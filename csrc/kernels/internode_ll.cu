#include "hip/hip_runtime.h"
#include "configs.cuh"
#include "exception.cuh"
#include "launch.cuh"
#include "ibgda_device.cuh"

#include "eager.cuh"
#include "utils.cuh"
#include <cstdint>
#include <cstdio>

namespace deep_ep {

namespace internode_ll {

template <int kNumThreads> __launch_bounds__(kNumThreads, 1)
__global__ void clean_low_latency_buffer(int* clean_0, int num_clean_int_0,
                                         int* clean_1, int num_clean_int_1) {
    // Barrier before cleaning (in case of unfinished chunked EP)
    nvshmemx_barrier_all_block();

    // Clean
    auto thread_id = static_cast<int>(threadIdx.x);
    #pragma unroll
    for (int i = thread_id; i < num_clean_int_0; i += kNumThreads)
        clean_0[i] = 0;
    #pragma unroll
    for (int i = thread_id; i < num_clean_int_1; i += kNumThreads)
        clean_1[i] = 0;

    // Barrier after cleaning (make sure the low-latency mode works fine)
    nvshmemx_barrier_all_block();
}

void clean_low_latency_buffer(int* clean_0, int num_clean_int_0,
                              int* clean_1, int num_clean_int_1,
                              hipStream_t stream) {
    constexpr int kNumThreads = 256;

    SETUP_LAUNCH_CONFIG(1, kNumThreads, stream);
    LAUNCH_KERNEL(&cfg, clean_low_latency_buffer<kNumThreads>,
                  clean_0, num_clean_int_0, clean_1, num_clean_int_1);
}

template <bool kUseFP8, bool kUseUE8M0, int kHidden, int kEager>
__global__ __launch_bounds__(1024, 1) void
dispatch(void* packed_recv_x, void* packed_recv_x_scales,
         int* packed_recv_src_info, int64_t* packed_recv_layout_range,
         int* packed_recv_count, int* per_rank_recv_count,
         int* cumulative_local_expert_recv_stats,
         int64_t* dispatch_wait_recv_cost_stats,
         void* rdma_recv_x, int* rdma_recv_count, void* rdma_x,
         const void* x, const int64_t* topk_idx,
         int* atomic_counter_per_expert, int* atomic_finish_counter_per_expert,
         int* next_clean, int num_next_clean_int,
         int num_tokens, int num_max_dispatch_tokens_per_rank,
         int num_topk, int num_experts, int rank, int num_ranks,
         int num_warp_groups, int num_warps_per_group,
         bool round_scale, int phases,
         int dispatch_round_n) {
    const auto sm_id = static_cast<int>(blockIdx.x);
    const auto thread_id = static_cast<int>(threadIdx.x);
    const auto warp_id = thread_id / 32, lane_id = get_lane_id();
    const auto num_sms = static_cast<int>(gridDim.x);
    const auto num_warps = num_warp_groups * num_warps_per_group;
    const auto num_local_experts = num_experts / num_ranks;
    const auto warp_group_id = warp_id / num_warps_per_group;
    const auto sub_warp_id = warp_id % num_warps_per_group;
    const auto responsible_expert_idx = sm_id * num_warp_groups + warp_group_id;

    // May extract UE8M0 from the scales
    using scale_t = std::conditional_t<kUseUE8M0, uint8_t, float>;
    using packed_t = std::conditional_t<kUseUE8M0, uint32_t, float>;
    EP_STATIC_ASSERT(sizeof(packed_t) % sizeof(scale_t) == 0, "Invalid vector length");

    // FP8 staffs
    constexpr int kNumPerChannels = 128;
    constexpr int num_scales = kHidden / kNumPerChannels;
    const size_t hidden_bytes = kHidden * (kUseFP8 ? sizeof(__hip_fp8_storage_t) : sizeof(hip_bfloat16));
    const size_t hidden_int4 = hidden_bytes / sizeof(int4);

    // Message package: hidden data, FP8 scales, index at source
    // NOTES: currently we have 3 reserved int fields for future use
    using vec_t = std::conditional_t<kUseFP8, int2, int4>;
    constexpr size_t num_bytes_per_msg_v = sizeof(int4) + (kUseFP8 ? (kHidden + num_scales * sizeof(float)) : (kHidden * sizeof(hip_bfloat16)));
    constexpr size_t num_bytes_per_msg = kEager != EAGER_OFF ? EXTEND_FOR_TAG_AND_ALIGN(num_bytes_per_msg_v + (kUseFP8 ? 0 : sizeof(int4)), AR_MSG_ALIGNMENT) : num_bytes_per_msg_v;
    // long message shall use extra int4 to jump the last tag of short message
    
    // WARNING!!! Two lines below must be consistent with msg def in config.hpp
    constexpr size_t dispatch_msg_max = sizeof(int4) + std::max(kHidden * sizeof(hip_bfloat16), kHidden + num_scales * sizeof(float));
    constexpr size_t combine_msg_max = num_scales * sizeof(nv_bfloat162) + kHidden * sizeof(hip_bfloat16);
    constexpr size_t msg_distance = kEager != EAGER_OFF ? EXTEND_FOR_TAG_AND_ALIGN(std::max(dispatch_msg_max, combine_msg_max) + sizeof(int4), AR_MSG_LONG_ALIGNMENT) : num_bytes_per_msg_v;
    constexpr size_t short_msg_len = sizeof(int4) + kHidden + num_scales * sizeof(float); // FP8 dispatch msg len, used for tag jump position
    //constexpr size_t long_msg_len = sizeof(int4) + kHidden * sizeof(hip_bfloat16); // BF16 dispatch / combine msg len;

    const size_t num_int4_per_msg = num_bytes_per_msg / sizeof(int4);
    EP_DEVICE_ASSERT(num_bytes_per_msg % sizeof(int4) == 0);
    // if (sm_id == 0 && thread_id == 0) {
    //     printf("[rank %d]: dispatch round 0x%08x short tag = 0x%08x, msg_distance = %lu, num_bytes_per_msg = %lu, short_msg = %lu\n", rank, ZTAG(dispatch_round_n), SHORT_TAG(dispatch_round_n), msg_distance, num_bytes_per_msg, short_msg_len);
    // }
    //if (sm_id == 0 && thread_id == 0) printf("[rank %d]: kEager = %d\n", rank, kEager);
    // Expert counts
    constexpr int kNumMaxWarpGroups = 32;
    __shared__ int shared_num_tokens_sent_per_expert[kNumMaxWarpGroups];

    // Sending phase
    if ((phases & LOW_LATENCY_SEND_PHASE) == 0)
        goto LOW_LATENCY_DISPATCH_RECV;

    // There are 2 kinds of warps in this part:
    // 1. The first-kind warps for FP8 cast and sending top-k tokens
    // 2. The last warp for reading `topk_idx` and count for per-expert information
    if (warp_id < num_warps - 1) {
        constexpr int kNumElemsPerRead = sizeof(int4) / sizeof(hip_bfloat16);
        EP_STATIC_ASSERT(kHidden % (32 * kNumElemsPerRead) == 0, "Invalid hidden");
        EP_STATIC_ASSERT(kNumElemsPerRead * 32 % kNumPerChannels == 0, "Invalid vectorization");
        const auto num_threads = (num_warps - 1) * 32;
        const size_t hidden_bf16_int4 = kHidden / kNumElemsPerRead;

        for (int token_idx = sm_id; token_idx < num_tokens; token_idx += num_sms) {
            const auto x_int4 = static_cast<const int4*>(x) + token_idx * hidden_bf16_int4;
            const auto rdma_x_src_idx = reinterpret_cast<int*>(static_cast<uint8_t*>(rdma_x) + token_idx * num_bytes_per_msg);
            const auto rdma_x_vec = reinterpret_cast<vec_t*>(reinterpret_cast<uint8_t*>(rdma_x_src_idx) + sizeof(int4));
            const auto rdma_x_scales = reinterpret_cast<float*>(reinterpret_cast<uint8_t*>(rdma_x_vec) + hidden_bytes);

            // Overlap top-k index read and source token index writes
            auto dst_expert_idx = warp_id < num_topk ? static_cast<int>(__ldg(topk_idx + token_idx * num_topk + warp_id)) : -1;
            //if (thread_id == 0) printf("[rank %d]: dispatch round 0x%08x copying token %d\n", rank, dispatch_round_n, token_idx);
            constexpr size_t scale_st_offset = sizeof(int4) * (kHidden == 8192 ? 2 : (kHidden < 4096 ? 0 : 1));
            constexpr size_t scale_st_float_offset = kEager != EAGER_OFF ? (scale_st_offset / sizeof(float)) : 0;
            constexpr size_t page_int4 = (PCIE_SEG_LEN - PCIE_TAIL_SZ) / sizeof(int4);
            constexpr size_t page_int2 = page_int4 << 1;
            constexpr size_t short_msg_int4 = short_msg_len / sizeof(int4);
            const size_t data_st_int4_offset = 
            kEager != EAGER_OFF ? 
                (kHidden == 8192 ?  
                    ((thread_id + 1) < page_int4 ? 0 : (thread_id + 1) < 2 * page_int4 ? 1 : (thread_id + 1) < short_msg_int4 ? 2 : (thread_id + 2) < 3 * page_int4 ? 3 : 4) :
                    (kHidden >= 4096 ? 
                        ((thread_id + 1) < page_int4 ? 0 : (thread_id + 1) < short_msg_int4 ? 1 : (thread_id + 2) < 2 * page_int4 ? 2 : (thread_id + 2) < 3 * page_int4 ? 3 : 4) : 
                        ((thread_id + 1) < short_msg_int4 ? 0 : (thread_id + 1) < page_int4 ? 1 : (thread_id + 2) < 2 * page_int4 ? 2 : (thread_id + 2) < 3 * page_int4 ? 3 : 4)))
                : 0;
            const size_t data_st_int2_offset = kEager != EAGER_OFF ? ((thread_id + 2) < page_int2 ? 0 : (thread_id + 2) < 2 * page_int2 ? 2 : 4) : 0;
            
            //const size_t int4_pos = thread_id + 1 + ((thread_id + 1) >= short_msg_int4 ? 1 : 0);
            //const size_t data_st_int4_offset = int4_pos < page_int4 ? 0 : (int4_pos < 2 * page_int4 ? 1 : (int4_pos < 3 * page_int4 ? 2 : 3));
#define DISPATCH_LD(ld_func, ptr) ((kEager != EAGER_OFF) ? LD_SHIFTED(ld_func, ptr, src_src_idx) : ld_func(ptr))
// #define DISPATCH_ST(PTR, VALUE) {\
//     if constexpr (kEager != EAGER_OFF) {\
//         if constexpr (kUseFP8) {\
//             auto sfp = SHIFTED_ADDR_P(PTR, rdma_x_src_idx);\
//             *sfp = VALUE;\
//             /*if (IS_PAGE_SUB_HEAD((PTR) + 1, rdma_x_src_idx, num_bytes_per_msg_v)) *reinterpret_cast<int*>(sfp + 1) = ZTAG(dispatch_round_n)*/;\
//         } else {\
//             auto sfp = SHIFTED_ADDR_PS(PTR, rdma_x_src_idx, short_msg_len);\
//             *sfp = VALUE;\
//             /*printf("[rank %d]: dispatch round 0x%08x st at offset %lu\n", rank, dispatch_round_n, PTR_DIFF(SHIFTED_ADDR_PS(PTR, rdma_x_src_idx, short_msg_len), rdma_x_src_idx))*/;\
//             /*if ((PTR_DIFF(sfp + 1, rdma_x_src_idx) & PCIE_SEG_LEN_MASK) == (PCIE_SEG_LEN - PCIE_TAIL_SZ) || PTR_DIFF(PTR + 1, rdma_x_src_idx) == num_bytes_per_msg_v) {\
//                 *reinterpret_cast<int*>(SHIFTED_ADDR_PS(PTR, rdma_x_src_idx, short_msg_len) + 1) = ZTAG(dispatch_round_n);\
//                 printf("[rank %d]: dispatch round 0x%08x st tag 0x%08x at offset %lu\n", rank, dispatch_round_n, ZTAG(dispatch_round_n), PTR_DIFF(SHIFTED_ADDR_PS(PTR, rdma_x_src_idx, short_msg_len) + 1, rdma_x_src_idx));\
//             }*/\
//         }\
//     } else {\
//         NORMAL_ST(PTR, VALUE);\
//     }\
// }

            if (thread_id == 0) {
                //DISPATCH_ST(rdma_x_src_idx, token_idx);
                *rdma_x_src_idx = token_idx; // first element never shift...
            }

            // FP8 cast
            EP_STATIC_ASSERT(hidden_bf16_int4 % 32 == 0, "Must use the full warp to reduce");
            #pragma unroll
            for (int i = thread_id; i < hidden_bf16_int4; i += num_threads) {
                // Read
                auto int4_value = __ldg(x_int4 + i);

                if constexpr (kUseFP8) {
                    // Calculate local amax
                    auto bf16_values = reinterpret_cast<hip_bfloat16*>(&int4_value);
                    float fp32_values[kNumElemsPerRead];
                    float amax = kFP8Margin, scale, scale_inv;
                    #pragma unroll
                    for (int j = 0; j < kNumElemsPerRead; ++ j) {
                        fp32_values[j] = static_cast<float>(bf16_values[j]);
                        amax = fmaxf(amax, fabsf(fp32_values[j]));
                    }

                    // Reduce amax and scale
                    EP_STATIC_ASSERT(kNumElemsPerRead * 32 / kNumPerChannels == 2, "Invalid vectorization");
                    amax = warp_reduce_max<16>(amax);
                    calculate_fp8_scales(amax, scale, scale_inv, round_scale);
                    if (lane_id == 0 or lane_id == 16) {
                        //DISPATCH_ST(&rdma_x_scales[i * kNumElemsPerRead / 128], scale_inv);
                        rdma_x_scales[i * kNumElemsPerRead / 128 + scale_st_float_offset] = scale_inv;
                    }

                    // Cast into send buffer
                    vec_t int2_value;
                    auto fp8x2_values = reinterpret_cast<__hip_fp8x2_storage_t*>(&int2_value);
                    #pragma unroll
                    for (int j = 0; j < kNumElemsPerRead; j += 2) {
                        float2 fp32x2 = {fp32_values[j] * scale, fp32_values[j + 1] * scale};
                        fp8x2_values[j / 2] = __hip_cvt_float2_to_fp8x2(fp32x2, __HIP_SATFINITE, __HIP_E4M3_FNUZ);
                    }
                    //DISPATCH_ST(&rdma_x_vec[i], int2_value);
                    rdma_x_vec[i + data_st_int2_offset] = int2_value;
                } else {
                    // Reinterpret-cast is for C++14 compatibility
                    //DISPATCH_ST(reinterpret_cast<int4*>(&rdma_x_vec[i]), int4_value);
                    reinterpret_cast<int4*>(rdma_x_vec)[i + data_st_int4_offset] = int4_value;
                }
            }
            if constexpr (kEager != EAGER_OFF) {
                //const auto warp_runs = hidden_bf16_int4 / 32;
                //const auto idle_warp_id = (warp_id + (num_warps - 1) - (warp_runs % (num_warps - 1))) % (num_warps - 1);
                if (warp_id == 0) {
                    PARALLEL_SET_TAG(rdma_x_src_idx, num_bytes_per_msg_v + (kUseFP8 ? 0 : sizeof(int4)), dispatch_round_n, thread_id, num_threads, NORMAL_ST);
                }
            }
            asm volatile("bar.sync 1, %0;" :: "r"(num_threads));
            //if (thread_id == 0) printf("[rank %d]: dispatch round 0x%08x token %d filled\n", rank, dispatch_round_n, token_idx);
            // Issue IBGDA sends
            if (dst_expert_idx >= 0) {
                int slot_idx = lane_id == 0 ? atomicAdd(atomic_counter_per_expert + dst_expert_idx, 1) : 0;
                slot_idx = __shfl_sync(0xffffffff, slot_idx, 0);
                const auto dst_rank = dst_expert_idx / num_local_experts;
                const auto dst_expert_local_idx = dst_expert_idx % num_local_experts;
                const auto src_ptr = reinterpret_cast<uint64_t>(rdma_x_src_idx);
                const auto dst_ptr = reinterpret_cast<uint64_t>(rdma_recv_x) +
                                     dst_expert_local_idx * num_ranks * num_max_dispatch_tokens_per_rank * msg_distance +
                                     rank * num_max_dispatch_tokens_per_rank * msg_distance +
                                     slot_idx * msg_distance;
                const auto dst_p2p_ptr = nvshmemi_get_p2p_ptr(dst_ptr, rank, dst_rank);
                if (dst_p2p_ptr == 0) {
                    if constexpr (kEager == EAGER_FULL) {
                        slot_idx = -1;
                    }
                    nvshmemi_ibgda_put_nbi_warp(dst_ptr, src_ptr, num_bytes_per_msg, dst_rank, dst_expert_local_idx, lane_id, slot_idx);
                    //if (lane_id == 0) printf("[rank %d]: dispatch round 0x%08x send token %d to rank %d expert %d\n", rank, dispatch_round_n, token_idx, dst_rank, dst_expert_idx);
                } else {
                    // NOTES: only 2 load iterations for 7K hidden with 8 unrolls
                    const auto* src_int4_ptr = reinterpret_cast<const int4*>(src_ptr);
                    const auto* dst_int4_ptr = reinterpret_cast<int4*>(dst_p2p_ptr);
                    UNROLLED_WARP_COPY(8, lane_id, num_int4_per_msg, dst_int4_ptr, src_int4_ptr, ld_nc_global, st_na_global);
                    WARP_SET_TAIL_TAG(dst_p2p_ptr, msg_distance - sizeof(int4), ZTAG(dispatch_round_n));
                    // if constexpr (kEager == EAGER_FULL) {
                    //     __syncwarp();
                    //     lane_id == 0 ? atomic_add_release_global(atomic_finish_counter_per_expert + dst_expert_idx, 1) : 0;
                    // }
                }

                __syncwarp();
                // Increase counter after finishing
                if constexpr (kEager != EAGER_FULL) {
                    lane_id == 0 ? atomic_add_release_global(atomic_finish_counter_per_expert + dst_expert_idx, 1) : 0;
                } 
                else {
                    if (lane_id == 0) {
                        auto cur_v = atomic_add_release_global(atomic_finish_counter_per_expert + dst_expert_idx, 1) + 1;
                        if (((cur_v) & (FINISHED_SUM_TAG - 1)) == 0) {
                            //printf("[rank %d]: dispatch round 0x%08x expert %d atom cnt %d\n", rank, dispatch_round_n, dst_expert_idx, cur_v);
                            atomic_counter_per_expert[dst_expert_idx] = 0;
                        }
                    }
                }
            }
        }
    } else if (warp_id == num_warps - 1) {
        EP_DEVICE_ASSERT(num_sms > 1);
        if (sm_id == 0) {
            // The first SM is also responsible for checking QPs
            EP_DEVICE_ASSERT(ibgda_get_state()->num_rc_per_pe >= num_local_experts);

            if constexpr (kEager != EAGER_FULL) {
                // The first SM is also responsible for cleaning the next buffer
                #pragma unroll
                for (int i = lane_id; i < num_next_clean_int; i += 32)
                    next_clean[i] = 0;

                // Notify before executing `int_p`
                __syncwarp();
                #pragma unroll
                for (int i = lane_id; i < num_experts; i += 32)
                    atomic_add_release_global(atomic_finish_counter_per_expert + i, FINISHED_SUM_TAG);
            }
        }

        // This SM should be responsible for some destination experts, read `topk_idx` for them
        int expert_count[kNumMaxWarpGroups] = {0};
        const auto expert_begin_idx = sm_id * num_warp_groups;
        const auto expert_end_idx = min(expert_begin_idx + num_warp_groups, num_experts);

        // Per lane count
        #pragma unroll 8
        for (int i = lane_id; i < num_tokens * num_topk; i += 32) {
            auto idx = static_cast<int>(__ldg(topk_idx + i));
            if (idx >= expert_begin_idx and idx < expert_end_idx)
                expert_count[idx - expert_begin_idx] ++;
        }
        
        //int remain_nvl_cnt = 0;
        //uint64_t p2p_cnt_ptr[kNumMaxWarpGroups];
        // Warp reduce
        #pragma unroll
        for (int i = expert_begin_idx; i < expert_end_idx; ++ i) {
            auto sum = warp_reduce_sum(expert_count[i - expert_begin_idx]);
            if (lane_id == 0) {
                if constexpr (kEager == EAGER_FULL) {
                    const auto dst_rank = i / num_local_experts;
                    const auto dst_expert_local_idx = i % num_local_experts;
                    const auto num_tokens_sent = ((-sum - 1) & 0xffff) | SHORT_TAG(dispatch_round_n);
                    //atomicAdd(atomic_finish_counter_per_expert, -1);
                    // EP_DEVICE_ASSERT(num_next_clean_int == num_experts); 
                    // next_clean[dst_expert_local_idx * num_ranks + dst_rank] = 0;
                    auto dst_ptr = reinterpret_cast<uint64_t>(rdma_recv_count + dst_expert_local_idx * num_ranks + rank);
                    auto dst_p2p_ptr = nvshmemi_get_p2p_ptr(dst_ptr, rank, dst_rank);
                    if (dst_p2p_ptr == 0) {
                        nvshmemi_ibgda_rma_p(reinterpret_cast<int*>(dst_ptr), num_tokens_sent, dst_rank, dst_expert_local_idx);
                    } else {
                        // atomic_add_release_global(atomic_finish_counter_per_expert + i, FINISHED_SUM_TAG - sum);
                        // if (ld_acquire_global(atomic_finish_counter_per_expert + i) % FINISHED_SUM_TAG == 0) {
                        st_release_sys_global(reinterpret_cast<int*>(dst_p2p_ptr), num_tokens_sent);
                            //if (-(num_tokens_sent | 0xffff0000)-1 != 0) printf("[rank %d]: dispatch round 0x%08x %d tokens to expert %d\n", rank, dispatch_round_n, -(num_tokens_sent | 0xffff0000)-1, i);
                        //     expert_count[i - expert_begin_idx] = 0;
                        // } else {
                        //     remain_nvl_cnt += 1;
                        //     p2p_cnt_ptr[i - expert_begin_idx] = dst_p2p_ptr;
                        //     expert_count[i - expert_begin_idx] = num_tokens_sent;
                        // }
                    }
                    //printf("[rank %d]: round 0x%x %d tokens to expert %d at rank %d\n", rank, dispatch_round_n, sum, i, i / num_local_experts);
                    auto cur_v = atomic_add_release_global(atomic_finish_counter_per_expert + i, FINISHED_SUM_TAG - sum) + FINISHED_SUM_TAG - sum;
                    
                    if ((cur_v & (FINISHED_SUM_TAG - 1)) == 0) {
                        //printf("[rank %d]: dispatch round 0x%08x expert %d atom cnt %d\n", rank, dispatch_round_n, i, cur_v);
                        atomic_counter_per_expert[i] = 0;
                    }
                } else {
                    shared_num_tokens_sent_per_expert[i - expert_begin_idx] = sum;
                    atomic_add_release_global(atomic_finish_counter_per_expert + i, FINISHED_SUM_TAG - sum);
                }
            }
        }
        // if constexpr (kEager == EAGER_FULL) {
        //     while (remain_nvl_cnt > 0) {
        //         for (int i = expert_begin_idx; i < expert_end_idx; ++i) {
        //             if (expert_count[i - expert_begin_idx] == 0) continue;
        //             if (ld_acquire_global(atomic_finish_counter_per_expert + i) % FINISHED_SUM_TAG == 0) {
        //                 st_release_sys_global(reinterpret_cast<int*>(p2p_cnt_ptr[i - expert_begin_idx]), expert_count[i - expert_begin_idx]);
        //                 //if (-(expert_count[i - expert_begin_idx] | 0xffff0000)-1 != 0) printf("[rank %d]: dispatch round 0x%08x %d tokens to expert %d\n", rank, dispatch_round_n, -(expert_count[i - expert_begin_idx] | 0xffff0000)-1, i);
        //                 remain_nvl_cnt -= 1;
        //             }
        //         }
        //     }
        // }
        // if constexpr (kEager == EAGER_FULL) { // flush all p2p write out
        //     if (lane_id == 0 && atomicAdd(atomic_finish_counter_per_expert + 1, 1) % num_sms == (num_sms - 1)) {
        //         __threadfence_system();
        //         //printf("[rank %d]: round 0x%x try to flush p2p write out\n", rank, dispatch_round_n);
        //     }
        // }
    }
    if constexpr (kEager != EAGER_FULL) {
        
        __syncthreads();
        // Issue count sends
        if (responsible_expert_idx < num_experts and sub_warp_id == 0 and lane_id == 0) {
            const auto dst_rank = responsible_expert_idx / num_local_experts;
            const auto dst_expert_local_idx = responsible_expert_idx % num_local_experts;
            const auto num_tokens_sent = shared_num_tokens_sent_per_expert[responsible_expert_idx - sm_id * num_warp_groups];

            // Wait local sends issued and send expert counts
            while (ld_acquire_global(atomic_finish_counter_per_expert + responsible_expert_idx) != FINISHED_SUM_TAG * 2);
            auto dst_ptr = reinterpret_cast<uint64_t>(rdma_recv_count + dst_expert_local_idx * num_ranks + rank);
            auto dst_p2p_ptr = nvshmemi_get_p2p_ptr(dst_ptr, rank, dst_rank);
            if (dst_p2p_ptr == 0) {
                if constexpr (kEager <= EAGER_OFF) {
                    nvshmemi_ibgda_amo_nonfetch_add(reinterpret_cast<int*>(dst_ptr), -num_tokens_sent - 1, dst_rank, dst_expert_local_idx);
                } else {
                    nvshmemi_ibgda_rma_p(reinterpret_cast<int*>(dst_ptr), -num_tokens_sent - 1, dst_rank, dst_expert_local_idx);
                }
            } else {
                st_release_sys_global(reinterpret_cast<int*>(dst_p2p_ptr), -num_tokens_sent - 1);
            }

            // Clean workspace for next use
            atomic_counter_per_expert[responsible_expert_idx] = 0;
            atomic_finish_counter_per_expert[responsible_expert_idx] = 0;

            // Clean `packed_recv_count`
            if (dst_rank == 0)
                packed_recv_count[dst_expert_local_idx] = 0;
        }
        __syncwarp();
    }

    // cg::this_grid().sync();
    // if (sm_id == 0 && thread_id == 0) printf("[rank %d]: round 0x%x dispatch send done\n", rank, dispatch_round_n);

    // Receiving phase
    LOW_LATENCY_DISPATCH_RECV:
    if ((phases & LOW_LATENCY_RECV_PHASE) == 0)
        return;

    // For send-and-recv kernels, we need a grid sync for making `packed_recv_count` visible
    if constexpr (kEager != EAGER_FULL)
        if (phases & LOW_LATENCY_SEND_PHASE)
            cg::this_grid().sync();

    // Receiving and packing
    if (responsible_expert_idx < num_experts) {
        const auto src_rank = responsible_expert_idx / num_local_experts;
        const auto local_expert_idx = responsible_expert_idx % num_local_experts;
        //const auto global_expert_idx = rank * num_local_experts + local_expert_idx;
        const auto rdma_recv_x_uint8 = static_cast<uint8_t*>(rdma_recv_x) +
                local_expert_idx * num_ranks * num_max_dispatch_tokens_per_rank * msg_distance +
                src_rank * num_max_dispatch_tokens_per_rank * msg_distance;
        const auto recv_x_int4 = static_cast<int4*>(packed_recv_x) +
                local_expert_idx * num_ranks * num_max_dispatch_tokens_per_rank * hidden_int4;
        const auto recv_src_info = packed_recv_src_info + local_expert_idx * num_ranks * num_max_dispatch_tokens_per_rank;
        const auto recv_range = packed_recv_layout_range + local_expert_idx * num_ranks;
        const auto token_src_bitmap = reinterpret_cast<int*>(packed_recv_layout_range) + local_expert_idx * num_ranks * num_max_dispatch_tokens_per_rank + src_rank * num_max_dispatch_tokens_per_rank; // used in EAGER_FULL
        const auto per_rank_recv_cnt_ptr = per_rank_recv_count + local_expert_idx * num_ranks + src_rank; // used in EAGER_FULL
        const auto ld_intra_node = (src_rank >> 3) == (rank >> 3); // TODO: identify intra node more elegantly
        const auto num_aligned_scales = align<int>(num_scales, sizeof(float) / sizeof(scale_t));
        const auto recv_x_scales = static_cast<scale_t*>(packed_recv_x_scales) + local_expert_idx * num_ranks * num_max_dispatch_tokens_per_rank * num_aligned_scales;

        auto recv_cnt_barrier = atomic_finish_counter_per_expert + num_experts + local_expert_idx;

        // Shared between sub-warps in warp groups
        __shared__ int shared_num_recv_tokens[kNumMaxWarpGroups], shared_recv_token_begin_idx[kNumMaxWarpGroups];

        // Wait tokens to arrive
        // NOTES: using sub-warp 1 to overlap with sub-warp 0
        int num_recv_tokens = 0, recv_token_begin_idx;
        auto rcv_cnt_ptr = rdma_recv_count + local_expert_idx * num_ranks + src_rank;
        //const auto normal_recv = kEager != EAGER_FULL || ld_intra_node;
        
        EP_DEVICE_ASSERT(num_warps_per_group > 1 and num_warp_groups < 15);
        if constexpr (kEager != EAGER_FULL) {
            if (sub_warp_id == 1 and lane_id == 0) {
                auto start_time = clock64();
                //if constexpr (kEager != EAGER_FULL) {
                    while ((num_recv_tokens = ld_acquire_sys_global(rdma_recv_count + local_expert_idx * num_ranks + src_rank)) == 0);
                // } else {
                //     while (((num_recv_tokens = ld_acquire_sys_global(rdma_recv_count + local_expert_idx * num_ranks + src_rank)) & 0xffff0000) != SHORT_TAG(dispatch_round_n));
                // }
                auto wait_recv_cost = clock64() - start_time;
                // if constexpr (kEager == EAGER_FULL) {
                //     num_recv_tokens = num_recv_tokens | 0xffff0000;
                // }
                num_recv_tokens = -num_recv_tokens - 1;
                //if (num_recv_tokens != 0) printf("[rank %d]: dispatch round 0x%08x expert %d from rank %d got %d tokens\n", rank, dispatch_round_n, global_expert_idx, src_rank, num_recv_tokens);
                recv_token_begin_idx = atomicAdd(packed_recv_count + local_expert_idx, num_recv_tokens);
                shared_num_recv_tokens[warp_group_id] = num_recv_tokens;
                shared_recv_token_begin_idx[warp_group_id] = recv_token_begin_idx;
                //if (kEager != EAGER_FULL) {
                    recv_range[src_rank] = pack2<int, int64_t>(num_recv_tokens, recv_token_begin_idx);
                // } else {
                //     atomicAdd(per_rank_recv_cnt_ptr + src_rank, num_recv_tokens);
                // }
                // Add stats for diagnosis
                if (cumulative_local_expert_recv_stats != nullptr)
                    atomicAdd(cumulative_local_expert_recv_stats + local_expert_idx, num_recv_tokens);
                if (dispatch_wait_recv_cost_stats != nullptr)
                    atomicAdd(reinterpret_cast<unsigned long long*>(dispatch_wait_recv_cost_stats + src_rank), wait_recv_cost);
            }
            asm volatile("bar.sync %0, %1;" :: "r"(warp_group_id + 2), "r"(num_warps_per_group * 32));
            num_recv_tokens = shared_num_recv_tokens[warp_group_id];
            recv_token_begin_idx = shared_recv_token_begin_idx[warp_group_id];
        } else {
            if (sub_warp_id == 1 && lane_id == 0) {
                //st_release_cta(packed_recv_count + local_expert_idx, 0);
                int cur_v = atomic_add_release_global(recv_cnt_barrier, 1);
                st_release_cta(per_rank_recv_cnt_ptr, 0);
                if (cur_v != num_ranks - 1) {
                    while (ld_nc_global(recv_cnt_barrier) != 0);
                } else {
                    st_na_global(packed_recv_count + local_expert_idx, 0);
                    st_na_global(recv_cnt_barrier, 0);
                }
            }
            asm volatile("bar.sync %0, %1;" :: "r"(warp_group_id + 2), "r"(num_warps_per_group * 32));
        }
        // Copy tokens
        EP_DEVICE_ASSERT(num_scales <= 64);
        int this_token_index_in_recv = 0;
        for (int i = sub_warp_id; (kEager != EAGER_FULL && i < num_recv_tokens) || (kEager == EAGER_FULL && (num_recv_tokens == 0 || i < -num_recv_tokens-1) && i < num_max_dispatch_tokens_per_rank); i += num_warps_per_group) {
            // Copy source info
            const auto src_src_idx = reinterpret_cast<int*>(rdma_recv_x_uint8 + i * msg_distance);
            
            if constexpr (kEager == EAGER_FULL) {
                //if (!ld_intra_node) {
                    WAIT_2BIT(src_src_idx, num_bytes_per_msg_v + (kUseFP8 ? 0 : sizeof(int4)), num_bytes_per_msg, lane_id, 32, dispatch_round_n, rcv_cnt_ptr, num_recv_tokens, i, ld_intra_node);
                    __syncwarp();
                    num_recv_tokens = warp_reduce_min(num_recv_tokens);
                    
                    if (num_recv_tokens != 0 && i >= (-num_recv_tokens-1)) {
                        //if (sub_warp_id == 0 && lane_id == 0 && (-num_recv_tokens-1) != 0) printf("[rank %d]: dispatch recv token expert %d get %d tokens from rank %d, done\n", rank, global_expert_idx, (-num_recv_tokens-1), src_rank);
                        break;
                    }
                //}
                
                // int offsets[4] = {4080, 8176, 12272, 14416};
                // char tmatch[2][30] = {"", " WRONG DISPATCH TAG!"};
                // if (lane_id < 4) {
                //     auto mtag = ld_nc_global(src_src_idx + offsets[lane_id] / sizeof(int));
                //     char *mtag_str = tmatch[mtag != ZTAG(dispatch_round_n)];
                //     printf("[rank %d]: dispatch round 0x%08x expert %3d from rank %d inter node tag 0x%08x at offset: %d %s\n", rank, dispatch_round_n, rank * num_local_experts + local_expert_idx, src_rank, mtag, offsets[lane_id], mtag_str);
                // }
                // __syncwarp();
                //if (!ld_intra_node) {
                    if (lane_id == 0) {
                        this_token_index_in_recv = atomic_add_release_global(packed_recv_count + local_expert_idx, 1);
                        atomicAdd(per_rank_recv_cnt_ptr, 1);
                        //printf("[rank %d]: dispatch recv token expert %d slot %d get rank %d token %d\n", rank, global_expert_idx, this_token_index_in_recv, src_rank, DISPATCH_LD(ld_nc_global, src_src_idx));
                        token_src_bitmap[i] = this_token_index_in_recv;
                    }
                    __syncwarp();
                    this_token_index_in_recv = __shfl_sync(0xffffffff, this_token_index_in_recv, 0);
                // } else {
                //     this_token_index_in_recv = recv_token_begin_idx + i;
                //     if (lane_id == 0) {
                //         token_src_bitmap[i] = this_token_index_in_recv;
                //     }
                // }
            } else if constexpr (kEager != EAGER_OFF) {
                WAIT_BIT(src_src_idx, num_bytes_per_msg_v + (kUseFP8 ? 0 : sizeof(int4)), num_bytes_per_msg, lane_id, 32, dispatch_round_n, ld_intra_node);
                __syncwarp();
            }
            
            if constexpr (kEager != EAGER_FULL) {
                this_token_index_in_recv = recv_token_begin_idx + i;
            }

            if (lane_id == 0)
                recv_src_info[this_token_index_in_recv] = DISPATCH_LD(ld_nc_global, src_src_idx);
            __syncwarp();

            // Copy data
            // NOTES: only 2 load iterations for 7K hidden with 7 unrolls
            const auto src_data = reinterpret_cast<int4*>(reinterpret_cast<uint8_t*>(src_src_idx) + sizeof(int4));
            const auto dst_data = recv_x_int4 + this_token_index_in_recv * hidden_int4;
            if constexpr (kEager != EAGER_OFF) {
                UNROLLED_WARP_COPY_SRC_AUTO_SHIFT(7, lane_id, hidden_int4, dst_data, src_data, src_src_idx, short_msg_len, ld_nc_global, st_na_global);
                // constexpr size_t page_int4 = (PCIE_SEG_LEN - PCIE_TAIL_SZ) / sizeof(int4);
                // if constexpr (kUseFP8) {
                //     size_t copy_done = 0;
                //     size_t offset = 0;
                //     while (copy_done < hidden_int4) {
                //         size_t to_copy = min(page_int4 - (offset == 0 ? 1 : 0), hidden_int4 - copy_done);
                //         auto src_data_ = src_data + copy_done + offset;
                //         auto dst_data_ = dst_data + copy_done;
                //         UNROLLED_WARP_COPY(7, lane_id, to_copy, dst_data_, src_data_, ld_nc_global, st_na_release);
                //         offset += 1;
                //         copy_done += to_copy;
                //     }
                // } else {
                //     UNROLLED_WARP_COPY_SRC_AUTO_SHIFT(7, lane_id, hidden_int4, dst_data, src_data, src_src_idx, short_msg_len, ld_nc_global, st_na_global);
                // }
            } else {
                UNROLLED_WARP_COPY(7, lane_id, hidden_int4, dst_data, src_data, ld_nc_global, st_na_global);
            }

            // Copy scales
            if constexpr (kUseFP8) {
                // Equivalent CuTe layout:
                //   (num_tokens, (num_packed, num_elems_per_pack)):(num_elems_per_pack, (num_tokens * num_elems_per_pack, 1))
                const auto src_scales = reinterpret_cast<float*>(reinterpret_cast<uint8_t*>(src_data) + hidden_bytes);
                const auto num_elems_per_pack = static_cast<int>(sizeof(packed_t) / sizeof(scale_t));
                const auto token_idx = this_token_index_in_recv;
                const auto token_stride = num_elems_per_pack;
                const auto pack_stride = num_ranks * num_max_dispatch_tokens_per_rank * num_elems_per_pack;
                if (lane_id < num_scales) {
                    const auto pack_idx = lane_id / num_elems_per_pack;
                    const auto elem_idx = lane_id % num_elems_per_pack;
                    auto scale = extract_required_scale_format<kUseUE8M0>(DISPATCH_LD(ld_nc_global, src_scales + lane_id));
                    recv_x_scales[token_idx * token_stride + pack_idx * pack_stride + elem_idx] = scale;
                }
                if (lane_id + 32 < num_scales) {
                    const auto pack_idx = (lane_id + 32) / num_elems_per_pack;
                    const auto elem_idx = (lane_id + 32) % num_elems_per_pack;
                    auto scale = extract_required_scale_format<kUseUE8M0>(DISPATCH_LD(ld_nc_global, src_scales + lane_id + 32));
                    recv_x_scales[token_idx * token_stride + pack_idx * pack_stride + elem_idx] = scale;
                }
            }
        }
        //if constexpr (kEager == EAGER_FULL) if ((-num_recv_tokens-1) != 0 && sub_warp_id < num_max_dispatch_tokens_per_rank && lane_id == 0) printf("[rank %d]: dispatch recv token expert %d get %d tokens from rank %d, subwarp %d done\n", rank, global_expert_idx, (-num_recv_tokens-1), src_rank, sub_warp_id);
        
    }
    // cg::this_grid().sync();
    // if (sm_id == 0 && thread_id == 0) printf("[rank %d]: dispatch round 0x%08x recv done\n", rank, dispatch_round_n);
}

void dispatch(void* packed_recv_x, void* packed_recv_x_scales,
              int* packed_recv_src_info, int64_t* packed_recv_layout_range,
              int* packed_recv_count, int* per_rank_recv_count,
              int* cumulative_local_expert_recv_stats,
              int64_t* dispatch_wait_recv_cost_stats,
              void* rdma_recv_x, int* rdma_recv_count, void* rdma_x,
              const void* x, const int64_t* topk_idx,
              int* next_clean, int num_next_clean_int,
              int num_tokens, int hidden, int num_max_dispatch_tokens_per_rank,
              int num_topk, int num_experts, int rank, int num_ranks,
              bool use_fp8, bool round_scale, bool use_ue8m0,
              void* workspace, int num_device_sms,
              hipStream_t stream, int phases,
              int dispatch_round_n, const int eager_opt) {
    constexpr int kNumMaxTopK = 9;
    const int num_warp_groups = ceil_div(num_experts, num_device_sms);
    const int num_warps_per_group = 32 / num_warp_groups;
    EP_HOST_ASSERT(num_warp_groups > 0 and num_warps_per_group > 0);
    EP_HOST_ASSERT(kNumMaxTopK + 1 <= num_warp_groups * num_warps_per_group);

    const auto num_warps = num_warp_groups * num_warps_per_group;
    const auto num_sms = ceil_div(num_experts, num_warp_groups);
    EP_HOST_ASSERT(num_topk <= kNumMaxTopK);

    // Workspace checks
    auto atomic_counter_per_expert = static_cast<int*>(workspace);
    auto atomic_finish_counter_per_expert = atomic_counter_per_expert + num_experts;
    EP_HOST_ASSERT(num_experts * sizeof(int) * 2 <= NUM_WORKSPACE_BYTES);
    //
    if ((eager_opt == EAGER_FULL) && (phases & LOW_LATENCY_SEND_PHASE)) {
        //CUDA_CHECK(hipMemsetAsync(workspace, 0, num_experts * sizeof(int), stream)); // only set atomic_counter_per_expert
        //CUDA_CHECK(hipMemsetAsync(packed_recv_count, 0, sizeof(int) * num_experts / num_ranks, stream));
        //CUDA_CHECK(hipMemsetAsync(per_rank_recv_count, 0, sizeof(int) * num_experts, stream));
    }
    // FP8 checks
    if (use_ue8m0)
        EP_HOST_ASSERT(round_scale and "UE8M0 SF requires `round_scale=True`");

#define DISPATCH_LAUNCH_CASE(eager, hidden) { \
auto dispatch_func = dispatch<false, false, hidden, eager>; \
if (use_fp8 and not use_ue8m0) \
    dispatch_func = dispatch<true, false, hidden, eager>; \
if (use_fp8 and use_ue8m0) \
    dispatch_func = dispatch<true, true, hidden, eager>; \
LAUNCH_KERNEL(&cfg, dispatch_func, \
              packed_recv_x, packed_recv_x_scales, \
              packed_recv_src_info, packed_recv_layout_range, \
              packed_recv_count, per_rank_recv_count, \
              cumulative_local_expert_recv_stats, \
              dispatch_wait_recv_cost_stats, \
              rdma_recv_x, rdma_recv_count, rdma_x, \
              x, topk_idx, \
              atomic_counter_per_expert, atomic_finish_counter_per_expert, \
              next_clean, num_next_clean_int, \
              num_tokens, num_max_dispatch_tokens_per_rank, \
              num_topk, num_experts, rank, num_ranks, \
              num_warp_groups, num_warps_per_group, \
              round_scale, phases, dispatch_round_n); } break

    SETUP_LAUNCH_CONFIG(num_sms, num_warps * 32, stream);

    #define EAGER_DISPATCH_LAUNCH_CASE(hidden) SWITCH_EAGER(DISPATCH_LAUNCH_CASE, hidden)
    
    SWITCH_HIDDEN(EAGER_DISPATCH_LAUNCH_CASE);
#undef DISPATCH_LAUNCH_CASE
}

template <int kNumSendUnrolls>
__forceinline__ __device__ int logfmt_encode(void* buffer, nv_bfloat162 *shared_amaxmin, const int& lane_id) {
    constexpr int kNumElemsPerInt4 = sizeof(int4) / sizeof(hip_bfloat16);
    constexpr float kLogThreshold = 0;
    constexpr float kMinClip = 32; // `== log_2(2 ^ (2 ^ 5))`
    constexpr int kNumBits = 10;
    constexpr int kNumValues = 1 << (kNumBits - 1);

    int4 int4_values[kNumSendUnrolls];
    const auto& uint32_values = reinterpret_cast<uint32_t*>(int4_values);
    const auto& bf162_values = reinterpret_cast<nv_bfloat162*>(int4_values);

    // Calculate lane offset
    const auto& ld_buffer = reinterpret_cast<uint32_t*>(static_cast<uint8_t*>(buffer) + lane_id * (kNumSendUnrolls * sizeof(int4)));
    const auto& st_buffer = reinterpret_cast<uint32_t*>(static_cast<uint8_t*>(buffer) + lane_id * (kNumSendUnrolls * sizeof(int4) * 10 / 16));

    // Local log amax
    auto bf162_amax = __hip_bfloat162(CUDART_ZERO_BF16, CUDART_ZERO_BF16);
    auto bf162_amin = __hip_bfloat162(CUDART_INF_BF16, CUDART_INF_BF16);
    uint32_t local_signs = 0;
    #pragma unroll
    for (int k = 0; k < kNumSendUnrolls * kNumElemsPerInt4 / 2; ++ k) {
        // TODO: eliminate bank conflicts
        uint32_values[k] = ld_buffer[k];
        local_signs |= ((uint32_values[k] >> 15) & 1) << (k * 2);
        local_signs |= ((uint32_values[k] >> 31) & 1) << (k * 2 + 1);
        uint32_values[k] &= 0x7fff7fff;

        bf162_amax = __hmax2(bf162_amax, bf162_values[k]);
        bf162_amin = __hmin2(bf162_amin, bf162_values[k]);
    }

    // Reduce per 128 channels
    // TODO: figure out how hardware do 2-byte min/max
    auto amax = std::max(static_cast<float>(bf162_amax.x), static_cast<float>(bf162_amax.y));
    auto amin = std::min(static_cast<float>(bf162_amin.x), static_cast<float>(bf162_amin.y));
    constexpr static int kNumLanesToReduce = 128 * sizeof(hip_bfloat16) / (kNumSendUnrolls * sizeof(int4));
    amax = warp_reduce_max<kNumLanesToReduce>(amax);
    amin = warp_reduce_min<kNumLanesToReduce>(amin);

    // Write min/max into the shared memory
    if (shared_amaxmin != nullptr)
        *shared_amaxmin = __hip_bfloat162(amax, amin);
    __syncwarp();

    // Calculate log amin/amax float
    const auto& log_amax = log2f_approx(amax);
    const auto& log_amin = fmaxf(log2f_approx(amin), log_amax - kMinClip);
    const bool& enable_cast = warp_reduce_and<kNumLanesToReduce, true>(log_amax < kLogThreshold and log_amin < log_amax);

    // Case into LogFMT-10 if satisfied
    if (enable_cast) {
        const auto step = (log_amax - log_amin) / static_cast<float>(kNumValues - 2);
        const auto step_inv = 1.0f / step;
        const auto rounding = 2.0f - log2f_approx((1.0f + exp2f_approx(step)) * 0.5f) * step_inv;
        const auto fused_rounding = rounding - log_amin * step_inv;

        // Pack every 256 bits into 160 bits
        EP_STATIC_ASSERT(kNumSendUnrolls == 2 or kNumSendUnrolls == 4, "kNumSendUnrolls == 2 or 4 only");
        uint32_t encoded[kNumElemsPerInt4 * 2];
        #pragma unroll 1
        for (int i = 0; i < kNumSendUnrolls / 2; ++ i) {
            #pragma unroll
            for (int k = 0; k < kNumElemsPerInt4; ++ k) {
                const auto& [x, y] = __bfloat1622float2(bf162_values[i * kNumElemsPerInt4 + k]);
                encoded[k * 2 + 0] = __float2uint_rd(fmaxf(log2f_approx(x) * step_inv + fused_rounding, 0));
                encoded[k * 2 + 1] = __float2uint_rd(fmaxf(log2f_approx(y) * step_inv + fused_rounding, 0));
            }
            st_buffer[i * 5 + 0] = (encoded[ 0] >> 0) | (encoded[ 1] << 9) | (encoded[ 2] << 18) | (encoded[ 3] << 27);
            st_buffer[i * 5 + 1] = (encoded[ 3] >> 5) | (encoded[ 4] << 4) | (encoded[ 5] << 13) | (encoded[ 6] << 22) | (encoded[7]  << 31);
            st_buffer[i * 5 + 2] = (encoded[ 7] >> 1) | (encoded[ 8] << 8) | (encoded[ 9] << 17) | (encoded[10] << 26);
            st_buffer[i * 5 + 3] = (encoded[10] >> 6) | (encoded[11] << 3) | (encoded[12] << 12) | (encoded[13] << 21) | (encoded[14] << 30);
            st_buffer[i * 5 + 4] = (encoded[14] >> 2) | (encoded[15] << 7) | ((i == 0) ? (local_signs << 16) : (local_signs & 0xffff0000u));
        }
        tma_store_fence();
        __syncwarp();
    }

    // Return TMA copy bytes
    return enable_cast ? (32 * (kNumSendUnrolls * sizeof(int4) * 8 * 10 / 16 / 8)):
                         (32 * (kNumSendUnrolls * sizeof(int4)));
}

template <int kNumLanes, int kNumSendUnrolls, int kNumRecvUnrolls>
__forceinline__ __device__ void logfmt_check_amaxmin(uint8_t* meta_buffer, float2* shared_log_amax,
                                                     float2* shared_log_amin, int* shared_cast_info,
                                                     const int lane_id) {
    constexpr float kLogThreshold = 0;
    constexpr float kMinClip = 32; // `== log_2(2 ^ (2 ^ 5))`

    bool enable_cast = true;
    if (lane_id < kNumLanes) {
        // Calculate log amin/amax float
        auto amaxmin2 = reinterpret_cast<uint64_t*>(meta_buffer)[lane_id];
        const auto& bf162_amaxmin = reinterpret_cast<__hip_bfloat162*>(&amaxmin2);
        float log_amax[2], log_amin[2];
        #pragma unroll
        for (int i = 0; i < 2; ++ i) { 
            auto amax = static_cast<float>(bf162_amaxmin[i].x);
            auto amin = static_cast<float>(bf162_amaxmin[i].y);
            log_amax[i] = log2f_approx(amax);
            log_amin[i] = amin == 0 ? log_amax[i] - kMinClip : fmaxf(log2f_approx(amin), log_amax[i] - kMinClip);
            enable_cast = enable_cast and log_amax[i] < kLogThreshold and log_amin[i] < log_amax[i];
        }
        shared_log_amax[lane_id] = make_float2(log_amax[0], log_amax[1]);
        shared_log_amin[lane_id] = make_float2(log_amin[0], log_amin[1]);
    }

    const auto& casted = warp_reduce_and<kNumSendUnrolls>(enable_cast) ? 1u << (lane_id / kNumRecvUnrolls): 0u;
    const auto& num_casted_prefix = __popc(warp_reduce_or<kNumRecvUnrolls, true>(casted) & ((1u << (lane_id / kNumRecvUnrolls)) - 1));

    if (lane_id < kNumLanes and lane_id % kNumRecvUnrolls == 0)
        shared_cast_info[lane_id / kNumRecvUnrolls] = (num_casted_prefix << 1) | (casted ? 1u : 0u);
    __syncwarp();
}

template <int kNumRecvUnrolls>
__forceinline__ __device__ void decode_and_accumulate(uint32_t* ld_buffer, float* accum,
                                                      const float& log_amax, const float& log_amin,
                                                      const bool& enable_cast, const float& weight) {
    if (enable_cast) {
        constexpr int kNumBits = 10;
        constexpr int kNumValues = 1 << (kNumBits - 1);

        const auto& step = (log_amax - log_amin) / static_cast<float>(kNumValues - 2);
        auto decode = [=](const uint32_t &encoded, const uint32_t &sign) {
            const auto decoded = encoded == 0 ? .0f : exp2f_approx((encoded - 1) * step + log_amin);
            return sign ? -decoded : decoded;
        };

        EP_STATIC_ASSERT(kNumRecvUnrolls == 2 or kNumRecvUnrolls == 4, "kNumRecvUnrolls == 2 or 4 only");
        #pragma unroll
        for (int i = 0; i < kNumRecvUnrolls / 2; ++ i) {
            uint32_t concat[6];
            concat[0] = ld_buffer[i * 5];
            #pragma unroll
            for (int k = 1; k < 5; ++ k)
                concat[k] = (ld_buffer[i * 5 + k - 1] >> (32 - k * 5)) | (ld_buffer[i * 5 + k] << (k * 5));
            concat[5] = ld_buffer[i * 5 + 4] >> 7;

            const uint32_t& local_signs = ld_buffer[i * 5 + 4] >> 16;
            #pragma unroll
            for (int k = 0; k < 5; ++ k) {
                accum[i * 16 + k * 3 + 0] += decode((concat[k] >>  0) & 0x1ff, (local_signs >> (k * 3 + 0)) & 1) * weight;
                accum[i * 16 + k * 3 + 1] += decode((concat[k] >>  9) & 0x1ff, (local_signs >> (k * 3 + 1)) & 1) * weight;
                accum[i * 16 + k * 3 + 2] += decode((concat[k] >> 18) & 0x1ff, (local_signs >> (k * 3 + 2)) & 1) * weight;
            }
            accum[i * 16 + 15] += decode(concat[5] & 0x1ff, (local_signs >> 15) & 1) * weight;
        }
    } else {
        #pragma unroll
        for (int k = 0; k < kNumRecvUnrolls * 4; ++ k) {
            auto bf16_pack = *reinterpret_cast<__hip_bfloat162*>(ld_buffer + k);
            accum[k * 2 + 0] += static_cast<float>(bf16_pack.x) * weight;
            accum[k * 2 + 1] += static_cast<float>(bf16_pack.y) * weight;
        }
    }
}

template <bool kUseLogFMT, int kHidden, int kNumMaxTopk, int kNumMaxUnrolls, int kEager>
__global__ __launch_bounds__(1024, 1) void
combine(void* combined_x,
        void* rdma_recv_x, int* rdma_recv_flag, void* rdma_send_x,
        const void* x, const int64_t* topk_idx, const float* topk_weights,
        const int* per_rank_src_count,
        const int* src_info, const int64_t* layout_range,
        int64_t* combine_wait_recv_cost_stats,
        int* next_clean, int num_next_clean_int,
        int* atomic_clean_flag,
        int num_combined_tokens, int hidden, int num_topk,
        int num_max_dispatch_tokens_per_rank,
        int num_experts, int rank, int num_ranks,
        int num_warp_groups, int num_warps_per_group,
        int phases, bool zero_copy, int combine_round_n) {
    const auto sm_id = __shfl_sync(0xffffffff, static_cast<int>(blockIdx.x), 0);
    const auto num_sms = __shfl_sync(0xffffffff, static_cast<int>(gridDim.x), 0);
    const auto thread_id = static_cast<int>(threadIdx.x);
    const auto num_threads = __shfl_sync(0xffffffff, static_cast<int>(blockDim.x), 0);
    const auto warp_id = __shfl_sync(0xffffffff, thread_id / 32, 0), lane_id = get_lane_id();
    const auto num_local_experts = num_experts / num_ranks;
    const auto warp_group_id = warp_id / num_warps_per_group;
    const auto sub_warp_id = warp_id % num_warps_per_group;
    const auto responsible_expert_idx = sm_id * num_warp_groups + warp_group_id;

    extern __shared__ __align__(1024) uint8_t smem_buffer[];

    // Data type staffs
    constexpr int kNumElemsPerInt4 = sizeof(int4) / sizeof(hip_bfloat16);
    constexpr int64_t hidden_bf16_int4 = kHidden / kNumElemsPerInt4;

    // Use different unroll factors for send and recv phases
    constexpr int kNumSendUnrolls = kHidden % (32 * 4 * sizeof(int4) / sizeof(hip_bfloat16)) == 0 ? 4 : 2;
    constexpr int kNumRecvUnrolls = 2;
    constexpr int hidden_bf16_int4_pad = align(static_cast<int>(hidden_bf16_int4), 32 * kNumSendUnrolls);
    EP_STATIC_ASSERT(kHidden % (32 * 2 * sizeof(int4) / sizeof(hip_bfloat16)) == 0, "Invalid hidden");
    EP_STATIC_ASSERT(kNumSendUnrolls <= kNumMaxUnrolls and kNumRecvUnrolls <= kNumMaxUnrolls, "Invalid unrolls");
    EP_STATIC_ASSERT(hidden_bf16_int4 % kNumSendUnrolls == 0, "Invalid hidden");
    EP_STATIC_ASSERT(kNumSendUnrolls >= kNumRecvUnrolls, "Invalid unroll factors");

    // Message package
    EP_STATIC_ASSERT(kHidden % 128 == 0, "Invalid hidden");
    constexpr int kNumDivisions = kHidden / 128;
    constexpr int kNumMetaBytes = kNumDivisions * sizeof(nv_bfloat162);
    constexpr size_t num_bytes_per_slot_v = kHidden * sizeof(hip_bfloat16) + kNumMetaBytes;
    constexpr size_t num_bytes_per_slot = kEager != EAGER_OFF ? EXTEND_FOR_TAG_AND_ALIGN(num_bytes_per_slot_v, AR_MSG_ALIGNMENT) : num_bytes_per_slot_v;
    
    // WARNING!!! Two lines below must be consistent with msg def in config.hpp
    constexpr int kNumPerChannels = 128;
    constexpr int num_scales = kHidden / kNumPerChannels;
    constexpr size_t dispatch_msg_max = sizeof(int4) + std::max(kHidden * sizeof(hip_bfloat16), kHidden + num_scales * sizeof(float));
    constexpr size_t combine_msg_max = num_scales * sizeof(nv_bfloat162) + kHidden * sizeof(hip_bfloat16);
    constexpr size_t msg_distance = kEager != EAGER_OFF ? EXTEND_FOR_TAG_AND_ALIGN(std::max(dispatch_msg_max, combine_msg_max) + sizeof(int4), AR_MSG_LONG_ALIGNMENT) : num_bytes_per_slot_v;

    // if (sm_id == 0 && thread_id == 0) {
    //     printf("[rank %d]: combine round 0x%08x short tag = 0x%08x, msg_distance = %lu, num_bytes_per_slot = %lu\n", rank, ZTAG(combine_round_n), SHORT_TAG(combine_round_n), msg_distance, num_bytes_per_slot);
    // }

    constexpr size_t short_msg_len = sizeof(int4) + kHidden + num_scales * sizeof(float); // FP8 dispatch msg len, used for tag jump position
    constexpr size_t short_msg_ext_len = SHIFTED_ADDR(short_msg_len);
    constexpr size_t long_msg_len = sizeof(int4) + kHidden * sizeof(hip_bfloat16); // BF16 dispatch / combine msg len;
    //constexpr size_t long_msg_len_int4 = long_msg_len / sizeof(int4);
    constexpr size_t long_msg_ext_len = EXTEND_FOR_TAG_AND_ALIGN(long_msg_len, sizeof(int4));
    constexpr size_t long_msg_ext_len_int4 = long_msg_ext_len / sizeof(int4);

    constexpr int kEager_combine = kUseLogFMT ? EAGER_OFF : kEager; // logfmt does not support eager combine
    constexpr int MAX_PAGES_DIV4 = 1;
    constexpr int MAX_PAGES = MAX_PAGES_DIV4 << 2;
    EP_STATIC_ASSERT(MAX_PAGES_DIV4 <= 31, "pages can not be dealt by warp");
    EP_STATIC_ASSERT(kHidden * sizeof(hip_bfloat16) <= MAX_PAGES * PCIE_SEG_LEN, "hidden too large"); // max 4 page, 16K bytes, 8k bf16 hidden
    int4 __tail_tags_int4[MAX_PAGES_DIV4 + 1];
    __tail_tags_int4[MAX_PAGES_DIV4].x = ZTAG(combine_round_n);
    int *__tail_tags = reinterpret_cast<int*>(__tail_tags_int4); // store tag position values

    //constexpr int page_int4 = (PCIE_SEG_LEN - PCIE_TAIL_SZ) / sizeof(int4);
    //constexpr int page_int4_slots = PCIE_SEG_LEN / sizeof(int4);

    EP_STATIC_ASSERT(num_bytes_per_slot % sizeof(int4) == 0, "Invalid vectorization");

    // Sending phase
    if ((phases & LOW_LATENCY_SEND_PHASE) == 0)
        goto LOW_LATENCY_COMBINE_RECV;

    if constexpr (kEager_combine != EAGER_FULL) {
        // Clean up next buffer
        if (sm_id == 0 and warp_group_id == 0 and sub_warp_id == 0) {
            #pragma unroll
            for (int i = lane_id; i < num_next_clean_int; i += 32)
                next_clean[i] = 0;

            // Notify before executing `int_p`
            __syncwarp();
            
            if (lane_id == 0) {
                atomic_add_release_global(atomic_clean_flag, num_experts);
            }
        }
    }

    // Issue IBGDA sends
    if (responsible_expert_idx < num_experts) {
        const auto dst_rank = responsible_expert_idx / num_local_experts;
        const auto local_expert_idx = responsible_expert_idx % num_local_experts;
        const auto global_expert_idx = rank * num_local_experts + local_expert_idx;
        const auto layout = kEager != EAGER_FULL ? __ldg(layout_range + local_expert_idx * num_ranks + dst_rank) : 0l;
        const auto layout_bitmap_ptr = reinterpret_cast<const int*>(layout_range) + local_expert_idx * num_ranks * num_max_dispatch_tokens_per_rank + dst_rank * num_max_dispatch_tokens_per_rank; // used in EAGER_FULL
        const int num_tokens_from_this_exp_to_dstrank = kEager != EAGER_FULL ? 0 : __ldg(per_rank_src_count + local_expert_idx * num_ranks + dst_rank); // used in EAGER_FULL

        const auto local_x = static_cast<const int4*>(x) +
                local_expert_idx * num_ranks * num_max_dispatch_tokens_per_rank * hidden_bf16_int4;
        const auto local_src_info = src_info + local_expert_idx * num_ranks * num_max_dispatch_tokens_per_rank;
        const auto rdma_send_x_vec = static_cast<uint8_t*>(rdma_send_x) +
                local_expert_idx * num_ranks * num_max_dispatch_tokens_per_rank * num_bytes_per_slot;

        // Unpack layout
        int offset = 0, num_tokens_to_send, begin_idx, end_idx;
        if constexpr (kEager != EAGER_FULL) {
            unpack2(layout, num_tokens_to_send, offset);
        }

        begin_idx = sub_warp_id + offset;
        if constexpr (kEager == EAGER_FULL) {
            end_idx = num_tokens_from_this_exp_to_dstrank;
        } else {
            end_idx = offset + num_tokens_to_send;
        }
        int token_idx; // position level index

        // TMA stuffs
        constexpr int kNumTMABufferBytes = sizeof(int4) * 32 * kNumSendUnrolls;
        constexpr int kNumStages = 3;
        constexpr int kNumPrefetch = 1;
        EP_STATIC_ASSERT(kNumStages == 3 and kNumPrefetch == 1, "Invalid stages");

        auto smem_ptr = smem_buffer + warp_id * (kNumStages * (kNumTMABufferBytes + 16) + kNumMetaBytes);
        uint32_t tma_phase = 0;
        auto tma_buffers   = PatternVisitor([=](const int& i) { return reinterpret_cast<int4*>(smem_ptr + i * (kNumTMABufferBytes + 16)); });
        auto full_barriers = PatternVisitor([=](const int& i) { return reinterpret_cast<uint64_t*>(smem_ptr + i * (kNumTMABufferBytes + 16) + kNumTMABufferBytes); });
        auto meta_buffers  = kUseLogFMT ? reinterpret_cast<nv_bfloat162*>(smem_ptr + kNumStages * (kNumTMABufferBytes + 16)) : nullptr;
        EP_STATIC_ASSERT(kNumSendUnrolls * kNumStages <= 12, "TMA buffer size exceed limit");

        // Initialize m-barriers
        if (lane_id < kNumStages) {
            mbarrier_init(full_barriers[lane_id], 1);
            fence_view_async_shared();
            fence_barrier_init();
        }
        __syncwarp();

        constexpr int kNumIters = hidden_bf16_int4_pad / (32 * kNumSendUnrolls);
        auto tma_load_and_arrive = [&](const int& stage_idx, const int4* gmem_ptr, const int& num_bytes) {
            tma_load_1d(tma_buffers[stage_idx], gmem_ptr, full_barriers[stage_idx], num_bytes);
            mbarrier_arrive_and_expect_tx(full_barriers[stage_idx], num_bytes);
        };
        auto get_num_tma_bytes = [&](const int& offset_int4) {
            return min(kNumTMABufferBytes, static_cast<int>((hidden_bf16_int4 - offset_int4) * sizeof(int4)));
        };

        // Issue IBGDA send
        for (int __token_idx = begin_idx; __token_idx < end_idx; __token_idx += num_warps_per_group) { // sequence level index
            if constexpr (kEager == EAGER_FULL) {
                token_idx = lane_id == 0 ? __ldg(layout_bitmap_ptr + __token_idx) : 0;
                token_idx = __shfl_sync(0xffffffff, token_idx, 0);
            } else {
                token_idx = __token_idx;
            }
            //EP_DEVICE_ASSERT(token_idx >= 0 && token_idx < num_ranks * num_max_dispatch_tokens_per_rank);
            auto message_idx = __token_idx - offset;

            const auto x_int4 = local_x + token_idx * hidden_bf16_int4;
            const auto rdma_send_type_row = reinterpret_cast<int*>(rdma_send_x_vec + token_idx * num_bytes_per_slot);
            const auto rdma_send_x_vec_row = reinterpret_cast<uint8_t*>(rdma_send_type_row);

            // Copy directly to local rank, or copy to buffer and issue RDMA
            const auto src_idx = __shfl_sync(0xffffffff, __ldg(local_src_info + token_idx), 0);
            const auto buf_ptr = reinterpret_cast<int64_t>(rdma_send_x_vec_row);
            const auto dst_ptr = reinterpret_cast<uint64_t>(rdma_recv_x) + (global_expert_idx * num_max_dispatch_tokens_per_rank + src_idx) * msg_distance;
            const auto dst_p2p_ptr = nvshmemi_get_p2p_ptr(dst_ptr, rank, dst_rank);
            const auto intra_node = (dst_p2p_ptr != 0);
            //if (!intra_node) printf("[rank %d]: an inter-node combine send\n", rank);
            int num_send_bytes = hidden * sizeof(hip_bfloat16);

            // eager mode disable zero copy feature...
            if ((kEager_combine != EAGER_OFF) or (not zero_copy or dst_p2p_ptr != 0)) {
                // Read from `cpy_src_int4_ptr` and copy into `cpy_dst_int4_ptr`
                const auto cpy_src_int4_ptr = (zero_copy && kEager_combine == EAGER_OFF) ? reinterpret_cast<int4*>(buf_ptr) : x_int4;
                const auto cpy_dst_int4_ptr = dst_p2p_ptr == 0 ? reinterpret_cast<int4*>(buf_ptr) : reinterpret_cast<int4*>(dst_p2p_ptr);

                // Prefetch
                if (elect_one_sync(lane_id))
                    tma_load_and_arrive(0, cpy_src_int4_ptr, get_num_tma_bytes(0));
                __syncwarp();

                int tma_offset_bytes = kNumMetaBytes;
                #pragma unroll
                for (int i = lane_id * kNumSendUnrolls, iter_idx = 0; i < hidden_bf16_int4_pad; i += 32 * kNumSendUnrolls, ++ iter_idx) {
                    // Load the next iteration
                    const int& stage_idx = iter_idx % kNumStages;
                    const int& next_stage_idx = (iter_idx + 1) % kNumStages;
                    if (iter_idx + 1 < kNumIters and elect_one_sync(lane_id)) {
                        tma_store_wait<kNumStages - kNumPrefetch - 1>();
                        //const auto& offset_int4 = i + 32 * kNumSendUnrolls;
                        const auto& offset_int4 = (iter_idx + 1) * 32 * kNumSendUnrolls; // why offset_int4 is runtime dynamic?
                        tma_load_and_arrive(next_stage_idx, cpy_src_int4_ptr + offset_int4, get_num_tma_bytes(offset_int4));
                    }
                    __syncwarp();

                    // Wait the current TMA arrival
                    EP_STATIC_ASSERT(kNumStages < 32, "Too many stages");
                    mbarrier_wait<true>(full_barriers[stage_idx], tma_phase, stage_idx);
                    if constexpr (kUseLogFMT) {
                        // Cast if possible
                        constexpr int kNumInt4PerDivision = 128 / kNumElemsPerInt4;
                        int num_tma_bytes = logfmt_encode<kNumSendUnrolls>(
                            tma_buffers[stage_idx],
                            // NOTES: only the leader lane will write the result
                            (i % kNumInt4PerDivision == 0) ? meta_buffers + i / kNumInt4PerDivision : nullptr,
                            lane_id);
                        if (elect_one_sync(lane_id))
                            tma_store_1d(tma_buffers[stage_idx], reinterpret_cast<uint8_t*>(cpy_dst_int4_ptr) + tma_offset_bytes, num_tma_bytes);
                        tma_offset_bytes += num_tma_bytes;
                    } else {
                        // BF16 original values
#define COMBINE_SEND_TMA(tma_func, smem_ptr, gmem_ptr, bytes) TMA_AUTO_TAG(tma_func, smem_ptr, gmem_ptr, bytes, cpy_dst_int4_ptr, __tail_tags, short_msg_ext_len, combine_round_n, kEager_combine)
                        if constexpr (kEager_combine == EAGER_OFF) {
                            if (elect_one_sync(lane_id)) {
                                auto w_offset_int4 = iter_idx * kNumSendUnrolls * 32; // i is dynamic...
                                tma_store_1d(tma_buffers[stage_idx], cpy_dst_int4_ptr + w_offset_int4, get_num_tma_bytes(w_offset_int4));
                            }
                        } else {
                            auto w_offset_int4 = iter_idx * kNumSendUnrolls * 32; // i is dynamic...
                            COMBINE_SEND_TMA(tma_store_1d, tma_buffers[stage_idx], cpy_dst_int4_ptr + w_offset_int4, get_num_tma_bytes(w_offset_int4));
                        }
                        // if ((kEager_combine != EAGER_OFF) ? (lane_id == 0) : elect_one_sync(lane_id)) { // force lane 0 to issue tma
                        //     auto w_offset_int4 = iter_idx * kNumSendUnrolls * 32; // i is dynamic...
                        //     COMBINE_SEND_TMA(tma_store_1d, tma_buffers[stage_idx], cpy_dst_int4_ptr + w_offset_int4, get_num_tma_bytes(w_offset_int4));
                        //     //tma_store_1d(tma_buffers[stage_idx], cpy_dst_int4_ptr + w_offset_int4, get_num_tma_bytes(w_offset_int4));
                        // }
                    }
                    __syncwarp();
                }

                // Store metadata (min/max values) for LogFMT
                if constexpr (kUseLogFMT) {
                    num_send_bytes = tma_offset_bytes;
                    if (elect_one_sync(lane_id))
                        tma_store_1d(meta_buffers, cpy_dst_int4_ptr, kNumMetaBytes);
                }

                if constexpr (kEager_combine != EAGER_OFF) {
                    // inter_node: write tag save value
                    
                    if (true) {
                        // reduce or because tags may be stored in different lanes, maybe there is a better way, why just use lane 0 to do tma? If so, reduce can be replaced by shfl_sync
                        __syncwarp();
                        #pragma unroll
                        for (int __pn = 0; __pn < MAX_PAGES; ++__pn) {
                            //__tail_tags[__pn] = warp_reduce_or(__tail_tags[__pn]);
                            __tail_tags[__pn] = __shfl_sync(0xffffffff, __tail_tags[__pn], 0);
                        }
                        __tail_tags[MAX_PAGES + 1] = __shfl_sync(0xffffffff, __tail_tags[MAX_PAGES + 1], 1);

                        //EP_DEVICE_ASSERT(lane_id == get_lane_id());
                        EP_STATIC_ASSERT((kHidden * sizeof(hip_bfloat16)) % sizeof(int4) == 0, "combine message len (no logfmt) shall be a multiple of int4");
                        if (lane_id < MAX_PAGES_DIV4 + 1) {
                            auto target_ptr = cpy_dst_int4_ptr + (lane_id < MAX_PAGES_DIV4 ? (hidden_bf16_int4 + lane_id) : (long_msg_ext_len_int4));
                            //EP_DEVICE_ASSERT((target_ptr + 1 - cpy_dst_int4_ptr) * sizeof(int4) <= msg_distance);
                            //printf("[rank %d]: exp %d send back rank %d token %d, put %d stored value and last tag, {0x%08x, 0x%08x, 0x%08x, 0x%08x} offset %lu\n", rank, global_expert_idx, dst_rank, src_idx, lane_id, __tail_tags_int4[lane_id].x, __tail_tags_int4[lane_id].y, __tail_tags_int4[lane_id].z, __tail_tags_int4[lane_id].w, (target_ptr - cpy_dst_int4_ptr) * sizeof(int4));
                            // if (lane_id == MAX_PAGES_DIV4) EP_DEVICE_ASSERT(__tail_tags_int4[lane_id].x == ZTAG(combine_round_n));
                            // if (lane_id < MAX_PAGES_DIV4) {
                            //     INT_VALUE_NO_NAN(__tail_tags_int4[lane_id].x);
                            //     INT_VALUE_NO_NAN(__tail_tags_int4[lane_id].y);
                            //     INT_VALUE_NO_NAN(__tail_tags_int4[lane_id].z);
                            //     INT_VALUE_NO_NAN(__tail_tags_int4[lane_id].w);
                            // }
                            //st_na_release(target_ptr, __tail_tags_int4[lane_id]);
                            //dst_p2p_ptr == 0 ? st_release_cta(target_ptr, __tail_tags_int4[lane_id]) : st_release_sys_global(target_ptr, __tail_tags_int4[lane_id]);
                            //st_release_cta(target_ptr, __tail_tags_int4[lane_id]);
                            *target_ptr = __tail_tags_int4[lane_id]; // TODO: validate whether there is any memory order problem...
                            // if ((*target_ptr).y != __tail_tags_int4[lane_id].y) {
                            //     printf("[rank %d]: exp %d send back rank %d token %d, put %d stored value and last tag, {0x%08x, 0x%08x, 0x%08x, 0x%08x} offset %lu, but check mismatch\n", rank, global_expert_idx, dst_rank, src_idx, lane_id, __tail_tags_int4[lane_id].x, __tail_tags_int4[lane_id].y, __tail_tags_int4[lane_id].z, __tail_tags_int4[lane_id].w, (target_ptr - cpy_dst_int4_ptr) * sizeof(int4));
                            // }
                        }
                        __syncwarp();
                        num_send_bytes = EXTEND_FOR_TAG_AND_ALIGN(long_msg_len + sizeof(int4), sizeof(int4)); // extra tag store space
                    }
                }

                // Flush all stores
                tma_store_wait();
                __syncwarp();
                if constexpr (kEager_combine != EAGER_OFF) {
                    if (intra_node && lane_id == 0) {
                        // intranode: only write last tag to signal token ready!
                        st_release_sys_global(reinterpret_cast<int*>(cpy_dst_int4_ptr + msg_distance / sizeof(int4) - 1), ZTAG(combine_round_n));
                        //printf("[rank %d]: exp %d send back rank %d token %d, put intra node tag, offset %lu\n", rank, global_expert_idx, dst_rank, src_idx, msg_distance - sizeof(int4));
                    }
                }
            }

            // Issue RDMA
            // NOTES: for zero-copy mode, we assume the data is already in the send buffer
            if (dst_p2p_ptr == 0)
                nvshmemi_ibgda_put_nbi_warp(dst_ptr, buf_ptr, num_send_bytes, dst_rank, local_expert_idx, lane_id, message_idx);
        }

        // Put the finishing flag
        EP_DEVICE_ASSERT(num_warps_per_group > 1 and num_warp_groups < 16);
        asm volatile("bar.sync %0, %1;" :: "r"(warp_group_id + 1), "r"(num_warps_per_group * 32));
        if (sub_warp_id == 1 and lane_id == 0) {
            if constexpr (kEager_combine != EAGER_FULL) {
                while (ld_acquire_global(atomic_clean_flag) == 0);
            }
            auto dst_ptr = reinterpret_cast<uint64_t>(rdma_recv_flag + global_expert_idx);
            auto dst_p2p_ptr = nvshmemi_get_p2p_ptr(dst_ptr, rank, dst_rank);
            const int st_value = kEager_combine == EAGER_FULL ? (1 | SHORT_TAG(combine_round_n)) : 1;
            if (dst_p2p_ptr == 0) {
                if (kEager_combine <= EAGER_OFF) {
                    nvshmemi_ibgda_amo_nonfetch_add(reinterpret_cast<int*>(dst_ptr), st_value, dst_rank, local_expert_idx);
                } else {
                    nvshmemi_ibgda_rma_p(reinterpret_cast<int*>(dst_ptr), st_value, dst_rank, local_expert_idx);
                }
            } else {
                st_release_sys_global(reinterpret_cast<int*>(dst_p2p_ptr), st_value);
                //printf("[rank %d]: combine round 0x%08x expert %d send signal to rank %d\n", rank, combine_round_n, global_expert_idx, dst_rank);
            }
            if constexpr (kEager_combine != EAGER_FULL) {
                atomic_add_release_global(atomic_clean_flag, -1);
            }
        }
        __syncwarp();

        // Destroy m-barriers
        if (lane_id < kNumStages) {
            mbarrier_inval(full_barriers[lane_id]);
            fence_view_async_shared();
            fence_barrier_init();
        }
        __syncwarp();
    }

    // cg::this_grid().sync();
    // if (sm_id == 0 && thread_id == 0) printf("[rank %d]: round 0x%x combine send done\n", rank, combine_round_n);

    // Receiving phase
    LOW_LATENCY_COMBINE_RECV:
    if ((phases & LOW_LATENCY_RECV_PHASE) == 0)
        return;
    
    if constexpr (kEager_combine != EAGER_FULL) {
        // Wait all ranks to arrive
        if (responsible_expert_idx < num_experts) {
            EP_DEVICE_ASSERT(num_warps_per_group > 1);
            if (sub_warp_id == 0 and lane_id == 0) {
                auto start_time = clock64();
                while (ld_acquire_sys_global(rdma_recv_flag + responsible_expert_idx) == 0);
                auto wait_recv_cost = clock64() - start_time;
                if (combine_wait_recv_cost_stats != nullptr) {
                    const auto& src_rank = responsible_expert_idx / num_local_experts;
                    atomicAdd(reinterpret_cast<unsigned long long*>(combine_wait_recv_cost_stats + src_rank), wait_recv_cost);
                }
            }
        }
        cg::this_grid().sync();
    } else {
        // if (responsible_expert_idx < num_experts) {
        //     const auto src_rank = responsible_expert_idx / num_local_experts;
        //     const auto intra_node = (src_rank >> 3) == (rank >> 3);
        //     if (intra_node) {
        //         EP_DEVICE_ASSERT(num_warps_per_group > 1);
        //         if (sub_warp_id == 0 and lane_id == 0) {
        //             const auto target_value = (SHORT_TAG(combine_round_n) | 1);
        //             printf("[rank %d]: combine round 0x%08x waiting signal from rank %d expert %d\n", rank, combine_round_n, src_rank, responsible_expert_idx);
        //             while (ld_acquire_sys_global(rdma_recv_flag + responsible_expert_idx) == target_value);
        //             printf("[rank %d]: combine round 0x%08x got signal from rank %d expert %d\n", rank, combine_round_n, src_rank, responsible_expert_idx);
        //         }
        //     }
        // }
        __syncthreads(); // two types of recv warp must be synchronized
    }
    

    // Reassign warp groups
    constexpr int kMaxNumGroups = 2;
    const int num_decode_warps = hidden_bf16_int4_pad / (kNumRecvUnrolls * 32);
    const int num_groups = min(kMaxNumGroups, (num_threads / 32) / (num_decode_warps + 1));
    const int decode_warp_idx = __shfl_sync(0xffffffff, warp_id % (num_decode_warps + 1), 0);
    const int group_idx = __shfl_sync(0xffffffff, warp_id / (num_decode_warps + 1), 0);
    EP_STATIC_ASSERT(kHidden % (32 * kNumElemsPerInt4) == 0, "Invalid vectorization");
    EP_DEVICE_ASSERT(num_topk <= 32);
    EP_DEVICE_ASSERT(num_groups > 0);

    if (group_idx < num_groups) {
        constexpr int kNumStages = 3;
        constexpr int kNumTMABufferBytes = 16 * 2 + ((kEager_combine == EAGER_OFF) ? (kHidden * sizeof(hip_bfloat16)) : (long_msg_ext_len + sizeof(int4))); // WARNING: use metabytes for logfmt, assume redundant region is enough!
        // if (sm_id == 0 && thread_id == 0) {
        //     printf("[rank %d]: TMABufferBytes = %d\n", rank, kNumTMABufferBytes);
        // }
        constexpr int kNumBF16PerWarpBytes = 32 * kNumRecvUnrolls * kNumElemsPerInt4 * 2;
        constexpr int kNumLogFMTPerWarpBytes = kNumBF16PerWarpBytes / 16 * 10;
        constexpr int kNumDivisionBytes = kNumDivisions * sizeof(uint32_t);
        constexpr int kNumBytesPerGroup = kNumStages * kNumTMABufferBytes + kHidden * 2 + kNumStages * kNumDivisionBytes * 3;

        //__shared__ int4 ld_tags_int4[kMaxNumGroups][kNumStages][MAX_PAGES_DIV4];

        // Reallocate shared memory
        const auto smem_group_buffer = smem_buffer + kNumBytesPerGroup * group_idx;
        auto full_barriers  = PatternVisitor([=](const int& i) { return reinterpret_cast<uint64_t*>(smem_group_buffer + i * kNumTMABufferBytes); });
        auto empty_barriers = PatternVisitor([=](const int& i) { return reinterpret_cast<uint64_t*>(smem_group_buffer + i * kNumTMABufferBytes + 8); });
        auto tma_ld_buffers = PatternVisitor([=](const int& i) { return reinterpret_cast<uint8_t* >(smem_group_buffer + i * kNumTMABufferBytes + 16); });
        auto tma_st_buffers = PatternVisitor([=](const int& i) { return reinterpret_cast<uint32_t*>(smem_group_buffer + kNumStages * kNumTMABufferBytes + i * kNumBF16PerWarpBytes); });

        // Redundant when logfmt is disabled
        const auto smem_group_ptr = smem_group_buffer + kNumStages * kNumTMABufferBytes + kHidden * 2;
        auto log_amax_buffers  = PatternVisitor([=](const int& i) { return reinterpret_cast<float*>(smem_group_ptr + i * kNumDivisionBytes); });
        auto log_amin_buffers  = PatternVisitor([=](const int& i) { return reinterpret_cast<float*>(smem_group_ptr + kNumStages * kNumDivisionBytes + i * kNumDivisionBytes); });
        auto cast_info_buffers = PatternVisitor([=](const int& i) { return reinterpret_cast<int*>  (smem_group_ptr + kNumStages * kNumDivisionBytes * 2 + i * kNumDivisionBytes); });

        uint32_t tma_phase = 0;
        EP_STATIC_ASSERT(kNumStages < 32, "Too many stages");
        if (decode_warp_idx == num_decode_warps)
            tma_phase = (1 << kNumStages) - 1;

        // Initialize m-barriers
        if (decode_warp_idx == num_decode_warps and lane_id < kNumStages) {
            mbarrier_init(full_barriers[lane_id], 1);
            mbarrier_init(empty_barriers[lane_id], num_decode_warps);
        }
        asm volatile("bar.sync %0, %1;" :: "r"(group_idx + 1), "r"((num_decode_warps + 1) * 32));

        int stage_idx = 0, topk_idx_by_lane = 0;
        EP_STATIC_ASSERT(kNumMaxTopk <= 32, "Invalid number of topks");
        if (decode_warp_idx == num_decode_warps) {
            // TMA load warp
            //if (sm_id == 0 && rank == 0 && group_idx == 0 && lane_id == 0) printf("tma load warp working\n");
            for (int token_idx = sm_id + num_sms * group_idx; token_idx < num_combined_tokens; token_idx += num_sms * num_groups) {
                if (lane_id < num_topk)
                    topk_idx_by_lane = static_cast<int>(__ldg(topk_idx + token_idx * num_topk + lane_id));
                for (int i = 0; i < num_topk; ++ i) {
                    int topk_idx_reg = __shfl_sync(0xffffffff, topk_idx_by_lane, i);
                    if (topk_idx_reg < 0)
                        continue;
                    const int src_rank = topk_idx_reg / num_local_experts;
                    const auto intra_node = (src_rank >> 3) == (rank >> 3);
                    //if (!intra_node) printf("[rank %d]: an inter-node combine recv\n", rank);
                    //if (sm_id == 0 && rank == 0 && group_idx == 0 && lane_id == 0) printf("tma load warp waiting on mbarrier\n");
                    mbarrier_wait<true>(empty_barriers[stage_idx], tma_phase, stage_idx);
                    auto buffer = static_cast<uint8_t*>(rdma_recv_x) + (topk_idx_reg * num_max_dispatch_tokens_per_rank + token_idx) * msg_distance;
                    //if (sm_id == 0 && rank == 0 && group_idx == 0 && lane_id == 0) printf("tma load warp pass mbarrier\n");
                    if constexpr (kEager_combine != EAGER_OFF) {
                        constexpr int pages = PAGE_N(long_msg_len);
                        if (intra_node) {
                            // intra node: just check tail tag
                            if (lane_id == 0) {
                                int *__check_ptr = reinterpret_cast<int*>(buffer + msg_distance - sizeof(int4));
                                //int _value = ld_acquire_sys_global(__check_ptr);
                                //int w_cnt = 0;
                                //printf("[rank %d]: combine round 0x%08x token %d topk %d from expert %d check intra node tag at offset %lu, init 0x%08x\n", rank, combine_round_n, token_idx, i, topk_idx_reg, msg_distance - sizeof(int4), _value);
                                // while (_value != ZTAG(combine_round_n)) {
                                //     _value = ld_acquire_sys_global(__check_ptr);
                                //     w_cnt += 1;
                                //     if ((w_cnt & CHECK_TIME_MASK) == 0) printf("[rank %d]: combine round 0x%08x token %d topk %d from exp %d at rank %d, check %d times, 0x%08x != 0x%08x\n", rank, combine_round_n, token_idx, i, topk_idx_reg, src_rank, w_cnt, _value, ZTAG(combine_round_n));
                                // }
                                while (ld_acquire_sys_global(__check_ptr) != ZTAG(combine_round_n));
                                //printf("[rank %d]: combine round 0x%08x token %d from expert %d check intra node tag done\n", rank, combine_round_n, token_idx, topk_idx_reg);
                                // char tmatch[2][30] = {"", " WRONG COMBINE TAG!"};
                                // for (int __pn = 0; __pn < pages; __pn += 1) {
                                //     int *__check_ptr = reinterpret_cast<int*>(buffer + ((__pn == pages - 1) ? (long_msg_ext_len) : ((__pn << PCIE_SEG_LEN_LOG) + PCIE_SEG_LEN - PCIE_TAIL_SZ)));
                                //     int __check_val = ld_nc_global(__check_ptr);
                                //     printf("[rank %d]: combine round 0x%08x token %d from expert %d inter node tag 0x%08x at offset %lu %s\n", rank, combine_round_n, token_idx, topk_idx_reg, __check_val, PTR_DIFF(__check_ptr, buffer), tmatch[__check_val != ZTAG(combine_round_n)]);
                                // }
                                // printf("[rank %d]: combine round 0x%08x token %d from expert %d last jump tag 0x%08x at offset %lu\n", rank, combine_round_n, token_idx, topk_idx_reg, ld_nc_global(reinterpret_cast<int*>(buffer + long_msg_ext_len + sizeof(int))), long_msg_ext_len + sizeof(int));
                            }
                        } else {
                            // inter node: check tags of all pages
                            #pragma unroll
                            for (int __pn = lane_id; __pn < pages; __pn += 32) {
                                int *__check_ptr = reinterpret_cast<int*>(buffer + ((__pn == pages - 1) ? (long_msg_ext_len) : ((__pn << PCIE_SEG_LEN_LOG) + PCIE_SEG_LEN - PCIE_TAIL_SZ)));
                                //EP_DEVICE_ASSERT(reinterpret_cast<uint8_t*>(__check_ptr) - buffer + sizeof(int) <= msg_distance);
                                //int _value = ld_acquire_sys_global(__check_ptr);
                                //printf("[rank %d]: combine round 0x%08x token %d topk %d from exp %d at rank %d, check offset %lu, init 0x%08x\n", rank, combine_round_n, token_idx, i, topk_idx_reg, src_rank, reinterpret_cast<uint8_t*>(__check_ptr) - buffer, _value);
                                // int w_cnt = 0;
                                // while (_value != ZTAG(combine_round_n)) {
                                //     _value = ld_acquire_sys_global(__check_ptr);
                                //     w_cnt += 1;
                                //     if ((w_cnt & CHECK_TIME_MASK) == 0) printf("[rank %d]: combine round 0x%08x token %d topk %d from exp %d at rank %d, check offset %lu, %d times, 0x%08x != 0x%08x\n", rank, combine_round_n, token_idx, i, topk_idx_reg, src_rank, PTR_DIFF(__check_ptr, buffer), w_cnt, _value, ZTAG(combine_round_n));
                                // }
                                while (ld_acquire_sys_global(__check_ptr) != ZTAG(combine_round_n));
                                //printf("[rank %d]: combine round 0x%08x token %d topk %d from exp %d at rank %d, check offset %lu done\n", rank, combine_round_n, token_idx, i, topk_idx_reg, src_rank, reinterpret_cast<uint8_t*>(__check_ptr) - buffer);
                            }
                        }
                        __syncwarp();
                        // if (!intra_node) {
                        //     if (lane_id < MAX_PAGES_DIV4) {
                        //         auto tag_pos = buffer + sizeof(hip_bfloat16) * kHidden + lane_id * sizeof(int4);
                        //         //EP_DEVICE_ASSERT(tag_pos - buffer + sizeof(int4) <= msg_distance);
                        //         st_release_cta(&ld_tags_int4[group_idx][stage_idx][lane_id], ld_nc_global(reinterpret_cast<int4*>(tag_pos)));
                        //     }
                        // }
                        // __syncwarp();
                    }
                    if constexpr (kUseLogFMT) {
                        logfmt_check_amaxmin<kNumDivisions / 2, kNumSendUnrolls, kNumRecvUnrolls>(
                            buffer, reinterpret_cast<float2*>(log_amax_buffers[stage_idx]),
                            reinterpret_cast<float2*>(log_amin_buffers[stage_idx]), cast_info_buffers[stage_idx], lane_id);
                    }
                    //if (sm_id == 0 && rank == 0 && group_idx == 0 && lane_id == 0) printf("tma load warp launching tma load\n");
                    if (elect_one_sync(lane_id)) {
                        int num_casted = 0;
                        if constexpr (kUseLogFMT) {
                            const auto& info = cast_info_buffers[stage_idx][num_decode_warps - 1];
                            num_casted = (info >> 1) + (info & 1);
                        }
                        int num_tma_bytes;
                        if constexpr (kEager_combine != EAGER_OFF) {
                            num_tma_bytes = long_msg_ext_len + sizeof(int4);
                        } else {
                            num_tma_bytes = num_casted * kNumLogFMTPerWarpBytes + (num_decode_warps - num_casted) * kNumBF16PerWarpBytes;
                        }
                        tma_load_1d(tma_ld_buffers[stage_idx], buffer + (kUseLogFMT ? kNumMetaBytes : 0), full_barriers[stage_idx], num_tma_bytes);
                        mbarrier_arrive_and_expect_tx(full_barriers[stage_idx], num_tma_bytes);
                    }
                    //if (sm_id == 0 && rank == 0 && group_idx == 0 && lane_id == 0) printf("tma load warp started tma load\n");
                    // if constexpr (kEager != EAGER_OFF) {
                    //     // inter node: tag postion shall be restored, load them first
                    //     if (!intra_node && lane_id < MAX_PAGES_DIV4) {
                    //         auto tag_pos = buffer + sizeof(hip_bfloat16) * kHidden + lane_id * sizeof(int4);
                    //         EP_DEVICE_ASSERT(tag_pos - buffer + sizeof(int4) <= msg_distance);
                    //         ld_tags_int4[group_idx][stage_idx][lane_id] = ld_nc_global(reinterpret_cast<int4*>(tag_pos));
                    //     }
                    // }
                    __syncwarp();
                    stage_idx = (stage_idx + 1) % kNumStages;
                    //if (sm_id == 0 && rank == 0 && group_idx == 0 && lane_id == 0) printf("tma load warp loop end\n");
                }
            }
        } else {
            // Reduction warps
            //if (sm_id == 0 && rank == 0 && group_idx == 0 && lane_id == 0 && decode_warp_idx == 0) printf("reduce warp working\n");
            float topk_weights_by_lane;
            for (int token_idx = sm_id + num_sms * group_idx; token_idx < num_combined_tokens; token_idx += num_sms * num_groups) {
                if (lane_id < num_topk) {
                    topk_idx_by_lane = static_cast<int>(__ldg(topk_idx + token_idx * num_topk + lane_id));
                    topk_weights_by_lane = __ldg(topk_weights + token_idx * num_topk + lane_id);
                }
                __syncwarp();
                //if (sm_id == 0 && rank == 0 && group_idx == 0 && lane_id == 0 && decode_warp_idx == 0) printf("reduce warp got topk info\n");
                float combined_values[kNumElemsPerInt4 * kNumRecvUnrolls] = {0.0f};
                for (int i = 0; i < num_topk; ++ i) {
                    const auto topk_idx = __shfl_sync(0xffffffff, topk_idx_by_lane, i);
                    if (topk_idx < 0)
                        continue;
                    const auto& topk_weight = __shfl_sync(0xffffffff, topk_weights_by_lane, i);
                    //const auto intra_node = ((topk_idx / num_local_experts) >> 3) == (rank >> 3);
                    //if (sm_id == 0 && rank == 0 && group_idx == 0 && lane_id == 0 && decode_warp_idx == 0) printf("reduce warp wait on barrier\n");
                    mbarrier_wait<true>(full_barriers[stage_idx], tma_phase, stage_idx);
                    //if (sm_id == 0 && rank == 0 && group_idx == 0 && lane_id == 0 && decode_warp_idx == 0) printf("reduce warp pass barrier\n");
                    if constexpr (kEager_combine != EAGER_OFF) {
                        //auto buffer = static_cast<uint8_t*>(rdma_recv_x) + (topk_idx * num_max_dispatch_tokens_per_rank + token_idx) * msg_distance;
                        constexpr int pages = PAGE_N(long_msg_len);
                        if (true) {
                            if (decode_warp_idx == 0 && lane_id < pages) {
                                //EP_DEVICE_ASSERT((lane_id << PCIE_SEG_LEN_LOG) + (PCIE_SEG_LEN - PCIE_TAIL_SZ) < kHidden * sizeof(hip_bfloat16) - sizeof(int));
                                // int space;
                                // hip_bfloat16* space_ptr = reinterpret_cast<hip_bfloat16*>(&space);
                                // space_ptr[0] = static_cast<hip_bfloat16>(rank - 128);
                                // space_ptr[1] = space_ptr[0];
                                //EP_DEVICE_ASSERT(reinterpret_cast<int*>(ld_tags_int4[group_idx][stage_idx])[lane_id] == space);
                                //printf("[rank %d]: debug: wriring shmem\n", rank);
                                const auto ld_offset = (lane_id < pages - 1) ? (kHidden * sizeof(hip_bfloat16) + sizeof(int) * lane_id) : (long_msg_ext_len + sizeof(int));
                                const auto st_offset = (lane_id < pages - 1) ? ((lane_id << PCIE_SEG_LEN_LOG) + (PCIE_SEG_LEN - PCIE_TAIL_SZ)) : short_msg_ext_len;
                                int save_value = *(reinterpret_cast<int*>(tma_ld_buffers[stage_idx] + ld_offset)); // reinterpret_cast<int*>(ld_tags_int4[group_idx][stage_idx])[lane_id]
                                //printf("[rank %d]: combine round 0x%08x token %d from expert %d restore value 0x%08x from %lu to %lu\n", rank, combine_round_n, token_idx, topk_idx, save_value, ld_offset, st_offset);
                                //INT_VALUE_NO_NAN(save_value);
                                st_release_cta(reinterpret_cast<int*>(tma_ld_buffers[stage_idx] + st_offset), save_value);
                            }
                            asm volatile("bar.sync %0, %1;" :: "r"(group_idx + 2), "r"(num_decode_warps * 32));
                        }
                    }
                    //if (sm_id == 0 && rank == 0 && group_idx == 0 && lane_id == 0 && decode_warp_idx == 0) printf("reduce warp begin reduce\n");
                    if constexpr (kUseLogFMT) {
                        const auto& info = cast_info_buffers[stage_idx][decode_warp_idx];
                        bool enable_cast = info & 1;
                        int num_casted_prefix = info >> 1;
                        int tma_offset = kNumLogFMTPerWarpBytes * num_casted_prefix + kNumBF16PerWarpBytes * (decode_warp_idx - num_casted_prefix);
                        int division_idx = decode_warp_idx * (kNumRecvUnrolls * 2) + lane_id * kNumRecvUnrolls / 16;
                        decode_and_accumulate<kNumRecvUnrolls>(
                            reinterpret_cast<uint32_t*>(tma_ld_buffers[stage_idx] + tma_offset + (enable_cast ? kNumLogFMTPerWarpBytes : kNumBF16PerWarpBytes) / 32 * lane_id),
                            combined_values, log_amax_buffers[stage_idx][division_idx], log_amin_buffers[stage_idx][division_idx], enable_cast, topk_weight
                        );
                    } else {
                        int tma_offset = kNumBF16PerWarpBytes * decode_warp_idx;
                        decode_and_accumulate<kNumRecvUnrolls>(
                            reinterpret_cast<uint32_t*>(tma_ld_buffers[stage_idx] + tma_offset + kNumBF16PerWarpBytes / 32 * lane_id),
                            combined_values, 0, 0, false, topk_weight
                        );
                    }
                    //if (sm_id == 0 && rank == 0 && group_idx == 0 && lane_id == 0 && decode_warp_idx == 0) printf("reduce warp finish reduce\n");
                    if (elect_one_sync(lane_id))
                        mbarrier_arrive(empty_barriers[stage_idx]);
                    stage_idx = (stage_idx + 1) % kNumStages;
                }
                //asm volatile("bar.sync %0, %1;" :: "r"(group_idx + 2), "r"(num_decode_warps * 32));
                //if (sm_id == 0 && rank == 0 && group_idx == 0 && lane_id == 0 && decode_warp_idx == 0) printf("reduce warp finish all reduce, waiting for tma store done\n");

                tma_store_wait<0>();

                //if (sm_id == 0 && rank == 0 && group_idx == 0 && lane_id == 0 && decode_warp_idx == 0) printf("reduce warp finish all reduce, load reduce result to share mem\n");
                #pragma unroll
                for (int k = 0; k < kNumRecvUnrolls * 4; ++ k) {
                    auto combined_pack = __hip_bfloat162(combined_values[k * 2], combined_values[k * 2 + 1]);
                    tma_st_buffers[decode_warp_idx][kNumRecvUnrolls * 4 * lane_id + k] = *reinterpret_cast<uint32_t*>(&combined_pack);
                }
                tma_store_fence();
                //if (sm_id == 0 && rank == 0 && group_idx == 0 && lane_id == 0 && decode_warp_idx == 0) printf("reduce warp finish all reduce, start tma store\n");
                if (elect_one_sync(lane_id)) {
                    tma_store_1d(tma_st_buffers[decode_warp_idx],
                                 static_cast<int4*>(combined_x) + token_idx * hidden_bf16_int4 + decode_warp_idx * kNumRecvUnrolls * 32,
                                 kNumBF16PerWarpBytes);
                }
                __syncwarp();
                //if (sm_id == 0 && rank == 0 && group_idx == 0 && lane_id == 0 && decode_warp_idx == 0) printf("reduce warp finish a token\n");
            }
        }

        // Flush all stores
        tma_store_wait<0>();
    }

    // cg::this_grid().sync();
    // if (sm_id == 0 && thread_id == 0) printf("[rank %d]: round 0x%x combine waiting signal\n", rank, combine_round_n);
    
    // This is done for making sure we do not exit this round of combine until all peers are combining at least
    // Without this, it may happen that we finish combine and begin next dispatch, but a peer is slow, still dispatching in last round.
    // Then two round of dispatch data may overwrite! 
    if (responsible_expert_idx < num_experts) {
        //const auto src_rank = responsible_expert_idx / num_local_experts;
        //const auto intra_node = (src_rank >> 3) == (rank >> 3);
        if constexpr (kEager_combine == EAGER_FULL) {
            EP_DEVICE_ASSERT(num_warps_per_group > 1);
            if (sub_warp_id == 0 and lane_id == 0) {
                const auto target_value = (SHORT_TAG(combine_round_n) | 1);
                int _value;
                while ((_value = ld_acquire_sys_global(rdma_recv_flag + responsible_expert_idx)) != target_value) {
                    // w_cnt += 1;
                    // if ((w_cnt & CHECK_TIME_MASK) == 0) {
                    //     printf("[rank %d]: combine round 0x%08x waiting signal from expert %d for %d times, 0x%08x != 0x%08x\n", rank, combine_round_n, responsible_expert_idx, w_cnt, _value, target_value);
                    // }
                }
            }
        }
    }
    // cg::this_grid().sync();
    // if (sm_id == 0 && thread_id == 0) printf("[rank %d]: round 0x%x combine recv done\n", rank, combine_round_n);
}

void combine(void* combined_x,
             void* rdma_recv_x, int* rdma_recv_flag, void* rdma_send_x,
             const void* x, const int64_t* topk_idx, const float* topk_weights,
             const int* per_rank_src_count,
             const int* src_info, const int64_t* layout_range,
             int64_t* combine_wait_recv_cost_stats,
             int* next_clean, int num_next_clean_int,
             int num_combined_tokens, int hidden, int num_max_dispatch_tokens_per_rank,
             int num_topk, int num_experts, int rank, int num_ranks,
             bool use_logfmt,
             void* workspace, int num_device_sms,
             hipStream_t stream, int phases, bool zero_copy,
             int combine_round_n, const int eager_opt) {
    constexpr int kNumMaxTopk = 9;
    const int num_warp_groups = ceil_div(num_experts, num_device_sms);
    const int num_warps_per_group = 32 / num_warp_groups;
    const int num_recv_per_sm = ceil_div(num_combined_tokens, num_device_sms);
    EP_HOST_ASSERT(num_warp_groups > 0 and num_warps_per_group > 0 and num_recv_per_sm > 0);

    const auto num_warps = num_warp_groups * num_warps_per_group;
    const auto num_sms = max(ceil_div(num_experts, num_warp_groups), ceil_div(num_combined_tokens, num_recv_per_sm));

    // Check workspace
    auto atomic_clean_flag = static_cast<int*>(workspace);
    EP_HOST_ASSERT(sizeof(int) <= NUM_WORKSPACE_BYTES);
    EP_HOST_ASSERT(num_topk <= kNumMaxTopk);

    // Online cast cannot use zero-copy
    EP_HOST_ASSERT(not (zero_copy and use_logfmt));

    constexpr int kNumStages = 3;
    constexpr int kNumMaxUnrolls = 4;
    constexpr int kMaxNumGroups = 2;

    // Send buffer size
    const int num_meta_bytes = hidden / 128 * 4;
    const int num_send_tma_bytes = 32 * sizeof(int4) * kNumMaxUnrolls + 16;
    const int smem_send_size = num_warps * (kNumStages * num_send_tma_bytes + num_meta_bytes);

    // Receive buffer size
    const int num_recv_tma_bytes = 16 + hidden * 2;
    const int smem_recv_size = kMaxNumGroups * (kNumStages * num_recv_tma_bytes + hidden * 2 + kNumStages * num_meta_bytes * 3);

    // Total requirement
    const int smem_size = max(smem_send_size, smem_recv_size);

#define COMBINE_LAUNCH_CASE(eager_opt, hidden) { \
auto combine_func = use_logfmt ? \
    combine<true, hidden, kNumMaxTopk, kNumMaxUnrolls, EAGER_OFF> : \
    combine<false, hidden, kNumMaxTopk, kNumMaxUnrolls, eager_opt>; \
SET_SHARED_MEMORY_FOR_TMA(combine_func); \
LAUNCH_KERNEL(&cfg, combine_func, \
              combined_x, \
              rdma_recv_x, rdma_recv_flag, rdma_send_x, \
              x, topk_idx, topk_weights, per_rank_src_count, src_info, layout_range, \
              combine_wait_recv_cost_stats, \
              next_clean, num_next_clean_int, \
              atomic_clean_flag, \
              num_combined_tokens, hidden, num_topk, \
              num_max_dispatch_tokens_per_rank, \
              num_experts, rank, num_ranks, \
              num_warp_groups, num_warps_per_group, \
              phases, zero_copy, combine_round_n); } break

    SETUP_LAUNCH_CONFIG(num_sms, num_warps * 32, stream);

    #define EAGER_COMBINE_LAUNCH_CASE(hidden) SWITCH_EAGER(COMBINE_LAUNCH_CASE, hidden)

    SWITCH_HIDDEN(EAGER_COMBINE_LAUNCH_CASE);
#undef COMBINE_LAUNCH_CASE
}

} // namespace internode_ll

} // namespace deep_ep
