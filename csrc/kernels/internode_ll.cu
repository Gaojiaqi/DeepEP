#include "hip/hip_runtime.h"
#include "configs.cuh"
#include "exception.cuh"
#include "launch.cuh"
#include "ibgda_device.cuh"

namespace deep_ep {

namespace internode_ll {

template <int kNumThreads> __launch_bounds__(kNumThreads, 1)
__global__ void clean_low_latency_buffer(int* clean_0, int num_clean_int_0,
                                         int* clean_1, int num_clean_int_1) {
    // Barrier before cleaning (in case of unfinished chunked EP)
    nvshmemx_barrier_all_block();

    // Clean
    auto thread_id = static_cast<int>(threadIdx.x);
    #pragma unroll
    for (int i = thread_id; i < num_clean_int_0; i += kNumThreads)
        clean_0[i] = 0;
    #pragma unroll
    for (int i = thread_id; i < num_clean_int_1; i += kNumThreads)
        clean_1[i] = 0;

    // Barrier after cleaning (make sure the low-latency mode works fine)
    nvshmemx_barrier_all_block();
}

void clean_low_latency_buffer(int* clean_0, int num_clean_int_0,
                              int* clean_1, int num_clean_int_1,
                              hipStream_t stream) {
    constexpr int kNumThreads = 256;

    SETUP_LAUNCH_CONFIG(1, kNumThreads, stream);
    LAUNCH_KERNEL(&cfg, clean_low_latency_buffer<kNumThreads>,
                  clean_0, num_clean_int_0, clean_1, num_clean_int_1);
}

template <bool kUseFP8, bool kUseUE8M0, int kHidden>
__global__ __launch_bounds__(1024, 1) void
dispatch(void* packed_recv_x, void* packed_recv_x_scales,
         int* packed_recv_src_info, int64_t* packed_recv_layout_range,
         int* packed_recv_count,
         int* cumulative_local_expert_recv_stats,
         void* rdma_recv_x, int* rdma_recv_count, void* rdma_x,
         const void* x, const int64_t* topk_idx,
         int* atomic_counter_per_expert, int* atomic_finish_counter_per_expert,
         int* next_clean, int num_next_clean_int,
         int num_tokens, int num_max_dispatch_tokens_per_rank,
         int num_topk, int num_experts, int rank, int num_ranks,
         int num_warp_groups, int num_warps_per_group,
         bool round_scale, int phases) {
    const auto sm_id = static_cast<int>(blockIdx.x);
    const auto thread_id = static_cast<int>(threadIdx.x);
    const auto warp_id = thread_id / 32, lane_id = get_lane_id();
    const auto num_sms = static_cast<int>(gridDim.x);
    const auto num_warps = num_warp_groups * num_warps_per_group;
    const auto num_local_experts = num_experts / num_ranks;
    const auto warp_group_id = warp_id / num_warps_per_group;
    const auto sub_warp_id = warp_id % num_warps_per_group;
    const auto responsible_expert_idx = sm_id * num_warp_groups + warp_group_id;

    // May extract UE8M0 from the scales
    using scale_t = std::conditional_t<kUseUE8M0, uint8_t, float>;
    using packed_t = std::conditional_t<kUseUE8M0, uint32_t, float>;
    EP_STATIC_ASSERT(sizeof(packed_t) % sizeof(scale_t) == 0, "Invalid vector length");

    // FP8 staffs
    constexpr int kNumPerChannels = 128;
    const int num_scales = kHidden / kNumPerChannels;
    const size_t hidden_bytes = kHidden * (kUseFP8 ? sizeof(__hip_fp8_storage_t) : sizeof(hip_bfloat16));
    const size_t hidden_int4 = hidden_bytes / sizeof(int4);

    // Message package: hidden data, FP8 scales, index at source
    // NOTES: currently we have 3 reserved int fields for future use
    using vec_t = typename std::conditional<kUseFP8, int2, int4>::type;
    const size_t num_bytes_per_msg = sizeof(int4) + (kUseFP8 ? (kHidden + num_scales * sizeof(float)) : (kHidden * sizeof(hip_bfloat16)));
    const size_t num_int4_per_msg = num_bytes_per_msg / sizeof(int4);
    EP_DEVICE_ASSERT(num_bytes_per_msg % sizeof(int4) == 0);

    // Expert counts
    constexpr int kNumMaxWarpGroups = 32;
    __shared__ int shared_num_tokens_sent_per_expert[kNumMaxWarpGroups];

    // Sending phase
    if ((phases & LOW_LATENCY_SEND_PHASE) == 0)
        goto LOW_LATENCY_DISPATCH_RECV;

    // There are 2 kinds of warps in this part:
    // 1. The first-kind warps for FP8 cast and sending top-k tokens
    // 2. The last warp for reading `topk_idx` and count for per-expert information
    if (warp_id < num_warps - 1) {
        constexpr int kNumElemsPerRead = sizeof(int4) / sizeof(hip_bfloat16);
        EP_STATIC_ASSERT(kHidden % (32 * kNumElemsPerRead) == 0, "Invalid hidden");
        EP_STATIC_ASSERT(kNumElemsPerRead * 32 % kNumPerChannels == 0, "Invalid vectorization");
        const auto num_threads = (num_warps - 1) * 32;
        const size_t hidden_bf16_int4 = kHidden / kNumElemsPerRead;

        for (int token_idx = sm_id; token_idx < num_tokens; token_idx += num_sms) {
            const auto x_int4 = static_cast<const int4*>(x) + token_idx * hidden_bf16_int4;
            const auto rdma_x_src_idx = reinterpret_cast<int*>(static_cast<uint8_t*>(rdma_x) + token_idx * num_bytes_per_msg);
            const auto rdma_x_vec = reinterpret_cast<vec_t*>(reinterpret_cast<uint8_t*>(rdma_x_src_idx) + sizeof(int4));
            const auto rdma_x_scales = reinterpret_cast<float*>(reinterpret_cast<uint8_t*>(rdma_x_vec) + hidden_bytes);

            // Overlap top-k index read and source token index writes
            auto dst_expert_idx = warp_id < num_topk ? static_cast<int>(__ldg(topk_idx + token_idx * num_topk + warp_id)) : -1;
            thread_id == 0 ? (*rdma_x_src_idx = token_idx) : 0;

            // FP8 cast
            #pragma unroll
            for (int i = thread_id; i < hidden_bf16_int4; i += num_threads) {
                // Read
                auto int4_value = __ldg(x_int4 + i);

                if constexpr (kUseFP8) {
                    // Calculate local amax
                    auto bf16_values = reinterpret_cast<hip_bfloat16*>(&int4_value);
                    float fp32_values[kNumElemsPerRead];
                    float amax = kFP8Margin, scale, scale_inv;
                    #pragma unroll
                    for (int j = 0; j < kNumElemsPerRead; ++ j) {
                        fp32_values[j] = static_cast<float>(bf16_values[j]);
                        amax = fmaxf(amax, fabsf(fp32_values[j]));
                    }

                    // Reduce amax and scale
                    EP_STATIC_ASSERT(kNumElemsPerRead * 32 / kNumPerChannels == 2, "Invalid vectorization");
                    amax = warp_reduce_max<16>(amax);
                    calculate_fp8_scales(amax, scale, scale_inv, round_scale);
                    if (lane_id == 0 or lane_id == 16)
                        rdma_x_scales[i * kNumElemsPerRead / 128] = scale_inv;

                    // Cast into send buffer
                    vec_t int2_value;
                    auto fp8x2_values = reinterpret_cast<__hip_fp8x2_storage_t*>(&int2_value);
                    #pragma unroll
                    for (int j = 0; j < kNumElemsPerRead; j += 2) {
                        float2 fp32x2 = {fp32_values[j] * scale, fp32_values[j + 1] * scale};
                        fp8x2_values[j / 2] = __hip_cvt_float2_to_fp8x2(fp32x2, __HIP_SATFINITE, __HIP_E4M3_FNUZ);
                    }
                    rdma_x_vec[i] = int2_value;
                } else {
                    // Reinterpret-cast is for C++14 compatibility
                    rdma_x_vec[i] = *reinterpret_cast<vec_t*>(&int4_value);
                }
            }
            asm volatile("bar.sync 1, %0;" :: "r"(num_threads));

            // Issue IBGDA sends
            if (dst_expert_idx >= 0) {
                int slot_idx = lane_id == 0 ? atomicAdd(atomic_counter_per_expert + dst_expert_idx, 1) : 0;
                slot_idx = __shfl_sync(0xffffffff, slot_idx, 0);
                const auto dst_rank = dst_expert_idx / num_local_experts;
                const auto dst_expert_local_idx = dst_expert_idx % num_local_experts;
                const auto src_ptr = reinterpret_cast<uint64_t>(rdma_x_src_idx);
                const auto dst_ptr = reinterpret_cast<uint64_t>(rdma_recv_x) +
                                     dst_expert_local_idx * num_ranks * num_max_dispatch_tokens_per_rank * num_bytes_per_msg +
                                     rank * num_max_dispatch_tokens_per_rank * num_bytes_per_msg +
                                     slot_idx * num_bytes_per_msg;
                const auto dst_p2p_ptr = nvshmemi_get_p2p_ptr(dst_ptr, rank, dst_rank);
                if (dst_p2p_ptr == 0) {
                    nvshmemi_ibgda_put_nbi_warp(dst_ptr, src_ptr, num_bytes_per_msg, dst_rank, dst_expert_local_idx, lane_id, slot_idx);
                } else {
                    // NOTES: only 2 load iterations for 7K hidden with 8 unrolls
                    const auto* src_int4_ptr = reinterpret_cast<const int4*>(src_ptr);
                    const auto* dst_int4_ptr = reinterpret_cast<int4*>(dst_p2p_ptr);
                    UNROLLED_WARP_COPY(8, lane_id, num_int4_per_msg, dst_int4_ptr, src_int4_ptr, ld_nc_global, st_na_global);
                }

                // Increase counter after finishing
                __syncwarp();
                lane_id == 0 ? atomic_add_release_global(atomic_finish_counter_per_expert + dst_expert_idx, 1) : 0;
            }
        }
    } else if (warp_id == num_warps - 1) {
        EP_DEVICE_ASSERT(num_sms > 1);
        if (sm_id == 0) {
            // The first SM is also responsible for checking QPs
            EP_DEVICE_ASSERT(ibgda_get_state()->num_rc_per_pe >= num_local_experts);

            // The first SM is also responsible for cleaning the next buffer
            #pragma unroll
            for (int i = lane_id; i < num_next_clean_int; i += 32)
                next_clean[i] = 0;

            // Notify before executing `int_p`
            __syncwarp();
            #pragma unroll
            for (int i = lane_id; i < num_experts; i += 32)
                atomic_add_release_global(atomic_finish_counter_per_expert + i, FINISHED_SUM_TAG);
        }

        // This SM should be responsible for some destination experts, read `topk_idx` for them
        int expert_count[kNumMaxWarpGroups] = {0};
        const auto expert_begin_idx = sm_id * num_warp_groups;
        const auto expert_end_idx = min(expert_begin_idx + num_warp_groups, num_experts);

        // Per lane count
        #pragma unroll 8
        for (int i = lane_id; i < num_tokens * num_topk; i += 32) {
            auto idx = static_cast<int>(__ldg(topk_idx + i));
            if (idx >= expert_begin_idx and idx < expert_end_idx)
                expert_count[idx - expert_begin_idx] ++;
        }

        // Warp reduce
        #pragma unroll
        for (int i = expert_begin_idx; i < expert_end_idx; ++ i) {
            auto sum = warp_reduce_sum(expert_count[i - expert_begin_idx]);
            if (lane_id == 0) {
                shared_num_tokens_sent_per_expert[i - expert_begin_idx] = sum;
                atomic_add_release_global(atomic_finish_counter_per_expert + i, FINISHED_SUM_TAG - sum);
            }
        }
    }
    __syncthreads();

    // Issue count sends
    if (responsible_expert_idx < num_experts and sub_warp_id == 0 and lane_id == 0) {
        const auto dst_rank = responsible_expert_idx / num_local_experts;
        const auto dst_expert_local_idx = responsible_expert_idx % num_local_experts;
        const auto num_tokens_sent = shared_num_tokens_sent_per_expert[responsible_expert_idx - sm_id * num_warp_groups];

        // Wait local sends issued and send expert counts
        while (ld_acquire_global(atomic_finish_counter_per_expert + responsible_expert_idx) != FINISHED_SUM_TAG * 2);
        auto dst_ptr = reinterpret_cast<uint64_t>(rdma_recv_count + dst_expert_local_idx * num_ranks + rank);
        auto dst_p2p_ptr = nvshmemi_get_p2p_ptr(dst_ptr, rank, dst_rank);
        if (dst_p2p_ptr == 0) {
            nvshmemi_ibgda_amo_nonfetch_add(reinterpret_cast<int*>(dst_ptr), -num_tokens_sent - 1, dst_rank, dst_expert_local_idx);
        } else {
            st_release_sys_global(reinterpret_cast<int*>(dst_p2p_ptr), -num_tokens_sent - 1);
        }

        // Clean workspace for next use
        atomic_counter_per_expert[responsible_expert_idx] = 0;
        atomic_finish_counter_per_expert[responsible_expert_idx] = 0;

        // Clean `packed_recv_count`
        if (dst_rank == 0)
            packed_recv_count[dst_expert_local_idx] = 0;
    }
    __syncwarp();

    // Receiving phase
    LOW_LATENCY_DISPATCH_RECV:
    if ((phases & LOW_LATENCY_RECV_PHASE) == 0)
        return;

    // For send-and-recv kernels, we need a grid sync for making `packed_recv_count` visible
    if (phases & LOW_LATENCY_SEND_PHASE)
        cg::this_grid().sync();

    // Receiving and packing
    if (responsible_expert_idx < num_experts) {
        const auto src_rank = responsible_expert_idx / num_local_experts;
        const auto local_expert_idx = responsible_expert_idx % num_local_experts;
        const auto rdma_recv_x_uint8 = static_cast<uint8_t*>(rdma_recv_x) +
                local_expert_idx * num_ranks * num_max_dispatch_tokens_per_rank * num_bytes_per_msg +
                src_rank * num_max_dispatch_tokens_per_rank * num_bytes_per_msg;
        const auto recv_x_int4 = static_cast<int4*>(packed_recv_x) +
                local_expert_idx * num_ranks * num_max_dispatch_tokens_per_rank * hidden_int4;
        const auto recv_src_info = packed_recv_src_info + local_expert_idx * num_ranks * num_max_dispatch_tokens_per_rank;
        const auto recv_range = packed_recv_layout_range + local_expert_idx * num_ranks;
        const auto num_aligned_scales = align<int>(num_scales, sizeof(float) / sizeof(scale_t));
        const auto recv_x_scales = static_cast<scale_t*>(packed_recv_x_scales) + local_expert_idx * num_ranks * num_max_dispatch_tokens_per_rank * num_aligned_scales;

        // Shared between sub-warps in warp groups
        __shared__ int shared_num_recv_tokens[kNumMaxWarpGroups], shared_recv_token_begin_idx[kNumMaxWarpGroups];

        // Wait tokens to arrive
        // NOTES: using sub-warp 1 to overlap with sub-warp 0
        int num_recv_tokens, recv_token_begin_idx;
        EP_DEVICE_ASSERT(num_warps_per_group > 1 and num_warp_groups < 15);
        if (sub_warp_id == 1 and lane_id == 0) {
            while ((num_recv_tokens = ld_acquire_sys_global(rdma_recv_count + local_expert_idx * num_ranks + src_rank)) == 0);
            num_recv_tokens = -num_recv_tokens - 1;
            recv_token_begin_idx = atomicAdd(packed_recv_count + local_expert_idx, num_recv_tokens);
            shared_num_recv_tokens[warp_group_id] = num_recv_tokens;
            shared_recv_token_begin_idx[warp_group_id] = recv_token_begin_idx;
            recv_range[src_rank] = pack2<int, int64_t>(num_recv_tokens, recv_token_begin_idx);
            if (cumulative_local_expert_recv_stats != nullptr)
                atomicAdd(cumulative_local_expert_recv_stats + local_expert_idx, num_recv_tokens);
        }
        asm volatile("bar.sync %0, %1;" :: "r"(warp_group_id + 2), "r"(num_warps_per_group * 32));
        num_recv_tokens = shared_num_recv_tokens[warp_group_id];
        recv_token_begin_idx = shared_recv_token_begin_idx[warp_group_id];

        // Copy tokens
        EP_DEVICE_ASSERT(num_scales <= 64);
        for (int i = sub_warp_id; i < num_recv_tokens; i += num_warps_per_group) {
            // Copy source info
            const auto src_src_idx = reinterpret_cast<int*>(rdma_recv_x_uint8 + i * num_bytes_per_msg);
            if (lane_id == 0)
                recv_src_info[recv_token_begin_idx + i] = ld_nc_global(src_src_idx);
            __syncwarp();

            // Copy data
            // NOTES: only 2 load iterations for 7K hidden with 7 unrolls
            const auto src_data = reinterpret_cast<int4*>(reinterpret_cast<uint8_t*>(src_src_idx) + sizeof(int4));
            const auto dst_data = recv_x_int4 + (recv_token_begin_idx + i) * hidden_int4;
            UNROLLED_WARP_COPY(7, lane_id, hidden_int4, dst_data, src_data, ld_nc_global, st_na_global);

            // Copy scales
            if constexpr (kUseFP8) {
                // Equivalent CuTe layout:
                //   (num_tokens, (num_packed, num_elems_per_pack)):(num_elems_per_pack, (num_tokens * num_elems_per_pack, 1))
                const auto src_scales = reinterpret_cast<float*>(reinterpret_cast<uint8_t*>(src_data) + hidden_bytes);
                const auto num_elems_per_pack = static_cast<int>(sizeof(packed_t) / sizeof(scale_t));
                const auto token_idx = recv_token_begin_idx + i;
                const auto token_stride = num_elems_per_pack;
                const auto pack_stride = num_ranks * num_max_dispatch_tokens_per_rank * num_elems_per_pack;
                if (lane_id < num_scales) {
                    const auto pack_idx = lane_id / num_elems_per_pack;
                    const auto elem_idx = lane_id % num_elems_per_pack;
                    auto scale = extract_required_scale_format<kUseUE8M0>(ld_nc_global(src_scales + lane_id));
                    recv_x_scales[token_idx * token_stride + pack_idx * pack_stride + elem_idx] = scale;
                }
                if (lane_id + 32 < num_scales) {
                    const auto pack_idx = (lane_id + 32) / num_elems_per_pack;
                    const auto elem_idx = (lane_id + 32) % num_elems_per_pack;
                    auto scale = extract_required_scale_format<kUseUE8M0>(ld_nc_global(src_scales + lane_id + 32));
                    recv_x_scales[token_idx * token_stride + pack_idx * pack_stride + elem_idx] = scale;
                }
            }
        }
    }
}

void dispatch(void* packed_recv_x, void* packed_recv_x_scales,
              int* packed_recv_src_info, int64_t* packed_recv_layout_range,
              int* packed_recv_count,
              int* cumulative_local_expert_recv_stats,
              void* rdma_recv_x, int* rdma_recv_count, void* rdma_x,
              const void* x, const int64_t* topk_idx,
              int* next_clean, int num_next_clean_int,
              int num_tokens, int hidden, int num_max_dispatch_tokens_per_rank,
              int num_topk, int num_experts, int rank, int num_ranks,
              bool use_fp8, bool round_scale, bool use_ue8m0,
              void* workspace, int num_device_sms,
              hipStream_t stream, int phases) {
    constexpr int kNumMaxTopK = 9;
    const int num_warp_groups = ceil_div(num_experts, num_device_sms);
    const int num_warps_per_group = 32 / num_warp_groups;
    EP_HOST_ASSERT(num_warp_groups > 0 and num_warps_per_group > 0);
    EP_HOST_ASSERT(kNumMaxTopK + 1 <= num_warp_groups * num_warps_per_group);

    const auto num_warps = num_warp_groups * num_warps_per_group;
    const auto num_sms = ceil_div(num_experts, num_warp_groups);
    EP_HOST_ASSERT(num_topk <= kNumMaxTopK);

    // Workspace checks
    auto atomic_counter_per_expert = static_cast<int*>(workspace);
    auto atomic_finish_counter_per_expert = atomic_counter_per_expert + num_experts;
    EP_HOST_ASSERT(num_experts * sizeof(int) * 2 <= NUM_WORKSPACE_BYTES);

    // FP8 checks
    if (use_ue8m0)
        EP_HOST_ASSERT(round_scale and "UE8M0 SF requires `round_scale=True`");

#define DISPATCH_LAUNCH_CASE(hidden) { \
auto dispatch_func = dispatch<false, false, hidden>; \
if (use_fp8 and not use_ue8m0) \
    dispatch_func = dispatch<true, false, hidden>; \
if (use_fp8 and use_ue8m0) \
    dispatch_func = dispatch<true, true, hidden>; \
LAUNCH_KERNEL(&cfg, dispatch_func, \
              packed_recv_x, packed_recv_x_scales, \
              packed_recv_src_info, packed_recv_layout_range, \
              packed_recv_count, \
              cumulative_local_expert_recv_stats, \
              rdma_recv_x, rdma_recv_count, rdma_x, \
              x, topk_idx, \
              atomic_counter_per_expert, atomic_finish_counter_per_expert, \
              next_clean, num_next_clean_int, \
              num_tokens, num_max_dispatch_tokens_per_rank, \
              num_topk, num_experts, rank, num_ranks, \
              num_warp_groups, num_warps_per_group, \
              round_scale, phases); } break

    SETUP_LAUNCH_CONFIG(num_sms, num_warps * 32, stream);
    SWITCH_HIDDEN(DISPATCH_LAUNCH_CASE);
#undef DISPATCH_LAUNCH_CASE
}

template <int kNumSendUnrolls>
__forceinline__ __device__ int logfmt_encode(uint32_t* ld_buffer, uint32_t* st_buffer, uint32_t *shared_amaxmin) {
    constexpr int kNumElemsPerInt4 = sizeof(int4) / sizeof(hip_bfloat16);
    constexpr float kLogThreshold = 0;
    constexpr float kMinClip = 32; // `== log_2(2 ^ (2 ^ 5))`
    constexpr int kNumBits = 10;
    constexpr int kNumValues = 1 << (kNumBits - 1);

    int4 int4_values[kNumSendUnrolls];
    const auto& uint32_values = reinterpret_cast<uint32_t*>(int4_values);
    EP_STATIC_ASSERT(sizeof(uint32_t) == sizeof(nv_bfloat162), "Invalid length");

    // Local log amax
    float log_abs[kNumElemsPerInt4 * kNumSendUnrolls];
    auto bf162_amax = __hip_bfloat162(CUDART_ZERO_BF16, CUDART_ZERO_BF16);
    auto bf162_amin = __hip_bfloat162(CUDART_INF_BF16, CUDART_INF_BF16);
    uint32_t local_signs = 0;
    #pragma unroll
    for (int k = 0; k < kNumSendUnrolls * sizeof(int4) / sizeof(nv_bfloat162); ++ k) {
        // TODO: eliminate bank conflicts
        uint32_values[k] = ld_buffer[k];
        auto uint32_abs = uint32_values[k] & 0x7fff7fff;
        local_signs |= ((uint32_values[k] >> 15) & 1) << (k * 2);
        local_signs |= ((uint32_values[k] >> 31) & 1) << (k * 2 + 1);

        const auto& bf162_abs = *reinterpret_cast<__hip_bfloat162*>(&uint32_abs);
        bf162_amax = __hmax2(bf162_amax, bf162_abs);
        bf162_amin = __hmin2(bf162_amin, bf162_abs);

        const auto& float2_abs = __bfloat1622float2(bf162_abs);
        log_abs[k * 2 + 0] = log2f_approx(float2_abs.x);
        log_abs[k * 2 + 1] = log2f_approx(float2_abs.y);
    }

    // Reduce per 128 channels
    // TODO: figure out how hardware do 2-byte min/max
    auto bf16_amax = __hmax(bf162_amax.x, bf162_amax.y);
    auto bf16_amin = __hmin(bf162_amin.x, bf162_amin.y);
    constexpr static int kNumLanesToReduce = 128 * sizeof(hip_bfloat16) / (kNumSendUnrolls * sizeof(int4));
    bf16_amax = warp_reduce_max<kNumLanesToReduce>(bf16_amax);
    bf16_amin = warp_reduce_min<kNumLanesToReduce>(bf16_amin);

    // Write min/max into the shared memory
    auto bf162_amaxmin = __hip_bfloat162(bf16_amax, bf16_amin);
    *shared_amaxmin = *reinterpret_cast<uint32_t*>(&bf162_amaxmin);

    // Calculate log amin/amax float
    const auto& amax = static_cast<float>(bf16_amax);
    const auto& amin = static_cast<float>(bf16_amin);
    const auto& log_amax = log2f_approx(amax);
    const auto& log_amin = amin == 0 ? log_amax - kMinClip : fmaxf(log2f_approx(amin), log_amax - kMinClip);
    const bool& enable_cast = warp_reduce_and<kNumLanesToReduce, true>(log_amax < kLogThreshold and log_amin < log_amax);

    // Case into LogFMT-10 if satisfied
    if (enable_cast) {
        const auto step = (log_amax - log_amin) / static_cast<float>(kNumValues - 2);
        const auto step_inv = 1.0f / step;
        const auto rounding = 2.0f - log2f_approx((1.0f + exp2f_approx(step)) * 0.5f) * step_inv;
        const auto fused_rounding = rounding - log_amin * step_inv;

        auto encode = [=](const float& x) {
            return __float2uint_rd(x < log_amax - kMinClip ? 0.0f : x * step_inv + fused_rounding);
        };

        // Pack every 256 bits into 160 bits
        EP_STATIC_ASSERT(kNumSendUnrolls == 2 or kNumSendUnrolls == 4, "kNumSendUnrolls == 2 or 4 only");
        #pragma unroll
        for (int i = 0; i < kNumSendUnrolls / 2; ++ i) {
            uint32_t concat[6];
            #pragma unroll
            for (int k = 0; k < 5; ++ k)
                concat[k] = encode(log_abs[i * 16 + k * 3]) | (encode(log_abs[i * 16 + k * 3 + 1]) << 9) | (encode(log_abs[i * 16 + k * 3 + 2]) << 18);
            concat[5] = encode(log_abs[i * 16 + 15]);
            #pragma unroll
            for (int k = 0; k < 5; ++ k)
                st_buffer[i * 5 + k] = (concat[k] >> (k * 5)) | (concat[k + 1] << (27 - k * 5));
            st_buffer[i * 5 + 4] |= (local_signs >> 16 * i) << 16;
        }
    }
    tma_store_fence();
    __syncwarp();
    // TODO: make `enable_cast` as a template?
    return 32 * (kNumSendUnrolls * sizeof(int4) * 8 * (enable_cast ? 10 : 16) / 16 / 8);
}

template <int kNumLanes, int kNumSendUnrolls, int kNumRecvUnrolls>
__forceinline__ __device__ void logfmt_check_amaxmin(uint8_t* meta_buffer, float2* shared_log_amax,
                                                     float2* shared_log_amin, int* shared_cast_info,
                                                     const int lane_id) {
    constexpr float kLogThreshold = 0;
    constexpr float kMinClip = 32; // `== log_2(2 ^ (2 ^ 5))`

    bool enable_cast = true;
    if (lane_id < kNumLanes) {
        // Calculate log amin/amax float
        auto amaxmin2 = reinterpret_cast<uint64_t*>(meta_buffer)[lane_id];
        const auto& bf162_amaxmin = reinterpret_cast<__hip_bfloat162*>(&amaxmin2);
        float log_amax[2], log_amin[2];
        #pragma unroll
        for (int i = 0; i < 2; ++ i) { 
            auto amax = static_cast<float>(bf162_amaxmin[i].x);
            auto amin = static_cast<float>(bf162_amaxmin[i].y);
            log_amax[i] = log2f_approx(amax);
            log_amin[i] = amin == 0 ? log_amax[i] - kMinClip : fmaxf(log2f_approx(amin), log_amax[i] - kMinClip);
            enable_cast = enable_cast and log_amax[i] < kLogThreshold and log_amin[i] < log_amax[i];
        }
        shared_log_amax[lane_id] = make_float2(log_amax[0], log_amax[1]);
        shared_log_amin[lane_id] = make_float2(log_amin[0], log_amin[1]);
    }

    const auto& cast_bit = warp_reduce_and<kNumSendUnrolls>(enable_cast) ? 1u << (lane_id / kNumRecvUnrolls): 0u;
    const auto& cast_prefix_count = std::__popcount(warp_reduce_or<kNumRecvUnrolls, true>(cast_bit) & ((1u << (lane_id / kNumRecvUnrolls)) - 1));

    if (lane_id < kNumLanes)
        shared_cast_info[lane_id / kNumRecvUnrolls] = (cast_prefix_count << 1) | (cast_bit ? 1u : 0u);
}

template <int kHidden, int kNumRecvUnrolls>
__forceinline__ __device__ void logfmt_decode_and_accumulate(uint32_t* ld_buffer, float* accum,
                                                             const float& log_amax, const float& log_amin,
                                                             const bool& enable_cast, const float& weight) {
    constexpr int kNumElemsPerInt4 = sizeof(int4) / sizeof(hip_bfloat16);
    constexpr int kNumBits = 10;
    constexpr int kNumValues = 1 << (kNumBits - 1);
    EP_STATIC_ASSERT(kHidden % (kNumElemsPerInt4 * 32) == 0 and kNumElemsPerInt4 == 8, "Invalid hidden");

    if (enable_cast) {
        const auto step = (log_amax - log_amin) / static_cast<float>(kNumValues - 2);
        auto decode = [=](const uint32_t &encoded, const uint32_t &op) {
            const auto decoded = encoded == 0 ? .0f : exp2f_approx((encoded - 1) * step + log_amin);
            return op ? -decoded : decoded;
        };

        EP_STATIC_ASSERT(kNumRecvUnrolls == 2 or kNumRecvUnrolls == 4, "kNumRecvUnrolls == 2 or 4 only");
        #pragma unroll
        for (int i = 0; i < kNumRecvUnrolls / 2; ++ i) {
            uint32_t concat[6];
            concat[0] = ld_buffer[i * 5];
            #pragma unroll
            for (int k = 1; k < 5; ++ k)
                concat[k] = (ld_buffer[i * 5 + k - 1] >> (32 - k * 5)) | (ld_buffer[i * 5 + k] << (k * 5));
            concat[5] = ld_buffer[i * 5 + 4] >> 7;

            const uint32_t& local_signs = ld_buffer[i * 5 + 4] >> 16;
            #pragma unroll
            for (int k = 0; k < 5; ++ k) {
                accum[i * 16 + k * 3 + 0] += decode((concat[k] >>  0) & 0x1ff, (local_signs >> (k * 3 + 0)) & 1) * weight;
                accum[i * 16 + k * 3 + 1] += decode((concat[k] >>  9) & 0x1ff, (local_signs >> (k * 3 + 1)) & 1) * weight;
                accum[i * 16 + k * 3 + 2] += decode((concat[k] >> 18) & 0x1ff, (local_signs >> (k * 3 + 2)) & 1) * weight;
            }
            accum[i * 16 + 15] += decode(concat[5] & 0x1ff, (local_signs >> 15) & 1) * weight;
        }
    } else {
        #pragma unroll
        for (int k = 0; k < kNumRecvUnrolls * 4; ++ k) {
            auto bf16_pack = *reinterpret_cast<__hip_bfloat162*>(ld_buffer + k);
            accum[k * 2 + 0] += static_cast<float>(bf16_pack.x) * weight;
            accum[k * 2 + 1] += static_cast<float>(bf16_pack.y) * weight;
        }
    }
}

template <bool kUseLogFMT, int kHidden, int kNumMaxTopk>
__global__ __launch_bounds__(1024, 1) void
combine(void* combined_x,
        void* rdma_recv_x, int* rdma_recv_flag, void* rdma_send_x,
        const void* x, const int64_t* topk_idx, const float* topk_weights,
        const int* src_info, const int64_t* layout_range,
        int* next_clean, int num_next_clean_int,
        int* atomic_clean_flag,
        int num_combined_tokens, int hidden, int num_topk,
        int num_max_dispatch_tokens_per_rank,
        int num_experts, int rank, int num_ranks,
        int num_warp_groups, int num_warps_per_group,
        int phases, bool zero_copy) {
    const auto sm_id = static_cast<int>(blockIdx.x);
    const auto num_sms = static_cast<int>(gridDim.x);
    const auto thread_id = static_cast<int>(threadIdx.x);
    const auto num_threads = static_cast<int>(blockDim.x);
    const auto warp_id = thread_id / 32, lane_id = get_lane_id();
    const auto num_local_experts = num_experts / num_ranks;
    const auto warp_group_id = warp_id / num_warps_per_group;
    const auto sub_warp_id = warp_id % num_warps_per_group;
    const auto responsible_expert_idx = sm_id * num_warp_groups + warp_group_id;

    extern __shared__ __align__(1024) uint8_t smem_buffer[];

    // Data type staffs
    constexpr int kNumElemsPerInt4 = sizeof(int4) / sizeof(hip_bfloat16);
    constexpr int64_t hidden_bf16_int4 = kHidden / kNumElemsPerInt4;
    // Use different unroll factors for send & recv phase
    constexpr int kNumSendUnrolls = 4;
    constexpr int kNumRecvUnrolls = 2;
    constexpr int hidden_bf16_int4_pad = align(static_cast<int>(hidden_bf16_int4), 32 * kNumSendUnrolls);
    EP_STATIC_ASSERT(hidden_bf16_int4 % kNumSendUnrolls == 0, "Invalid hidden");
    EP_STATIC_ASSERT(kNumSendUnrolls >= kNumRecvUnrolls, "Invalid unroll factors");

    // Message package
    EP_STATIC_ASSERT(kHidden % 128 == 0, "Invalid hidden");
    constexpr int kNumDivisions = kHidden / 128;
    constexpr int kNumMetaBytes = kNumDivisions * sizeof(uint32_t);
    constexpr size_t num_bytes_per_slot = kHidden * sizeof(hip_bfloat16) + kNumMetaBytes;
    EP_STATIC_ASSERT(num_bytes_per_slot % sizeof(int4) == 0, "Invalid vectorization");

    // Sending phase
    if ((phases & LOW_LATENCY_SEND_PHASE) == 0)
        goto LOW_LATENCY_COMBINE_RECV;

    // Clean up next buffer
    if (sm_id == 0 and warp_group_id == 0 and sub_warp_id == 0) {
        #pragma unroll
        for (int i = lane_id; i < num_next_clean_int; i += 32)
            next_clean[i] = 0;

        // Notify before executing `int_p`
        __syncwarp();
        if (lane_id == 0)
            atomic_add_release_global(atomic_clean_flag, num_experts);
    }

    // Issue IBGDA sends
    if (responsible_expert_idx < num_experts) {
        const auto dst_rank = responsible_expert_idx / num_local_experts;
        const auto local_expert_idx = responsible_expert_idx % num_local_experts;
        const auto global_expert_idx = rank * num_local_experts + local_expert_idx;
        const auto layout = __ldg(layout_range + local_expert_idx * num_ranks + dst_rank);
        const auto local_x = static_cast<const int4*>(x) +
                local_expert_idx * num_ranks * num_max_dispatch_tokens_per_rank * hidden_bf16_int4;
        const auto local_src_info = src_info + local_expert_idx * num_ranks * num_max_dispatch_tokens_per_rank;
        const auto rdma_send_x_vec = static_cast<uint8_t*>(rdma_send_x) +
                local_expert_idx * num_ranks * num_max_dispatch_tokens_per_rank * num_bytes_per_slot;

        // Unpack layout
        int offset, num_tokens_to_send;
        unpack2(layout, num_tokens_to_send, offset);

        // TMA stuffs
        constexpr int kNumTMABufferBytes = sizeof(int4) * 32 * kNumSendUnrolls;
        constexpr int kNumStages = 3;
        constexpr int kNumPrefetch = 1;
        EP_STATIC_ASSERT(kNumStages == 3 and kNumPrefetch == 1, "Invalid stages");

        auto smem_ptr = smem_buffer + warp_id * (kNumStages * (kNumTMABufferBytes + 16) + kNumMetaBytes);
        uint32_t tma_phase[kNumStages] = {0};
        auto tma_buffer   = PatternVisitor([=](const int& i) { return reinterpret_cast<int4*>(smem_ptr + i * (kNumTMABufferBytes + 16)); });
        auto tma_mbarrier = PatternVisitor([=](const int& i) { return reinterpret_cast<uint64_t*>(smem_ptr + i * (kNumTMABufferBytes + 16) + kNumTMABufferBytes); });
        auto meta_buffer  = kUseLogFMT ? reinterpret_cast<uint32_t*>(smem_ptr + kNumStages * (kNumTMABufferBytes + 16)) : nullptr;
        EP_STATIC_ASSERT(kNumSendUnrolls * kNumStages <= 12, "TMA buffer size exceed limit");

        // Initialize m-barriers
        if (lane_id < kNumStages) {
            mbarrier_init(tma_mbarrier[lane_id], 1);
            fence_view_async_shared();
            fence_barrier_init();
        }
        __syncwarp();

        constexpr int kNumIters = hidden_bf16_int4_pad / (32 * kNumSendUnrolls);
        auto tma_load_and_arrive = [&](const int& stage_idx, const int4* gmem_ptr, const int& num_bytes) {
            tma_load_1d(tma_buffer[stage_idx], gmem_ptr, tma_mbarrier[stage_idx], num_bytes);
            mbarrier_arrive_and_expect_tx(tma_mbarrier[stage_idx], num_bytes);
        };
        auto get_num_tma_bytes = [&](const int& offset_int4) {
            return min(kNumTMABufferBytes, static_cast<int>((hidden_bf16_int4 - offset_int4) * sizeof(int4)));
        };

        // Issue IBGDA send
        for (int token_idx = offset + sub_warp_id; token_idx < offset + num_tokens_to_send; token_idx += num_warps_per_group) {
            const auto x_int4 = local_x + token_idx * hidden_bf16_int4;
            const auto rdma_send_type_row = reinterpret_cast<int*>(rdma_send_x_vec + token_idx * num_bytes_per_slot);
            const auto rdma_send_x_vec_row = reinterpret_cast<uint8_t*>(rdma_send_type_row);

            // Copy directly to local rank, or copy to buffer and issue RDMA
            const auto src_idx = __shfl_sync(0xffffffff, __ldg(local_src_info + token_idx), 0);
            const auto buf_ptr = reinterpret_cast<int64_t>(rdma_send_x_vec_row);
            const auto dst_ptr = reinterpret_cast<uint64_t>(rdma_recv_x) + (global_expert_idx * num_max_dispatch_tokens_per_rank + src_idx) * num_bytes_per_slot;
            const auto dst_p2p_ptr = nvshmemi_get_p2p_ptr(dst_ptr, rank, dst_rank);
            int num_send_bytes = hidden * sizeof(hip_bfloat16);

            if (not zero_copy or dst_p2p_ptr != 0) {
                // Read from `cpy_src_int4_ptr` and copy into `cpy_dst_int4_ptr`
                const auto cpy_src_int4_ptr = zero_copy ? reinterpret_cast<int4*>(buf_ptr) : x_int4;
                const auto cpy_dst_int4_ptr = dst_p2p_ptr == 0 ? reinterpret_cast<int4*>(buf_ptr) : reinterpret_cast<int4*>(dst_p2p_ptr);

                // Prefetch
                if (elect_one_sync(lane_id))
                    tma_load_and_arrive(0, cpy_src_int4_ptr, get_num_tma_bytes(0));
                __syncwarp();

                int tma_offset_bytes = kNumMetaBytes;
                #pragma unroll
                for (int i = lane_id * kNumSendUnrolls, iter_idx = 0; i < hidden_bf16_int4_pad; i += 32 * kNumSendUnrolls, ++ iter_idx) {
                    // Load the next iteration
                    const int& stage_idx = iter_idx % kNumStages;
                    const int& next_stage_idx = (iter_idx + 1) % kNumStages;
                    if (iter_idx + 1 < kNumIters and elect_one_sync(lane_id)) {
                        tma_store_wait<kNumStages - kNumPrefetch - 1>();
                        const auto& offset_int4 = i + 32 * kNumSendUnrolls;
                        tma_load_and_arrive(next_stage_idx, cpy_src_int4_ptr + offset_int4, get_num_tma_bytes(offset_int4));
                    }
                    __syncwarp();

                    // Wait the current TMA arrival
                    mbarrier_wait(tma_mbarrier[stage_idx], tma_phase[stage_idx]);
                    const auto& ld_buffer = reinterpret_cast<uint32_t*>(tma_buffer[stage_idx] + lane_id * kNumSendUnrolls);

                    if constexpr (kUseLogFMT) {
                        // Cast if possible
                        auto st_buffer = reinterpret_cast<uint32_t*>(tma_buffer[stage_idx]) + lane_id * kNumSendUnrolls * kNumElemsPerInt4 * 10 / 32;
                        int num_tma_bytes = logfmt_encode<kNumSendUnrolls>(ld_buffer, st_buffer, meta_buffer + i * kNumElemsPerInt4 / 128);
                        if (elect_one_sync(lane_id))
                            tma_store_1d(tma_buffer[stage_idx], reinterpret_cast<uint8_t*>(cpy_dst_int4_ptr) + tma_offset_bytes, num_tma_bytes);
                        tma_offset_bytes += num_tma_bytes;
                    } else {
                        // BF16 original values
                        if (elect_one_sync(lane_id))
                            tma_store_1d(tma_buffer[stage_idx], cpy_dst_int4_ptr + i, get_num_tma_bytes(i));
                    }
                }

                // Store metadata (min/max values) for LogFMT
                if constexpr (kUseLogFMT) {
                    num_send_bytes = tma_offset_bytes;
                    if (elect_one_sync(lane_id))
                        tma_store_1d(meta_buffer, cpy_dst_int4_ptr, kNumMetaBytes);
                }

                // Flush all stores
                tma_store_wait();
                __syncwarp();
            }

            // Issue RDMA
            // NOTES: for zero-copy mode, we assume the data is already in the send buffer
            if (dst_p2p_ptr == 0)
                nvshmemi_ibgda_put_nbi_warp(dst_ptr, buf_ptr, num_send_bytes, dst_rank, local_expert_idx, lane_id, token_idx - offset);
        }

        // Put the finishing flag
        EP_DEVICE_ASSERT(num_warps_per_group > 1 and num_warp_groups < 16);
        asm volatile("bar.sync %0, %1;" :: "r"(warp_group_id + 1), "r"(num_warps_per_group * 32));
        if (sub_warp_id == 1 and lane_id == 0) {
            while (ld_acquire_global(atomic_clean_flag) == 0);
            auto dst_ptr = reinterpret_cast<uint64_t>(rdma_recv_flag + global_expert_idx);
            auto dst_p2p_ptr = nvshmemi_get_p2p_ptr(dst_ptr, rank, dst_rank);
            if (dst_p2p_ptr == 0) {
                nvshmemi_ibgda_amo_nonfetch_add(reinterpret_cast<int*>(dst_ptr), 1, dst_rank, local_expert_idx);
            } else {
                st_release_sys_global(reinterpret_cast<int*>(dst_p2p_ptr), 1);
            }
            atomic_add_release_global(atomic_clean_flag, -1);
        }
        __syncwarp();

        // Destroy m-barriers
        if (lane_id < kNumStages) {
            mbarrier_inval(tma_mbarrier[lane_id]);
            fence_view_async_shared();
            fence_barrier_init();
        }
        __syncwarp();
    }

    // Receiving phase
    LOW_LATENCY_COMBINE_RECV:
    if ((phases & LOW_LATENCY_RECV_PHASE) == 0)
        return;

    // Wait all ranks to arrive
    if (responsible_expert_idx < num_experts) {
        EP_DEVICE_ASSERT(num_warps_per_group > 1);
        if (sub_warp_id == 0 and lane_id == 0) {
            while (ld_acquire_sys_global(rdma_recv_flag + responsible_expert_idx) == 0);
        }
    }
    cg::this_grid().sync();

    // Reduce tokens
    EP_DEVICE_ASSERT(num_topk <= 32);
    EP_STATIC_ASSERT(kHidden % (32 * kNumElemsPerInt4) == 0, "Invalid vectorization");
    if constexpr (kUseLogFMT) {
        // TODO: reduce registers usage
        // Reassign warp groups
        constexpr int kMaxNumGroups = 2;
        const int num_decode_warps = hidden_bf16_int4_pad / (kNumRecvUnrolls * 32);
        const int num_groups = min(kMaxNumGroups, (num_threads / 32) / (num_decode_warps + 1));
        const int decode_warp_idx = __shfl_sync(0xffffffff, warp_id % (num_decode_warps + 1), 0);
        const int group_idx = __shfl_sync(0xffffffff, warp_id / (num_decode_warps + 1), 0);
        EP_DEVICE_ASSERT(num_groups > 0);
        if (group_idx >= num_groups)
            return;

        // Reallocate shared memory
        constexpr int kNumStages = 3;
        constexpr int kNumTMABufferBytes = 16 * 2 + kHidden * 2;
        constexpr int kNumBF16PerWarpBytes = 32 * kNumRecvUnrolls * kNumElemsPerInt4 * 2;
        constexpr int kNumLogFMTPerWarpBytes = kNumBF16PerWarpBytes / 16 * 10;
        constexpr int kNumDivisionBytes = kNumDivisions * sizeof(uint32_t);
        constexpr int kNumBytesPerGroup = kNumStages * kNumTMABufferBytes + kHidden * 2 + kNumStages * kNumDivisionBytes * 3;

        uint32_t tma_phase[kNumStages];
        #pragma unroll
        for (int i = 0; i < kNumStages; ++ i)
            tma_phase[i] = (decode_warp_idx == num_decode_warps ? 1 : 0);

        const auto smem_group_buffer = smem_buffer + kNumBytesPerGroup * group_idx;
        auto full_barriers  = PatternVisitor([=](const int& k) { return reinterpret_cast<uint64_t*>(smem_group_buffer + k * kNumTMABufferBytes); });
        auto empty_barriers = PatternVisitor([=](const int& k) { return reinterpret_cast<uint64_t*>(smem_group_buffer + k * kNumTMABufferBytes + 16); });
        auto tma_ld_buffer  = PatternVisitor([=](const int& k) { return reinterpret_cast<uint8_t* >(smem_group_buffer + k * kNumTMABufferBytes + 32); });
        auto tma_st_buffer  = PatternVisitor([=](const int& i) { return reinterpret_cast<uint32_t*>(smem_group_buffer + kNumStages * kNumTMABufferBytes + i * kNumBF16PerWarpBytes); });

        const auto smem_group_ptr = smem_group_buffer + kNumStages * kNumTMABufferBytes + kHidden * 2;
        auto log_amax  = PatternVisitor([=](const int& k) { return reinterpret_cast<float*>(smem_group_ptr + k * kNumDivisionBytes); });
        auto log_amin  = PatternVisitor([=](const int& k) { return reinterpret_cast<float*>(smem_group_ptr + kNumStages * kNumDivisionBytes + k * kNumDivisionBytes); });
        auto cast_info = PatternVisitor([=](const int& k) { return reinterpret_cast<int*>  (smem_group_ptr + kNumStages * kNumDivisionBytes * 2 + k * kNumDivisionBytes); });

        // Initialize m-barriers
        if (decode_warp_idx == num_decode_warps and lane_id < kNumStages) {
            mbarrier_init(full_barriers[lane_id], 1);
            mbarrier_init(empty_barriers[lane_id], num_decode_warps);
        }
        asm volatile("bar.sync %0, %1;" :: "r"(group_idx + 1), "r"((num_decode_warps + 1) * 32));

        int stage_idx = 0;
        for (int token_idx = sm_id + num_sms * group_idx; token_idx < num_combined_tokens; token_idx += num_sms * num_groups) {
            if (decode_warp_idx == num_decode_warps) {
                // TMA load warp
                #pragma unroll
                for (int i = 0; i < num_topk; ++ i) {
                    int reg_topk_idx = static_cast<int>(__ldg(topk_idx + token_idx * num_topk + i));
                    if (reg_topk_idx < 0)
                        continue;

                    mbarrier_wait(empty_barriers[stage_idx], tma_phase[stage_idx]);
                    auto buffer = static_cast<uint8_t*>(rdma_recv_x) + (reg_topk_idx * num_max_dispatch_tokens_per_rank + token_idx) * num_bytes_per_slot;
                    logfmt_check_amaxmin<kNumDivisions / 2, kNumSendUnrolls, kNumRecvUnrolls>(buffer, reinterpret_cast<float2*>(log_amax[stage_idx]),
                                                                                              reinterpret_cast<float2*>(log_amin[stage_idx]), cast_info[stage_idx], lane_id);
                    if (elect_one_sync(lane_id)) {
                        int cast_count = (cast_info[stage_idx][num_decode_warps - 1] >> 1) + (cast_info[stage_idx][num_decode_warps - 1] & 1);
                        int num_tma_bytes = cast_count * kNumLogFMTPerWarpBytes + (num_decode_warps - cast_count) * kNumBF16PerWarpBytes;
                        tma_load_1d(tma_ld_buffer[stage_idx], buffer + kNumMetaBytes, full_barriers[stage_idx], num_tma_bytes);
                        mbarrier_arrive_and_expect_tx(full_barriers[stage_idx], num_tma_bytes);
                    }
                    __syncwarp();
                    stage_idx = (stage_idx + 1) % kNumStages;
                }
            } else if (decode_warp_idx < num_decode_warps) {
                // Reduction warps
                float combined_values[kNumElemsPerInt4 * kNumRecvUnrolls] = {0.0f};
                #pragma unroll
                for (int i = 0; i < num_topk; ++ i) {
                    if (__ldg(topk_idx + token_idx * num_topk + i) < 0)
                        continue;

                    float topk_weight = __ldg(topk_weights + token_idx * num_topk + i);
                    mbarrier_wait(full_barriers[stage_idx], tma_phase[stage_idx]);

                    int cast_prefix_count = cast_info[stage_idx][decode_warp_idx] >> 1;
                    bool enable_cast = cast_info[stage_idx][decode_warp_idx] & 1;
                    int tma_offset = kNumLogFMTPerWarpBytes * cast_prefix_count + kNumBF16PerWarpBytes * (decode_warp_idx - cast_prefix_count);
                    int division_idx = decode_warp_idx * kNumRecvUnrolls * 2 + lane_id * kNumRecvUnrolls / 16;

                    logfmt_decode_and_accumulate<kHidden, kNumRecvUnrolls>(
                        reinterpret_cast<uint32_t*>(tma_ld_buffer[stage_idx] + tma_offset + (enable_cast ? kNumLogFMTPerWarpBytes : kNumBF16PerWarpBytes) / 32 * lane_id),
                        combined_values, log_amax[stage_idx][division_idx], log_amin[stage_idx][division_idx], enable_cast, topk_weight
                    );

                    if (elect_one_sync(lane_id))
                        mbarrier_arrive(empty_barriers[stage_idx]);
                    stage_idx = (stage_idx + 1) % kNumStages;
                }
                tma_store_wait<0>();

                #pragma unroll
                for (int k = 0; k < kNumRecvUnrolls * 4; ++ k) {
                    auto combined_pack = __hip_bfloat162(combined_values[k * 2], combined_values[k * 2 + 1]);
                    tma_st_buffer[decode_warp_idx][kNumRecvUnrolls * 4 * lane_id + k] = *reinterpret_cast<uint32_t*>(&combined_pack);
                }
                tma_store_fence();
                if (elect_one_sync(lane_id))
                    tma_store_1d(tma_st_buffer[decode_warp_idx], static_cast<int4*>(combined_x) + token_idx * hidden_bf16_int4 + decode_warp_idx * kNumRecvUnrolls * 32, kNumBF16PerWarpBytes);
            }
        }
        tma_store_wait<0>();
    } else {
        for (int hidden_idx = thread_id; hidden_idx < hidden_bf16_int4; hidden_idx += num_threads) {
            for (int token_idx = sm_id; token_idx < num_combined_tokens; token_idx += num_sms) {
                // Read top-k indices and weights
                int reg_topk_idx[kNumMaxTopk];
                float reg_topk_weights[kNumMaxTopk];
                #pragma unroll
                for (int i = 0; i < num_topk; ++ i) {
                    reg_topk_idx[i] = static_cast<int>(__ldg(topk_idx + token_idx * num_topk + i));
                    reg_topk_weights[i] = __ldg(topk_weights + token_idx * num_topk + i);
                }

                float combined_values[kNumElemsPerInt4] = {0.0f};
                #pragma unroll
                for (int i = 0; i < num_topk; ++ i) if (reg_topk_idx[i] >= 0) {
                    // Read from sources
                    auto rdma_buffer_type = reinterpret_cast<const int*>(static_cast<uint8_t*>(rdma_recv_x) + (reg_topk_idx[i] * num_max_dispatch_tokens_per_rank + token_idx) * num_bytes_per_slot);
                    auto rdma_buffer_row = reinterpret_cast<const uint8_t*>(rdma_buffer_type);

                    // Reduce
                    auto x_vec = ld_nc_global(reinterpret_cast<const int4*>(rdma_buffer_row) + hidden_idx);
                    const auto x_bf16 = reinterpret_cast<hip_bfloat16*>(&x_vec);
                    #pragma unroll
                    for (int j = 0; j < kNumElemsPerInt4; ++ j)
                        combined_values[j] += static_cast<float>(x_bf16[j]) * reg_topk_weights[i];
                }

                // Write results
                int4& combined_int4 = *reinterpret_cast<int4*>(combined_values);
                auto combined_bf16 = reinterpret_cast<hip_bfloat16*>(&combined_values);
                #pragma unroll
                for (int j = 0; j < kNumElemsPerInt4; ++ j)
                    combined_bf16[j] = static_cast<hip_bfloat16>(combined_values[j]);
                (static_cast<int4*>(combined_x) + token_idx * hidden_bf16_int4)[hidden_idx] = combined_int4;
            }
        }
    }
}

void combine(void* combined_x,
             void* rdma_recv_x, int* rdma_recv_flag, void* rdma_send_x,
             const void* x, const int64_t* topk_idx, const float* topk_weights,
             const int* src_info, const int64_t* layout_range,
             int* next_clean, int num_next_clean_int,
             int num_combined_tokens, int hidden, int num_max_dispatch_tokens_per_rank,
             int num_topk, int num_experts, int rank, int num_ranks,
             bool use_logfmt,
             void* workspace, int num_device_sms,
             hipStream_t stream, int phases, bool zero_copy) {
    constexpr int kNumMaxTopk = 9;
    const int num_warp_groups = ceil_div(num_experts, num_device_sms);
    const int num_warps_per_group = 32 / num_warp_groups;
    const int num_recv_per_sm = ceil_div(num_combined_tokens, num_device_sms);
    EP_HOST_ASSERT(num_warp_groups > 0 and num_warps_per_group > 0 and num_recv_per_sm > 0);

    const auto num_warps = num_warp_groups * num_warps_per_group;
    const auto num_sms = max(ceil_div(num_experts, num_warp_groups), ceil_div(num_combined_tokens, num_recv_per_sm));

    // Check workspace
    auto atomic_clean_flag = static_cast<int*>(workspace);
    EP_HOST_ASSERT(sizeof(int) <= NUM_WORKSPACE_BYTES);
    EP_HOST_ASSERT(num_topk <= kNumMaxTopk);

    // Online cast cannot use zero-copy
    EP_HOST_ASSERT(not (zero_copy and use_logfmt));

    constexpr int kNumStages = 3;
    constexpr int kNumUnrolls = 4;
    constexpr int kMaxNumGroups = 2;
    const int num_meta_bytes = hidden / 128 * 4;
    const int num_send_tma_bytes = 32 * sizeof(int4) * kNumUnrolls + 16;
    const int smem_send_size = num_warps * (kNumStages * num_send_tma_bytes + num_meta_bytes);
    const int num_recv_tma_bytes = 16 * 2 + hidden * 2;
    const int smem_recv_size = kMaxNumGroups * (kNumStages * num_recv_tma_bytes + hidden * 2 + kNumStages * num_meta_bytes * 3);
    const int smem_size = max(smem_send_size, smem_recv_size);

#define COMBINE_LAUNCH_CASE(hidden) { \
auto combine_func = use_logfmt ? \
    combine<true, hidden, kNumMaxTopk> : \
    combine<false, hidden, kNumMaxTopk>; \
SET_SHARED_MEMORY_FOR_TMA(combine_func); \
LAUNCH_KERNEL(&cfg, combine_func, \
              combined_x, \
              rdma_recv_x, rdma_recv_flag, rdma_send_x, \
              x, topk_idx, topk_weights, src_info, layout_range, \
              next_clean, num_next_clean_int, \
              atomic_clean_flag, \
              num_combined_tokens, hidden, num_topk, \
              num_max_dispatch_tokens_per_rank, \
              num_experts, rank, num_ranks, \
              num_warp_groups, num_warps_per_group, \
              phases, zero_copy); } break

    SETUP_LAUNCH_CONFIG(num_sms, num_warps * 32, stream);
    SWITCH_HIDDEN(COMBINE_LAUNCH_CASE);
#undef COMBINE_LAUNCH_CASE
}

} // namespace internode_ll

} // namespace deep_ep
