#include "hip/hip_runtime.h"
#include "configs.cuh"
#include "exception.cuh"
#include "launch.cuh"
#include "ibgda_device.cuh"

#include "eager.cuh"
#include "utils.cuh"
#include <cstdint>
#include <cstdio>

namespace deep_ep {

namespace internode_ll {

template <int kNumThreads> __launch_bounds__(kNumThreads, 1)
__global__ void clean_low_latency_buffer(int* clean_0, int num_clean_int_0,
                                         int* clean_1, int num_clean_int_1) {
    // Barrier before cleaning (in case of unfinished chunked EP)
    nvshmemx_barrier_all_block();

    // Clean
    auto thread_id = static_cast<int>(threadIdx.x);
    #pragma unroll
    for (int i = thread_id; i < num_clean_int_0; i += kNumThreads)
        clean_0[i] = 0;
    #pragma unroll
    for (int i = thread_id; i < num_clean_int_1; i += kNumThreads)
        clean_1[i] = 0;

    // Barrier after cleaning (make sure the low-latency mode works fine)
    nvshmemx_barrier_all_block();
}

void clean_low_latency_buffer(int* clean_0, int num_clean_int_0,
                              int* clean_1, int num_clean_int_1,
                              hipStream_t stream) {
    constexpr int kNumThreads = 256;

    SETUP_LAUNCH_CONFIG(1, kNumThreads, stream);
    LAUNCH_KERNEL(&cfg, clean_low_latency_buffer<kNumThreads>,
                  clean_0, num_clean_int_0, clean_1, num_clean_int_1);
}


template <bool kUseFP8, bool kUseUE8M0, int kHidden>
__device__ __inline__ bool eager_dispatch_recv_token(
    void *rdma_recv_x, 
    int* rdma_recv_count,
    int responsible_expert_idx,
    int num_local_experts, 
    int num_experts,
    int rank, 
    int num_ranks, 
    int num_max_dispatch_tokens_per_rank, 
    size_t msg_distance,
    size_t num_bytes_per_msg,
    size_t num_bytes_per_msg_v,
    void* packed_recv_x, 
    void* packed_recv_x_scales,
    int* packed_recv_src_info,
    int64_t* packed_recv_layout_range,
    int* packed_recv_count,
    int* per_rank_recv_count,
    int& num_recv_tokens,
    int i,
    int dispatch_round_n
) {
    using scale_t = std::conditional_t<kUseUE8M0, uint8_t, float>;
    using packed_t = std::conditional_t<kUseUE8M0, uint32_t, float>;
    const auto lane_id = get_lane_id();
    constexpr int kNumPerChannels = 128;
    constexpr int num_scales = kHidden / kNumPerChannels;
    const size_t hidden_bytes = kHidden * (kUseFP8 ? sizeof(__hip_fp8_storage_t) : sizeof(hip_bfloat16));
    const size_t hidden_int4 = hidden_bytes / sizeof(int4);
    const auto src_rank = responsible_expert_idx / num_local_experts;
    const auto local_expert_idx = responsible_expert_idx % num_local_experts;
    const auto global_expert_idx = rank * num_local_experts + local_expert_idx;
    const auto rdma_recv_x_uint8 = static_cast<uint8_t*>(rdma_recv_x) +
            local_expert_idx * num_ranks * num_max_dispatch_tokens_per_rank * msg_distance +
            src_rank * num_max_dispatch_tokens_per_rank * msg_distance;
    const auto recv_x_int4 = static_cast<int4*>(packed_recv_x) +
            local_expert_idx * num_ranks * num_max_dispatch_tokens_per_rank * hidden_int4;
    const auto recv_src_info = packed_recv_src_info + local_expert_idx * num_ranks * num_max_dispatch_tokens_per_rank;
    //const auto recv_range = packed_recv_layout_range + local_expert_idx * num_ranks;
    const auto token_src_bitmap = reinterpret_cast<int*>(packed_recv_layout_range) + local_expert_idx * num_ranks * num_max_dispatch_tokens_per_rank + src_rank * num_max_dispatch_tokens_per_rank; // used in EAGER_FULL
    const auto per_rank_recv_cnt_ptr = per_rank_recv_count + local_expert_idx * num_ranks; // used in EAGER_FULL
    const auto ld_intra_node = (src_rank >> 3) == (rank >> 3); // TODO: identify intra node more elegantly
    const auto num_aligned_scales = align<int>(num_scales, sizeof(float) / sizeof(scale_t));
    const auto recv_x_scales = static_cast<scale_t*>(packed_recv_x_scales) + local_expert_idx * num_ranks * num_max_dispatch_tokens_per_rank * num_aligned_scales;

    const auto src_src_idx = reinterpret_cast<int*>(rdma_recv_x_uint8 + i * msg_distance);
    //const auto src_topk_idx = reinterpret_cast<int*>(reinterpret_cast<uint8_t*>(src_src_idx) + sizeof(int4));

    auto rcv_cnt_ptr = rdma_recv_count + local_expert_idx * num_ranks + src_rank;
    int this_token_index_in_recv = 0, shfl_value;
    int ready = 1;
    constexpr int kEager = EAGER_FULL;
    
    TRY_2BIT(src_src_idx, num_bytes_per_msg_v, num_bytes_per_msg, lane_id, 32, dispatch_round_n, rcv_cnt_ptr, num_recv_tokens, ready, i, ld_intra_node);
    ready = warp_reduce_min(ready);
    if (ready == 0) return false; // this token is not ready, go to next token
    
    if (lane_id == 0) {
        shfl_value = atomic_add_release_global(packed_recv_count + local_expert_idx, 1);
        int zzz = atomicAdd(per_rank_recv_cnt_ptr + src_rank, 1);
        //printf("[rank %d]: dispatch recv token expert %d slot %d get rank %d token %d, it is %d-th\n", rank, global_expert_idx, shfl_value, src_rank, LD_SHIFTED(ld_nc_global, src_src_idx, src_src_idx), zzz);
        token_src_bitmap[i] = shfl_value;
    }
    __syncwarp();

    this_token_index_in_recv = __shfl_sync(0xffffffff, shfl_value, 0);
    
#define DISPATCH_LD(ld_func, ptr) ((kEager != EAGER_OFF) ? LD_SHIFTED(ld_func, ptr, src_src_idx) : ld_func(ptr))

    if (lane_id == 0)
        recv_src_info[this_token_index_in_recv] = DISPATCH_LD(ld_nc_global, src_src_idx);
    __syncwarp();

    // Copy data
    // NOTES: only 2 load iterations for 7K hidden with 7 unrolls
    const auto src_data = reinterpret_cast<int4*>(reinterpret_cast<uint8_t*>(src_src_idx) + sizeof(int4));
    const auto dst_data = recv_x_int4 + this_token_index_in_recv * hidden_int4;
    //if (kEager != EAGER_OFF) {
    UNROLLED_WARP_COPY_SRC_AUTO_SHIFT(5, lane_id, hidden_int4, dst_data, src_data, src_src_idx, ld_nc_global, st_na_global);  
    // } else {
    //     UNROLLED_WARP_COPY(7, lane_id, hidden_int4, dst_data, src_data, ld_nc_global, st_na_global);
    // }

    // Copy scales
    if constexpr (kUseFP8) {
        // Equivalent CuTe layout:
        //   (num_tokens, (num_packed, num_elems_per_pack)):(num_elems_per_pack, (num_tokens * num_elems_per_pack, 1))
        const auto src_scales = reinterpret_cast<float*>(reinterpret_cast<uint8_t*>(src_data) + hidden_bytes);
        const auto num_elems_per_pack = static_cast<int>(sizeof(packed_t) / sizeof(scale_t));
        const auto token_idx = this_token_index_in_recv;
        const auto token_stride = num_elems_per_pack;
        const auto pack_stride = num_ranks * num_max_dispatch_tokens_per_rank * num_elems_per_pack;
        if (lane_id < num_scales) {
            const auto pack_idx = lane_id / num_elems_per_pack;
            const auto elem_idx = lane_id % num_elems_per_pack;
            auto scale = extract_required_scale_format<kUseUE8M0>(DISPATCH_LD(ld_nc_global, src_scales + lane_id));
            recv_x_scales[token_idx * token_stride + pack_idx * pack_stride + elem_idx] = scale;
        }
        if (lane_id + 32 < num_scales) {
            const auto pack_idx = (lane_id + 32) / num_elems_per_pack;
            const auto elem_idx = (lane_id + 32) % num_elems_per_pack;
            auto scale = extract_required_scale_format<kUseUE8M0>(DISPATCH_LD(ld_nc_global, src_scales + lane_id + 32));
            recv_x_scales[token_idx * token_stride + pack_idx * pack_stride + elem_idx] = scale;
        }
    }
    return true; // got a token
}

template <bool kUseFP8, bool kUseUE8M0, int kHidden, int kEager>
__global__ __launch_bounds__(1024, 1) void
dispatch(void* packed_recv_x, void* packed_recv_x_scales,
         int* packed_recv_src_info, int64_t* packed_recv_layout_range,
         int* packed_recv_count, int* per_rank_recv_count,
         int* cumulative_local_expert_recv_stats,
         int64_t* dispatch_wait_recv_cost_stats,
         void* rdma_recv_x, int* rdma_recv_count, void* rdma_x,
         const void* x, const int64_t* topk_idx,
         int* atomic_counter_per_expert, int* atomic_finish_counter_per_expert,
         int* next_clean, int num_next_clean_int,
         int num_tokens, int num_max_dispatch_tokens_per_rank,
         int num_topk, int num_experts, int rank, int num_ranks,
         int num_warp_groups, int num_warps_per_group,
         bool round_scale, int phases,
         int dispatch_round_n) {
    const auto sm_id = static_cast<int>(blockIdx.x);
    const auto thread_id = static_cast<int>(threadIdx.x);
    const auto warp_id = thread_id / 32, lane_id = get_lane_id();
    const auto num_sms = static_cast<int>(gridDim.x);
    const auto num_warps = num_warp_groups * num_warps_per_group;
    const auto num_local_experts = num_experts / num_ranks;
    const auto warp_group_id = warp_id / num_warps_per_group;
    const auto sub_warp_id = warp_id % num_warps_per_group;
    const auto responsible_expert_idx = sm_id * num_warp_groups + warp_group_id;

    // May extract UE8M0 from the scales
    using scale_t = std::conditional_t<kUseUE8M0, uint8_t, float>;
    using packed_t = std::conditional_t<kUseUE8M0, uint32_t, float>;
    EP_STATIC_ASSERT(sizeof(packed_t) % sizeof(scale_t) == 0, "Invalid vector length");

    // FP8 staffs
    constexpr int kNumPerChannels = 128;
    constexpr int num_scales = kHidden / kNumPerChannels;
    const size_t hidden_bytes = kHidden * (kUseFP8 ? sizeof(__hip_fp8_storage_t) : sizeof(hip_bfloat16));
    const size_t hidden_int4 = hidden_bytes / sizeof(int4);

    // Message package: hidden data, FP8 scales, index at source
    // NOTES: currently we have 3 reserved int fields for future use
    using vec_t = std::conditional_t<kUseFP8, int2, int4>;
    const size_t num_bytes_per_msg_v = sizeof(int4) + (kUseFP8 ? (kHidden + num_scales * sizeof(float)) : (kHidden * sizeof(hip_bfloat16)));
    const size_t num_bytes_per_msg = kEager != EAGER_OFF ? EXTEND_FOR_TAG_AND_ALIGN(num_bytes_per_msg_v, AR_MSG_ALIGNMENT) : num_bytes_per_msg_v;
    
    // WARNING!!! Two lines below must be consistent with msg def in config.hpp
    constexpr size_t dispatch_msg_max = sizeof(int4) + std::max(kHidden * sizeof(hip_bfloat16), kHidden + num_scales * sizeof(float));
    constexpr size_t combine_msg_max = num_scales * sizeof(nv_bfloat162) + kHidden * sizeof(hip_bfloat16);
    constexpr size_t msg_distance = kEager != EAGER_OFF ? EXTEND_FOR_TAG_AND_ALIGN(std::max(dispatch_msg_max, combine_msg_max) + sizeof(int), AR_MSG_LONG_ALIGNMENT) : num_bytes_per_msg_v;

    const size_t num_int4_per_msg = num_bytes_per_msg / sizeof(int4);
    EP_DEVICE_ASSERT(num_bytes_per_msg % sizeof(int4) == 0);
    
    //if (sm_id == 0 && thread_id == 0) printf("[rank %d]: kEager = %d\n", rank, kEager);
    // Expert counts
    constexpr int kNumMaxWarpGroups = 32;
    __shared__ int shared_num_tokens_sent_per_expert[kNumMaxWarpGroups];

    // Sending phase
    if ((phases & LOW_LATENCY_SEND_PHASE) == 0)
        goto LOW_LATENCY_DISPATCH_RECV;

    // There are 2 kinds of warps in this part:
    // 1. The first-kind warps for FP8 cast and sending top-k tokens
    // 2. The last warp for reading `topk_idx` and count for per-expert information
    if (warp_id < num_warps - 1) {
        constexpr int kNumElemsPerRead = sizeof(int4) / sizeof(hip_bfloat16);
        EP_STATIC_ASSERT(kHidden % (32 * kNumElemsPerRead) == 0, "Invalid hidden");
        EP_STATIC_ASSERT(kNumElemsPerRead * 32 % kNumPerChannels == 0, "Invalid vectorization");
        const auto num_threads = (num_warps - 1) * 32;
        const size_t hidden_bf16_int4 = kHidden / kNumElemsPerRead;

        for (int token_idx = sm_id; token_idx < num_tokens; token_idx += num_sms) {
            const auto x_int4 = static_cast<const int4*>(x) + token_idx * hidden_bf16_int4;
            const auto rdma_x_src_idx = reinterpret_cast<int*>(static_cast<uint8_t*>(rdma_x) + token_idx * num_bytes_per_msg);
            const auto rdma_x_vec = reinterpret_cast<vec_t*>(reinterpret_cast<uint8_t*>(rdma_x_src_idx) + sizeof(int4));
            const auto rdma_x_scales = reinterpret_cast<float*>(reinterpret_cast<uint8_t*>(rdma_x_vec) + hidden_bytes);

            // Overlap top-k index read and source token index writes
            auto dst_expert_idx = warp_id < num_topk ? static_cast<int>(__ldg(topk_idx + token_idx * num_topk + warp_id)) : -1;

#define DISPATCH_ST(PTR, VALUE) {\
    if (kEager != EAGER_OFF) {\
        N_ST_SHIFTED(PTR, VALUE, rdma_x_src_idx);\
    } else {\
        NORMAL_ST(PTR, VALUE);\
    }\
}

            if (thread_id == 0) {
                DISPATCH_ST(rdma_x_src_idx, token_idx);
            }

            // FP8 cast
            EP_STATIC_ASSERT(hidden_bf16_int4 % 32 == 0, "Must use the full warp to reduce");
            #pragma unroll
            for (int i = thread_id; i < hidden_bf16_int4; i += num_threads) {
                // Read
                auto int4_value = __ldg(x_int4 + i);

                if constexpr (kUseFP8) {
                    // Calculate local amax
                    auto bf16_values = reinterpret_cast<hip_bfloat16*>(&int4_value);
                    float fp32_values[kNumElemsPerRead];
                    float amax = kFP8Margin, scale, scale_inv;
                    #pragma unroll
                    for (int j = 0; j < kNumElemsPerRead; ++ j) {
                        fp32_values[j] = static_cast<float>(bf16_values[j]);
                        amax = fmaxf(amax, fabsf(fp32_values[j]));
                    }

                    // Reduce amax and scale
                    EP_STATIC_ASSERT(kNumElemsPerRead * 32 / kNumPerChannels == 2, "Invalid vectorization");
                    amax = warp_reduce_max<16>(amax);
                    calculate_fp8_scales(amax, scale, scale_inv, round_scale);
                    if (lane_id == 0 or lane_id == 16) {
                        DISPATCH_ST(&rdma_x_scales[i * kNumElemsPerRead / 128], scale_inv);
                    }

                    // Cast into send buffer
                    vec_t int2_value;
                    auto fp8x2_values = reinterpret_cast<__hip_fp8x2_storage_t*>(&int2_value);
                    #pragma unroll
                    for (int j = 0; j < kNumElemsPerRead; j += 2) {
                        float2 fp32x2 = {fp32_values[j] * scale, fp32_values[j + 1] * scale};
                        fp8x2_values[j / 2] = __hip_cvt_float2_to_fp8x2(fp32x2, __HIP_SATFINITE, __HIP_E4M3_FNUZ);
                    }
                    DISPATCH_ST(&rdma_x_vec[i], int2_value);
                } else {
                    // Reinterpret-cast is for C++14 compatibility
                    DISPATCH_ST(reinterpret_cast<int4*>(&rdma_x_vec[i]), int4_value);
                }
            }
            if (kEager != EAGER_OFF) {
                const auto warp_runs = hidden_bf16_int4 / 32;
                const auto idle_warp_id = (warp_id + (num_warps - 1) - (warp_runs % (num_warps - 1))) % (num_warps - 1);
                if (idle_warp_id == 0) {
                    PARALLEL_SET_TAG(rdma_x_src_idx, num_bytes_per_msg_v, dispatch_round_n, lane_id, 32, NORMAL_ST);
                }
            }
            asm volatile("bar.sync 1, %0;" :: "r"(num_threads));

            // Issue IBGDA sends
            if (dst_expert_idx >= 0) {
                int slot_idx = lane_id == 0 ? atomicAdd(atomic_counter_per_expert + dst_expert_idx, 1) : 0;
                slot_idx = __shfl_sync(0xffffffff, slot_idx, 0);
                const auto dst_rank = dst_expert_idx / num_local_experts;
                const auto dst_expert_local_idx = dst_expert_idx % num_local_experts;
                const auto src_ptr = reinterpret_cast<uint64_t>(rdma_x_src_idx);
                const auto dst_ptr = reinterpret_cast<uint64_t>(rdma_recv_x) +
                                     dst_expert_local_idx * num_ranks * num_max_dispatch_tokens_per_rank * msg_distance +
                                     rank * num_max_dispatch_tokens_per_rank * msg_distance +
                                     slot_idx * msg_distance;
                const auto dst_p2p_ptr = nvshmemi_get_p2p_ptr(dst_ptr, rank, dst_rank);
                if (dst_p2p_ptr == 0) {
                    if constexpr (kEager == EAGER_FULL) {
                        slot_idx = -1;
                    }
                    nvshmemi_ibgda_put_nbi_warp(dst_ptr, src_ptr, num_bytes_per_msg, dst_rank, dst_expert_local_idx, lane_id, slot_idx);
                } else {
                    // NOTES: only 2 load iterations for 7K hidden with 8 unrolls
                    const auto* src_int4_ptr = reinterpret_cast<const int4*>(src_ptr);
                    const auto* dst_int4_ptr = reinterpret_cast<int4*>(dst_p2p_ptr);
                    UNROLLED_WARP_COPY(8, lane_id, num_int4_per_msg, dst_int4_ptr, src_int4_ptr, ld_nc_global, st_na_global);
                    WARP_SET_TAIL_TAG(dst_p2p_ptr, num_bytes_per_msg, dispatch_round_n);
                }

                // Increase counter after finishing
                if (kEager != EAGER_FULL) {
                    __syncwarp();
                    lane_id == 0 ? atomic_add_release_global(atomic_finish_counter_per_expert + dst_expert_idx, 1) : 0;
                }
            }
        }
    } else if (warp_id == num_warps - 1) {
        EP_DEVICE_ASSERT(num_sms > 1);
        if (sm_id == 0) {
            // The first SM is also responsible for checking QPs
            EP_DEVICE_ASSERT(ibgda_get_state()->num_rc_per_pe >= num_local_experts);

            // The first SM is also responsible for cleaning the next buffer
            #pragma unroll
            for (int i = lane_id; i < num_next_clean_int; i += 32)
                next_clean[i] = 0;

            // Notify before executing `int_p`
            __syncwarp();
            if (kEager != EAGER_FULL) {
                #pragma unroll
                for (int i = lane_id; i < num_experts; i += 32)
                    atomic_add_release_global(atomic_finish_counter_per_expert + i, FINISHED_SUM_TAG);
            }
        }

        // This SM should be responsible for some destination experts, read `topk_idx` for them
        int expert_count[kNumMaxWarpGroups] = {0};
        const auto expert_begin_idx = sm_id * num_warp_groups;
        const auto expert_end_idx = min(expert_begin_idx + num_warp_groups, num_experts);

        // Per lane count
        #pragma unroll 8
        for (int i = lane_id; i < num_tokens * num_topk; i += 32) {
            auto idx = static_cast<int>(__ldg(topk_idx + i));
            if (idx >= expert_begin_idx and idx < expert_end_idx)
                expert_count[idx - expert_begin_idx] ++;
        }

        // Warp reduce
        #pragma unroll
        for (int i = expert_begin_idx; i < expert_end_idx; ++ i) {
            auto sum = warp_reduce_sum(expert_count[i - expert_begin_idx]);
            if (lane_id == 0) {
                if (kEager == EAGER_FULL) {
                    const auto dst_rank = i / num_local_experts;
                    const auto dst_expert_local_idx = i % num_local_experts;
                    const auto num_tokens_sent = -sum - 1;
                    auto dst_ptr = reinterpret_cast<uint64_t>(rdma_recv_count + dst_expert_local_idx * num_ranks + rank);
                    auto dst_p2p_ptr = nvshmemi_get_p2p_ptr(dst_ptr, rank, dst_rank);
                    //if (sum != 0) printf("[rank %d]: %d tokens to expert %d\n", rank, sum, i);
                    if (dst_p2p_ptr == 0) {
                        nvshmemi_ibgda_rma_p(reinterpret_cast<int*>(dst_ptr), num_tokens_sent, dst_rank, dst_expert_local_idx);
                    } else {
                        st_release_sys_global(reinterpret_cast<int*>(dst_p2p_ptr), num_tokens_sent);
                    }
                } else {
                    shared_num_tokens_sent_per_expert[i - expert_begin_idx] = sum;
                    atomic_add_release_global(atomic_finish_counter_per_expert + i, FINISHED_SUM_TAG - sum);
                }
            }
        }
    }

    if (kEager != EAGER_FULL) {
        __syncthreads();

        // Issue count sends
        if (responsible_expert_idx < num_experts and sub_warp_id == 0 and lane_id == 0) {
            const auto dst_rank = responsible_expert_idx / num_local_experts;
            const auto dst_expert_local_idx = responsible_expert_idx % num_local_experts;
            const auto num_tokens_sent = shared_num_tokens_sent_per_expert[responsible_expert_idx - sm_id * num_warp_groups];

            // Wait local sends issued and send expert counts
            while (ld_acquire_global(atomic_finish_counter_per_expert + responsible_expert_idx) != FINISHED_SUM_TAG * 2);
            auto dst_ptr = reinterpret_cast<uint64_t>(rdma_recv_count + dst_expert_local_idx * num_ranks + rank);
            auto dst_p2p_ptr = nvshmemi_get_p2p_ptr(dst_ptr, rank, dst_rank);
            if (dst_p2p_ptr == 0) {
                if (kEager <= EAGER_OFF) {
                    nvshmemi_ibgda_amo_nonfetch_add(reinterpret_cast<int*>(dst_ptr), -num_tokens_sent - 1, dst_rank, dst_expert_local_idx);
                } else {
                    nvshmemi_ibgda_rma_p(reinterpret_cast<int*>(dst_ptr), -num_tokens_sent - 1, dst_rank, dst_expert_local_idx);
                }
            } else {
                st_release_sys_global(reinterpret_cast<int*>(dst_p2p_ptr), -num_tokens_sent - 1);
            }

            // Clean workspace for next use
            atomic_counter_per_expert[responsible_expert_idx] = 0;
            atomic_finish_counter_per_expert[responsible_expert_idx] = 0;

            // Clean `packed_recv_count`
            if (dst_rank == 0)
                packed_recv_count[dst_expert_local_idx] = 0;
        }
        __syncwarp();
    }

    // Receiving phase
    LOW_LATENCY_DISPATCH_RECV:
    if ((phases & LOW_LATENCY_RECV_PHASE) == 0)
        return;

    // For send-and-recv kernels, we need a grid sync for making `packed_recv_count` visible
    if ((kEager != EAGER_FULL) && (phases & LOW_LATENCY_SEND_PHASE))
        cg::this_grid().sync();

    // Receiving and packing
    if (responsible_expert_idx < num_experts) {
        const auto src_rank = responsible_expert_idx / num_local_experts;
        const auto local_expert_idx = responsible_expert_idx % num_local_experts;
        const auto global_expert_idx = rank * num_local_experts + local_expert_idx;
        const auto rdma_recv_x_uint8 = static_cast<uint8_t*>(rdma_recv_x) +
                local_expert_idx * num_ranks * num_max_dispatch_tokens_per_rank * msg_distance +
                src_rank * num_max_dispatch_tokens_per_rank * msg_distance;
        const auto recv_x_int4 = static_cast<int4*>(packed_recv_x) +
                local_expert_idx * num_ranks * num_max_dispatch_tokens_per_rank * hidden_int4;
        const auto recv_src_info = packed_recv_src_info + local_expert_idx * num_ranks * num_max_dispatch_tokens_per_rank;
        const auto recv_range = packed_recv_layout_range + local_expert_idx * num_ranks;
        const auto token_src_bitmap = reinterpret_cast<int*>(packed_recv_layout_range) + local_expert_idx * num_ranks * num_max_dispatch_tokens_per_rank + src_rank * num_max_dispatch_tokens_per_rank; // used in EAGER_FULL
        const auto per_rank_recv_cnt_ptr = per_rank_recv_count + local_expert_idx * num_ranks; // used in EAGER_FULL
        const auto ld_intra_node = (src_rank >> 3) == (rank >> 3); // TODO: identify intra node more elegantly
        const auto num_aligned_scales = align<int>(num_scales, sizeof(float) / sizeof(scale_t));
        const auto recv_x_scales = static_cast<scale_t*>(packed_recv_x_scales) + local_expert_idx * num_ranks * num_max_dispatch_tokens_per_rank * num_aligned_scales;

        // Shared between sub-warps in warp groups
        __shared__ int shared_num_recv_tokens[kNumMaxWarpGroups], shared_recv_token_begin_idx[kNumMaxWarpGroups];

        // Wait tokens to arrive
        // NOTES: using sub-warp 1 to overlap with sub-warp 0
        int num_recv_tokens = 0, recv_token_begin_idx;
        auto rcv_cnt_ptr = rdma_recv_count + local_expert_idx * num_ranks + src_rank;
        if constexpr (kEager != EAGER_FULL) {
            EP_DEVICE_ASSERT(num_warps_per_group > 1 and num_warp_groups < 15);
            if constexpr (kEager != EAGER_FULL) {
                if (sub_warp_id == 1 and lane_id == 0) {
                    auto start_time = clock64();
                    while ((num_recv_tokens = ld_acquire_sys_global(rdma_recv_count + local_expert_idx * num_ranks + src_rank)) == 0);
                    auto wait_recv_cost = clock64() - start_time;
                    num_recv_tokens = -num_recv_tokens - 1;
                    recv_token_begin_idx = atomicAdd(packed_recv_count + local_expert_idx, num_recv_tokens);
                    shared_num_recv_tokens[warp_group_id] = num_recv_tokens;
                    shared_recv_token_begin_idx[warp_group_id] = recv_token_begin_idx;
                    recv_range[src_rank] = pack2<int, int64_t>(num_recv_tokens, recv_token_begin_idx);

                    // Add stats for diagnosis
                    if (cumulative_local_expert_recv_stats != nullptr)
                        atomicAdd(cumulative_local_expert_recv_stats + local_expert_idx, num_recv_tokens);
                    if (dispatch_wait_recv_cost_stats != nullptr)
                        atomicAdd(reinterpret_cast<unsigned long long*>(dispatch_wait_recv_cost_stats + src_rank), wait_recv_cost);
                }
                asm volatile("bar.sync %0, %1;" :: "r"(warp_group_id + 2), "r"(num_warps_per_group * 32));
                num_recv_tokens = shared_num_recv_tokens[warp_group_id];
                recv_token_begin_idx = shared_recv_token_begin_idx[warp_group_id];
            }
            // Copy tokens
            EP_DEVICE_ASSERT(num_scales <= 64);
            int this_token_index_in_recv = 0;
            for (int i = sub_warp_id; (kEager != EAGER_FULL && i < num_recv_tokens) || (kEager == EAGER_FULL && (num_recv_tokens == 0 || i < -num_recv_tokens-1) && i < num_max_dispatch_tokens_per_rank); i += num_warps_per_group) {
                // Copy source info
                const auto src_src_idx = reinterpret_cast<int*>(rdma_recv_x_uint8 + i * msg_distance);

                if (kEager == EAGER_FULL) {
                    WAIT_2BIT(src_src_idx, num_bytes_per_msg_v, num_bytes_per_msg, lane_id, 32, dispatch_round_n, rcv_cnt_ptr, num_recv_tokens, i, ld_intra_node);
                    __syncwarp();
                    num_recv_tokens = warp_reduce_min(num_recv_tokens);
                    
                    if (num_recv_tokens != 0 && i >= (-num_recv_tokens-1)) {
                        //if (sub_warp_id == 0 && lane_id == 0 && (-num_recv_tokens-1) != 0) printf("[rank %d]: dispatch recv token expert %d get %d tokens from rank %d, done\n", rank, global_expert_idx, (-num_recv_tokens-1), src_rank);
                        break;
                    }
                    if (lane_id == 0) {
                        this_token_index_in_recv = atomic_add_release_global(packed_recv_count + local_expert_idx, 1);
                        atomicAdd(per_rank_recv_cnt_ptr + src_rank, 1);
                        //printf("[rank %d]: dispatch recv token expert %d slot %d get rank %d token %d\n", rank, global_expert_idx, this_token_index_in_recv, src_rank, LD_SHIFTED(ld_nc_global, src_src_idx, src_src_idx));
                        token_src_bitmap[i] = this_token_index_in_recv;
                    }
                    __syncwarp();
                    this_token_index_in_recv = __shfl_sync(0xffffffff, this_token_index_in_recv, 0);
                } else if constexpr (kEager != EAGER_OFF) {
                    WAIT_BIT(src_src_idx, num_bytes_per_msg_v, num_bytes_per_msg, lane_id, 32, dispatch_round_n, ld_intra_node);
                    __syncwarp();
                }
                if constexpr (kEager != EAGER_FULL) {
                    this_token_index_in_recv = recv_token_begin_idx + i;
                }

                if (lane_id == 0)
                    recv_src_info[this_token_index_in_recv] = DISPATCH_LD(ld_nc_global, src_src_idx);
                __syncwarp();

                // Copy data
                // NOTES: only 2 load iterations for 7K hidden with 7 unrolls
                const auto src_data = reinterpret_cast<int4*>(reinterpret_cast<uint8_t*>(src_src_idx) + sizeof(int4));
                const auto dst_data = recv_x_int4 + this_token_index_in_recv * hidden_int4;
                if (kEager != EAGER_OFF) {
                    UNROLLED_WARP_COPY_SRC_AUTO_SHIFT(7, lane_id, hidden_int4, dst_data, src_data, src_src_idx, ld_nc_global, st_na_global);  
                } else {
                    UNROLLED_WARP_COPY(7, lane_id, hidden_int4, dst_data, src_data, ld_nc_global, st_na_global);
                }

                // Copy scales
                if constexpr (kUseFP8) {
                    // Equivalent CuTe layout:
                    //   (num_tokens, (num_packed, num_elems_per_pack)):(num_elems_per_pack, (num_tokens * num_elems_per_pack, 1))
                    const auto src_scales = reinterpret_cast<float*>(reinterpret_cast<uint8_t*>(src_data) + hidden_bytes);
                    const auto num_elems_per_pack = static_cast<int>(sizeof(packed_t) / sizeof(scale_t));
                    const auto token_idx = this_token_index_in_recv;
                    const auto token_stride = num_elems_per_pack;
                    const auto pack_stride = num_ranks * num_max_dispatch_tokens_per_rank * num_elems_per_pack;
                    if (lane_id < num_scales) {
                        const auto pack_idx = lane_id / num_elems_per_pack;
                        const auto elem_idx = lane_id % num_elems_per_pack;
                        auto scale = extract_required_scale_format<kUseUE8M0>(DISPATCH_LD(ld_nc_global, src_scales + lane_id));
                        recv_x_scales[token_idx * token_stride + pack_idx * pack_stride + elem_idx] = scale;
                    }
                    if (lane_id + 32 < num_scales) {
                        const auto pack_idx = (lane_id + 32) / num_elems_per_pack;
                        const auto elem_idx = (lane_id + 32) % num_elems_per_pack;
                        auto scale = extract_required_scale_format<kUseUE8M0>(DISPATCH_LD(ld_nc_global, src_scales + lane_id + 32));
                        recv_x_scales[token_idx * token_stride + pack_idx * pack_stride + elem_idx] = scale;
                    }
                }
            }
            //if constexpr (kEager == EAGER_FULL) if ((-num_recv_tokens-1) != 0 && sub_warp_id < num_max_dispatch_tokens_per_rank && lane_id == 0) printf("[rank %d]: dispatch recv token expert %d get %d tokens from rank %d, subwarp %d done\n", rank, global_expert_idx, (-num_recv_tokens-1), src_rank, sub_warp_id);
        } else {
            // EAGER_FULL use soft warp scheduling to find ready tokens as soon as possible
            constexpr int hard_warps = 4; // hardware warp units
            const int working_queues = num_warps;
            __shared__ int current_token_index[32]; // working token index
            __shared__ int token_cnt_rcv[32]; // token count
            int still_working_queues = 0;
            int token_index = 0;
            if (warp_id < hard_warps) {
                for (int i = thread_id; i < working_queues; i += hard_warps * 32) {
                    current_token_index[i] = i % num_warps_per_group;
                    if (i < num_warp_groups) {
                        token_cnt_rcv[i] = 0;
                    }
                }
                asm volatile("bar.sync 1, %0;" :: "r"(hard_warps * 32));
                still_working_queues = (working_queues - warp_id - 1) / hard_warps + 1;
                while (still_working_queues > 0) {
                    for (int i = warp_id; i < working_queues; i += hard_warps) {
                        token_index = lane_id == 0 ? current_token_index[i] : 0; // pick a virtual warp
                        token_index = __shfl_sync(0xffffffff, token_index, 0);
                        auto rsp_exp_idx = (sm_id * num_warp_groups + (i / num_warps_per_group));
                        if (token_index < 0) { // virtual warp is already done
                            continue;
                        }
                        while (true) {
                            // try recv token
                            int num_recv_tokens = lane_id == 0 ? token_cnt_rcv[i / num_warps_per_group] : 0; // rank -> exp token cnt (-x-1 format)
                            num_recv_tokens = __shfl_sync(0xffffffff, num_recv_tokens, 0);
                            if (token_index >= num_max_dispatch_tokens_per_rank || (num_recv_tokens != 0 && token_index >= -num_recv_tokens-1)) { // out of range, so this virtual warp is done
                                token_index = -1; // this virtual warp is done
                                still_working_queues -= 1; // one less queue to work on
                                break;
                            }
                            int num_recv_tokens_ref = num_recv_tokens;
                            bool got_token = eager_dispatch_recv_token<kUseFP8, kUseUE8M0, kHidden>(rdma_recv_x, rdma_recv_count, rsp_exp_idx, num_local_experts, num_experts, rank, num_ranks, num_max_dispatch_tokens_per_rank, msg_distance, num_bytes_per_msg, num_bytes_per_msg_v, packed_recv_x, packed_recv_x_scales, packed_recv_src_info, packed_recv_layout_range, packed_recv_count, per_rank_recv_count, num_recv_tokens_ref, token_index, dispatch_round_n);
                            if (lane_id == 0 && num_recv_tokens == 0 && num_recv_tokens_ref != 0) {
                                // new count info
                                token_cnt_rcv[i / num_warps_per_group] = num_recv_tokens_ref;
                            }
                            __syncwarp();
                            if (!got_token) break; // no new token in this virtual warp, try next virtual warp
                            token_index += num_warps_per_group; // goto next token in this virtual warp
                        }
                        if (lane_id == 0) {
                            current_token_index[i] = token_index;
                        }
                        __syncwarp();
                    }
                }
            }
        }
    }
}

void dispatch(void* packed_recv_x, void* packed_recv_x_scales,
              int* packed_recv_src_info, int64_t* packed_recv_layout_range,
              int* packed_recv_count, int* per_rank_recv_count,
              int* cumulative_local_expert_recv_stats,
              int64_t* dispatch_wait_recv_cost_stats,
              void* rdma_recv_x, int* rdma_recv_count, void* rdma_x,
              const void* x, const int64_t* topk_idx,
              int* next_clean, int num_next_clean_int,
              int num_tokens, int hidden, int num_max_dispatch_tokens_per_rank,
              int num_topk, int num_experts, int rank, int num_ranks,
              bool use_fp8, bool round_scale, bool use_ue8m0,
              void* workspace, int num_device_sms,
              hipStream_t stream, int phases,
              int dispatch_round_n, const int eager_opt) {
    constexpr int kNumMaxTopK = 9;
    const int num_warp_groups = ceil_div(num_experts, num_device_sms);
    const int num_warps_per_group = 32 / num_warp_groups;
    EP_HOST_ASSERT(num_warp_groups > 0 and num_warps_per_group > 0);
    EP_HOST_ASSERT(kNumMaxTopK + 1 <= num_warp_groups * num_warps_per_group);

    const auto num_warps = num_warp_groups * num_warps_per_group;
    const auto num_sms = ceil_div(num_experts, num_warp_groups);
    EP_HOST_ASSERT(num_topk <= kNumMaxTopK);

    // Workspace checks
    auto atomic_counter_per_expert = static_cast<int*>(workspace);
    auto atomic_finish_counter_per_expert = atomic_counter_per_expert + num_experts;
    EP_HOST_ASSERT(num_experts * sizeof(int) * 2 <= NUM_WORKSPACE_BYTES);
    if ((eager_opt == EAGER_FULL) && (phases & LOW_LATENCY_SEND_PHASE)) {
        CUDA_CHECK(hipMemsetAsync(workspace, 0, num_experts * sizeof(int), stream)); // only set atomic_counter_per_expert
        CUDA_CHECK(hipMemsetAsync(packed_recv_count, 0, sizeof(int) * num_experts / num_ranks, stream));
        CUDA_CHECK(hipMemsetAsync(per_rank_recv_count, 0, sizeof(int) * num_experts, stream));
    }
    // FP8 checks
    if (use_ue8m0)
        EP_HOST_ASSERT(round_scale and "UE8M0 SF requires `round_scale=True`");

#define DISPATCH_LAUNCH_CASE(eager, hidden) { \
auto dispatch_func = dispatch<false, false, hidden, eager>; \
if (use_fp8 and not use_ue8m0) \
    dispatch_func = dispatch<true, false, hidden, eager>; \
if (use_fp8 and use_ue8m0) \
    dispatch_func = dispatch<true, true, hidden, eager>; \
LAUNCH_KERNEL(&cfg, dispatch_func, \
              packed_recv_x, packed_recv_x_scales, \
              packed_recv_src_info, packed_recv_layout_range, \
              packed_recv_count, per_rank_recv_count, \
              cumulative_local_expert_recv_stats, \
              dispatch_wait_recv_cost_stats, \
              rdma_recv_x, rdma_recv_count, rdma_x, \
              x, topk_idx, \
              atomic_counter_per_expert, atomic_finish_counter_per_expert, \
              next_clean, num_next_clean_int, \
              num_tokens, num_max_dispatch_tokens_per_rank, \
              num_topk, num_experts, rank, num_ranks, \
              num_warp_groups, num_warps_per_group, \
              round_scale, phases, dispatch_round_n); } break

    SETUP_LAUNCH_CONFIG(num_sms, num_warps * 32, stream);

    #define EAGER_DISPATCH_LAUNCH_CASE(hidden) SWITCH_EAGER(DISPATCH_LAUNCH_CASE, hidden)
    
    SWITCH_HIDDEN(EAGER_DISPATCH_LAUNCH_CASE);
#undef DISPATCH_LAUNCH_CASE
}

template <int kNumSendUnrolls>
__forceinline__ __device__ int logfmt_encode(void* buffer, nv_bfloat162 *shared_amaxmin, const int& lane_id) {
    constexpr int kNumElemsPerInt4 = sizeof(int4) / sizeof(hip_bfloat16);
    constexpr float kLogThreshold = 0;
    constexpr float kMinClip = 32; // `== log_2(2 ^ (2 ^ 5))`
    constexpr int kNumBits = 10;
    constexpr int kNumValues = 1 << (kNumBits - 1);

    int4 int4_values[kNumSendUnrolls];
    const auto& uint32_values = reinterpret_cast<uint32_t*>(int4_values);
    const auto& bf162_values = reinterpret_cast<nv_bfloat162*>(int4_values);

    // Calculate lane offset
    const auto& ld_buffer = reinterpret_cast<uint32_t*>(static_cast<uint8_t*>(buffer) + lane_id * (kNumSendUnrolls * sizeof(int4)));
    const auto& st_buffer = reinterpret_cast<uint32_t*>(static_cast<uint8_t*>(buffer) + lane_id * (kNumSendUnrolls * sizeof(int4) * 10 / 16));

    // Local log amax
    auto bf162_amax = __hip_bfloat162(CUDART_ZERO_BF16, CUDART_ZERO_BF16);
    auto bf162_amin = __hip_bfloat162(CUDART_INF_BF16, CUDART_INF_BF16);
    uint32_t local_signs = 0;
    #pragma unroll
    for (int k = 0; k < kNumSendUnrolls * kNumElemsPerInt4 / 2; ++ k) {
        // TODO: eliminate bank conflicts
        uint32_values[k] = ld_buffer[k];
        local_signs |= ((uint32_values[k] >> 15) & 1) << (k * 2);
        local_signs |= ((uint32_values[k] >> 31) & 1) << (k * 2 + 1);
        uint32_values[k] &= 0x7fff7fff;

        bf162_amax = __hmax2(bf162_amax, bf162_values[k]);
        bf162_amin = __hmin2(bf162_amin, bf162_values[k]);
    }

    // Reduce per 128 channels
    // TODO: figure out how hardware do 2-byte min/max
    auto amax = std::max(static_cast<float>(bf162_amax.x), static_cast<float>(bf162_amax.y));
    auto amin = std::min(static_cast<float>(bf162_amin.x), static_cast<float>(bf162_amin.y));
    constexpr static int kNumLanesToReduce = 128 * sizeof(hip_bfloat16) / (kNumSendUnrolls * sizeof(int4));
    amax = warp_reduce_max<kNumLanesToReduce>(amax);
    amin = warp_reduce_min<kNumLanesToReduce>(amin);

    // Write min/max into the shared memory
    if (shared_amaxmin != nullptr)
        *shared_amaxmin = __hip_bfloat162(amax, amin);
    __syncwarp();

    // Calculate log amin/amax float
    const auto& log_amax = log2f_approx(amax);
    const auto& log_amin = fmaxf(log2f_approx(amin), log_amax - kMinClip);
    const bool& enable_cast = warp_reduce_and<kNumLanesToReduce, true>(log_amax < kLogThreshold and log_amin < log_amax);

    // Case into LogFMT-10 if satisfied
    if (enable_cast) {
        const auto step = (log_amax - log_amin) / static_cast<float>(kNumValues - 2);
        const auto step_inv = 1.0f / step;
        const auto rounding = 2.0f - log2f_approx((1.0f + exp2f_approx(step)) * 0.5f) * step_inv;
        const auto fused_rounding = rounding - log_amin * step_inv;

        // Pack every 256 bits into 160 bits
        EP_STATIC_ASSERT(kNumSendUnrolls == 2 or kNumSendUnrolls == 4, "kNumSendUnrolls == 2 or 4 only");
        uint32_t encoded[kNumElemsPerInt4 * 2];
        #pragma unroll 1
        for (int i = 0; i < kNumSendUnrolls / 2; ++ i) {
            #pragma unroll
            for (int k = 0; k < kNumElemsPerInt4; ++ k) {
                const auto& [x, y] = __bfloat1622float2(bf162_values[i * kNumElemsPerInt4 + k]);
                encoded[k * 2 + 0] = __float2uint_rd(fmaxf(log2f_approx(x) * step_inv + fused_rounding, 0));
                encoded[k * 2 + 1] = __float2uint_rd(fmaxf(log2f_approx(y) * step_inv + fused_rounding, 0));
            }
            st_buffer[i * 5 + 0] = (encoded[ 0] >> 0) | (encoded[ 1] << 9) | (encoded[ 2] << 18) | (encoded[ 3] << 27);
            st_buffer[i * 5 + 1] = (encoded[ 3] >> 5) | (encoded[ 4] << 4) | (encoded[ 5] << 13) | (encoded[ 6] << 22) | (encoded[7]  << 31);
            st_buffer[i * 5 + 2] = (encoded[ 7] >> 1) | (encoded[ 8] << 8) | (encoded[ 9] << 17) | (encoded[10] << 26);
            st_buffer[i * 5 + 3] = (encoded[10] >> 6) | (encoded[11] << 3) | (encoded[12] << 12) | (encoded[13] << 21) | (encoded[14] << 30);
            st_buffer[i * 5 + 4] = (encoded[14] >> 2) | (encoded[15] << 7) | ((i == 0) ? (local_signs << 16) : (local_signs & 0xffff0000u));
        }
        tma_store_fence();
        __syncwarp();
    }

    // Return TMA copy bytes
    return enable_cast ? (32 * (kNumSendUnrolls * sizeof(int4) * 8 * 10 / 16 / 8)):
                         (32 * (kNumSendUnrolls * sizeof(int4)));
}

template <int kNumLanes, int kNumSendUnrolls, int kNumRecvUnrolls>
__forceinline__ __device__ void logfmt_check_amaxmin(uint8_t* meta_buffer, float2* shared_log_amax,
                                                     float2* shared_log_amin, int* shared_cast_info,
                                                     const int lane_id) {
    constexpr float kLogThreshold = 0;
    constexpr float kMinClip = 32; // `== log_2(2 ^ (2 ^ 5))`

    bool enable_cast = true;
    if (lane_id < kNumLanes) {
        // Calculate log amin/amax float
        auto amaxmin2 = reinterpret_cast<uint64_t*>(meta_buffer)[lane_id];
        const auto& bf162_amaxmin = reinterpret_cast<__hip_bfloat162*>(&amaxmin2);
        float log_amax[2], log_amin[2];
        #pragma unroll
        for (int i = 0; i < 2; ++ i) { 
            auto amax = static_cast<float>(bf162_amaxmin[i].x);
            auto amin = static_cast<float>(bf162_amaxmin[i].y);
            log_amax[i] = log2f_approx(amax);
            log_amin[i] = amin == 0 ? log_amax[i] - kMinClip : fmaxf(log2f_approx(amin), log_amax[i] - kMinClip);
            enable_cast = enable_cast and log_amax[i] < kLogThreshold and log_amin[i] < log_amax[i];
        }
        shared_log_amax[lane_id] = make_float2(log_amax[0], log_amax[1]);
        shared_log_amin[lane_id] = make_float2(log_amin[0], log_amin[1]);
    }

    const auto& casted = warp_reduce_and<kNumSendUnrolls>(enable_cast) ? 1u << (lane_id / kNumRecvUnrolls): 0u;
    const auto& num_casted_prefix = __popc(warp_reduce_or<kNumRecvUnrolls, true>(casted) & ((1u << (lane_id / kNumRecvUnrolls)) - 1));

    if (lane_id < kNumLanes and lane_id % kNumRecvUnrolls == 0)
        shared_cast_info[lane_id / kNumRecvUnrolls] = (num_casted_prefix << 1) | (casted ? 1u : 0u);
    __syncwarp();
}

template <int kNumRecvUnrolls>
__forceinline__ __device__ void decode_and_accumulate(uint32_t* ld_buffer, float* accum,
                                                      const float& log_amax, const float& log_amin,
                                                      const bool& enable_cast, const float& weight) {
    if (enable_cast) {
        constexpr int kNumBits = 10;
        constexpr int kNumValues = 1 << (kNumBits - 1);

        const auto& step = (log_amax - log_amin) / static_cast<float>(kNumValues - 2);
        auto decode = [=](const uint32_t &encoded, const uint32_t &sign) {
            const auto decoded = encoded == 0 ? .0f : exp2f_approx((encoded - 1) * step + log_amin);
            return sign ? -decoded : decoded;
        };

        EP_STATIC_ASSERT(kNumRecvUnrolls == 2 or kNumRecvUnrolls == 4, "kNumRecvUnrolls == 2 or 4 only");
        #pragma unroll
        for (int i = 0; i < kNumRecvUnrolls / 2; ++ i) {
            uint32_t concat[6];
            concat[0] = ld_buffer[i * 5];
            #pragma unroll
            for (int k = 1; k < 5; ++ k)
                concat[k] = (ld_buffer[i * 5 + k - 1] >> (32 - k * 5)) | (ld_buffer[i * 5 + k] << (k * 5));
            concat[5] = ld_buffer[i * 5 + 4] >> 7;

            const uint32_t& local_signs = ld_buffer[i * 5 + 4] >> 16;
            #pragma unroll
            for (int k = 0; k < 5; ++ k) {
                accum[i * 16 + k * 3 + 0] += decode((concat[k] >>  0) & 0x1ff, (local_signs >> (k * 3 + 0)) & 1) * weight;
                accum[i * 16 + k * 3 + 1] += decode((concat[k] >>  9) & 0x1ff, (local_signs >> (k * 3 + 1)) & 1) * weight;
                accum[i * 16 + k * 3 + 2] += decode((concat[k] >> 18) & 0x1ff, (local_signs >> (k * 3 + 2)) & 1) * weight;
            }
            accum[i * 16 + 15] += decode(concat[5] & 0x1ff, (local_signs >> 15) & 1) * weight;
        }
    } else {
        #pragma unroll
        for (int k = 0; k < kNumRecvUnrolls * 4; ++ k) {
            auto bf16_pack = *reinterpret_cast<__hip_bfloat162*>(ld_buffer + k);
            accum[k * 2 + 0] += static_cast<float>(bf16_pack.x) * weight;
            accum[k * 2 + 1] += static_cast<float>(bf16_pack.y) * weight;
        }
    }
}

template <bool kUseLogFMT, int kHidden, int kNumMaxTopk, int kNumMaxUnrolls, int kEager>
__global__ __launch_bounds__(1024, 1) void
combine(void* combined_x,
        void* rdma_recv_x, int* rdma_recv_flag, void* rdma_send_x,
        const void* x, const int64_t* topk_idx, const float* topk_weights,
        const int* per_rank_src_count,
        const int* src_info, const int64_t* layout_range,
        int64_t* combine_wait_recv_cost_stats,
        int* next_clean, int num_next_clean_int,
        int* atomic_clean_flag,
        int num_combined_tokens, int hidden, int num_topk,
        int num_max_dispatch_tokens_per_rank,
        int num_experts, int rank, int num_ranks,
        int num_warp_groups, int num_warps_per_group,
        int phases, bool zero_copy, int combine_round_n) {
    const auto sm_id = __shfl_sync(0xffffffff, static_cast<int>(blockIdx.x), 0);
    const auto num_sms = __shfl_sync(0xffffffff, static_cast<int>(gridDim.x), 0);
    const auto thread_id = static_cast<int>(threadIdx.x);
    const auto num_threads = __shfl_sync(0xffffffff, static_cast<int>(blockDim.x), 0);
    const auto warp_id = __shfl_sync(0xffffffff, thread_id / 32, 0), lane_id = get_lane_id();
    const auto num_local_experts = num_experts / num_ranks;
    const auto warp_group_id = warp_id / num_warps_per_group;
    const auto sub_warp_id = warp_id % num_warps_per_group;
    const auto responsible_expert_idx = sm_id * num_warp_groups + warp_group_id;

    extern __shared__ __align__(1024) uint8_t smem_buffer[];

    // Data type staffs
    constexpr int kNumElemsPerInt4 = sizeof(int4) / sizeof(hip_bfloat16);
    constexpr int64_t hidden_bf16_int4 = kHidden / kNumElemsPerInt4;

    // Use different unroll factors for send and recv phases
    constexpr int kNumSendUnrolls = kHidden % (32 * 4 * sizeof(int4) / sizeof(hip_bfloat16)) == 0 ? 4 : 2;
    constexpr int kNumRecvUnrolls = 2;
    constexpr int hidden_bf16_int4_pad = align(static_cast<int>(hidden_bf16_int4), 32 * kNumSendUnrolls);
    EP_STATIC_ASSERT(kHidden % (32 * 2 * sizeof(int4) / sizeof(hip_bfloat16)) == 0, "Invalid hidden");
    EP_STATIC_ASSERT(kNumSendUnrolls <= kNumMaxUnrolls and kNumRecvUnrolls <= kNumMaxUnrolls, "Invalid unrolls");
    EP_STATIC_ASSERT(hidden_bf16_int4 % kNumSendUnrolls == 0, "Invalid hidden");
    EP_STATIC_ASSERT(kNumSendUnrolls >= kNumRecvUnrolls, "Invalid unroll factors");

    // Message package
    EP_STATIC_ASSERT(kHidden % 128 == 0, "Invalid hidden");
    constexpr int kNumDivisions = kHidden / 128;
    constexpr int kNumMetaBytes = kNumDivisions * sizeof(nv_bfloat162);
    constexpr size_t num_bytes_per_slot_v = kHidden * sizeof(hip_bfloat16) + kNumMetaBytes;
    constexpr size_t num_bytes_per_slot = kEager != EAGER_OFF ? EXTEND_FOR_TAG_AND_ALIGN(num_bytes_per_slot_v, AR_MSG_ALIGNMENT) : num_bytes_per_slot_v;
    
    // WARNING!!! Two lines below must be consistent with msg def in config.hpp
    constexpr int kNumPerChannels = 128;
    constexpr int num_scales = kHidden / kNumPerChannels;
    constexpr size_t dispatch_msg_max = sizeof(int4) + std::max(kHidden * sizeof(hip_bfloat16), kHidden + num_scales * sizeof(float));
    constexpr size_t combine_msg_max = num_scales * sizeof(nv_bfloat162) + kHidden * sizeof(hip_bfloat16);
    constexpr size_t msg_distance = kEager != EAGER_OFF ? EXTEND_FOR_TAG_AND_ALIGN(std::max(dispatch_msg_max, combine_msg_max) + sizeof(int), AR_MSG_LONG_ALIGNMENT) : num_bytes_per_slot_v;

    constexpr int kEager_combine = kUseLogFMT ? EAGER_OFF : kEager; // logfmt does not support eager combine
    constexpr int MAX_PAGES_DIV4 = 1;
    constexpr int MAX_PAGES = MAX_PAGES_DIV4 << 2;
    EP_STATIC_ASSERT(MAX_PAGES_DIV4 <= 31, "pages can not be dealt by warp");
    EP_STATIC_ASSERT(kHidden * sizeof(hip_bfloat16) <= MAX_PAGES * PCIE_SEG_LEN, "hidden too large"); // max 4 page, 16K bytes, 8k bf16 hidden
    int4 __tail_tags_int4[MAX_PAGES_DIV4 + 1];
    __tail_tags_int4[MAX_PAGES_DIV4].x = ZTAG(combine_round_n);
    int *__tail_tags = reinterpret_cast<int*>(__tail_tags_int4); // store tag position values

    //constexpr int page_int4 = (PCIE_SEG_LEN - PCIE_TAIL_SZ) / sizeof(int4);
    //constexpr int page_int4_slots = PCIE_SEG_LEN / sizeof(int4);

    EP_STATIC_ASSERT(num_bytes_per_slot % sizeof(int4) == 0, "Invalid vectorization");

    // Sending phase
    if ((phases & LOW_LATENCY_SEND_PHASE) == 0)
        goto LOW_LATENCY_COMBINE_RECV;

    // Clean up next buffer
    if (sm_id == 0 and warp_group_id == 0 and sub_warp_id == 0) {
        #pragma unroll
        for (int i = lane_id; i < num_next_clean_int; i += 32)
            next_clean[i] = 0;

        // Notify before executing `int_p`
        __syncwarp();
        if (kEager_combine != EAGER_FULL) {
            if (lane_id == 0)
                atomic_add_release_global(atomic_clean_flag, num_experts);
        }
    }

    // Issue IBGDA sends
    if (responsible_expert_idx < num_experts) {
        const auto dst_rank = responsible_expert_idx / num_local_experts;
        const auto local_expert_idx = responsible_expert_idx % num_local_experts;
        const auto global_expert_idx = rank * num_local_experts + local_expert_idx;
        const auto layout = kEager != EAGER_FULL ? __ldg(layout_range + local_expert_idx * num_ranks + dst_rank) : 0l;
        const auto layout_bitmap_ptr = reinterpret_cast<const int*>(layout_range) + local_expert_idx * num_ranks * num_max_dispatch_tokens_per_rank + dst_rank * num_max_dispatch_tokens_per_rank; // used in EAGER_FULL
        const int num_tokens_from_this_exp_to_dstrank = kEager != EAGER_FULL ? 0 : __ldg(per_rank_src_count + local_expert_idx * num_ranks + dst_rank); // used in EAGER_FULL

        const auto local_x = static_cast<const int4*>(x) +
                local_expert_idx * num_ranks * num_max_dispatch_tokens_per_rank * hidden_bf16_int4;
        const auto local_src_info = src_info + local_expert_idx * num_ranks * num_max_dispatch_tokens_per_rank;
        const auto rdma_send_x_vec = static_cast<uint8_t*>(rdma_send_x) +
                local_expert_idx * num_ranks * num_max_dispatch_tokens_per_rank * num_bytes_per_slot;

        // Unpack layout
        int offset = 0, num_tokens_to_send, begin_idx, end_idx;
        if (kEager != EAGER_FULL) {
            unpack2(layout, num_tokens_to_send, offset);
        }

        begin_idx = sub_warp_id + offset;
        end_idx = kEager == EAGER_FULL ? num_tokens_from_this_exp_to_dstrank : (offset + num_tokens_to_send);
        int token_idx; // position level index

        // TMA stuffs
        constexpr int kNumTMABufferBytes = sizeof(int4) * 32 * kNumSendUnrolls;
        constexpr int kNumStages = 3;
        constexpr int kNumPrefetch = 1;
        EP_STATIC_ASSERT(kNumStages == 3 and kNumPrefetch == 1, "Invalid stages");

        auto smem_ptr = smem_buffer + warp_id * (kNumStages * (kNumTMABufferBytes + 16) + kNumMetaBytes);
        uint32_t tma_phase = 0;
        auto tma_buffers   = PatternVisitor([=](const int& i) { return reinterpret_cast<int4*>(smem_ptr + i * (kNumTMABufferBytes + 16)); });
        auto full_barriers = PatternVisitor([=](const int& i) { return reinterpret_cast<uint64_t*>(smem_ptr + i * (kNumTMABufferBytes + 16) + kNumTMABufferBytes); });
        auto meta_buffers  = kUseLogFMT ? reinterpret_cast<nv_bfloat162*>(smem_ptr + kNumStages * (kNumTMABufferBytes + 16)) : nullptr;
        EP_STATIC_ASSERT(kNumSendUnrolls * kNumStages <= 12, "TMA buffer size exceed limit");

        // Initialize m-barriers
        if (lane_id < kNumStages) {
            mbarrier_init(full_barriers[lane_id], 1);
            fence_view_async_shared();
            fence_barrier_init();
        }
        __syncwarp();

        constexpr int kNumIters = hidden_bf16_int4_pad / (32 * kNumSendUnrolls);
        auto tma_load_and_arrive = [&](const int& stage_idx, const int4* gmem_ptr, const int& num_bytes) {
            tma_load_1d(tma_buffers[stage_idx], gmem_ptr, full_barriers[stage_idx], num_bytes);
            mbarrier_arrive_and_expect_tx(full_barriers[stage_idx], num_bytes);
        };
        auto get_num_tma_bytes = [&](const int& offset_int4) {
            return min(kNumTMABufferBytes, static_cast<int>((hidden_bf16_int4 - offset_int4) * sizeof(int4)));
        };

        // Issue IBGDA send
        for (int __token_idx = begin_idx; __token_idx < end_idx; __token_idx += num_warps_per_group) { // sequence level index
            if (kEager == EAGER_FULL) {
                token_idx = lane_id == 0 ? __ldg(layout_bitmap_ptr + __token_idx) : 0;
                token_idx = __shfl_sync(0xffffffff, token_idx, 0);
            } else {
                token_idx = __token_idx;
            }
            //EP_DEVICE_ASSERT(token_idx >= 0 && token_idx < num_ranks * num_max_dispatch_tokens_per_rank);
            auto message_idx = __token_idx - offset;

            const auto x_int4 = local_x + token_idx * hidden_bf16_int4;
            const auto rdma_send_type_row = reinterpret_cast<int*>(rdma_send_x_vec + token_idx * num_bytes_per_slot);
            const auto rdma_send_x_vec_row = reinterpret_cast<uint8_t*>(rdma_send_type_row);

            // Copy directly to local rank, or copy to buffer and issue RDMA
            const auto src_idx = __shfl_sync(0xffffffff, __ldg(local_src_info + token_idx), 0);
            const auto buf_ptr = reinterpret_cast<int64_t>(rdma_send_x_vec_row);
            const auto dst_ptr = reinterpret_cast<uint64_t>(rdma_recv_x) + (global_expert_idx * num_max_dispatch_tokens_per_rank + src_idx) * msg_distance;
            const auto dst_p2p_ptr = nvshmemi_get_p2p_ptr(dst_ptr, rank, dst_rank);
            const auto intra_node = (dst_p2p_ptr != 0);
            //if (!intra_node) printf("[rank %d]: an inter-node combine send\n", rank);
            int num_send_bytes = hidden * sizeof(hip_bfloat16);

            // eager mode disable zero copy feature...
            if ((kEager_combine != EAGER_OFF) or (not zero_copy or dst_p2p_ptr != 0)) {
                // Read from `cpy_src_int4_ptr` and copy into `cpy_dst_int4_ptr`
                const auto cpy_src_int4_ptr = (zero_copy && kEager_combine == EAGER_OFF) ? reinterpret_cast<int4*>(buf_ptr) : x_int4;
                const auto cpy_dst_int4_ptr = dst_p2p_ptr == 0 ? reinterpret_cast<int4*>(buf_ptr) : reinterpret_cast<int4*>(dst_p2p_ptr);

                // Prefetch
                if (elect_one_sync(lane_id))
                    tma_load_and_arrive(0, cpy_src_int4_ptr, get_num_tma_bytes(0));
                __syncwarp();

                int tma_offset_bytes = kNumMetaBytes;
                #pragma unroll
                for (int i = lane_id * kNumSendUnrolls, iter_idx = 0; i < hidden_bf16_int4_pad; i += 32 * kNumSendUnrolls, ++ iter_idx) {
                    // Load the next iteration
                    const int& stage_idx = iter_idx % kNumStages;
                    const int& next_stage_idx = (iter_idx + 1) % kNumStages;
                    if (iter_idx + 1 < kNumIters and elect_one_sync(lane_id)) {
                        tma_store_wait<kNumStages - kNumPrefetch - 1>();
                        //const auto& offset_int4 = i + 32 * kNumSendUnrolls;
                        const auto& offset_int4 = (iter_idx + 1) * 32 * kNumSendUnrolls; // why offset_int4 is runtime dynamic?
                        tma_load_and_arrive(next_stage_idx, cpy_src_int4_ptr + offset_int4, get_num_tma_bytes(offset_int4));
                    }
                    __syncwarp();

                    // Wait the current TMA arrival
                    EP_STATIC_ASSERT(kNumStages < 32, "Too many stages");
                    mbarrier_wait<true>(full_barriers[stage_idx], tma_phase, stage_idx);
                    if constexpr (kUseLogFMT) {
                        // Cast if possible
                        constexpr int kNumInt4PerDivision = 128 / kNumElemsPerInt4;
                        int num_tma_bytes = logfmt_encode<kNumSendUnrolls>(
                            tma_buffers[stage_idx],
                            // NOTES: only the leader lane will write the result
                            (i % kNumInt4PerDivision == 0) ? meta_buffers + i / kNumInt4PerDivision : nullptr,
                            lane_id);
                        if (elect_one_sync(lane_id))
                            tma_store_1d(tma_buffers[stage_idx], reinterpret_cast<uint8_t*>(cpy_dst_int4_ptr) + tma_offset_bytes, num_tma_bytes);
                        tma_offset_bytes += num_tma_bytes;
                    } else {
                        // BF16 original values
#define COMBINE_SEND_TMA(tma_func, smem_ptr, gmem_ptr, bytes) TMA_AUTO_TAG(tma_func, smem_ptr, gmem_ptr, bytes, cpy_dst_int4_ptr, __tail_tags, combine_round_n, kEager_combine, intra_node)
                        if ((kEager_combine != EAGER_OFF) ? (lane_id == 0) : elect_one_sync(lane_id)) { // force lane 0 to issue tma
                            auto w_offset_int4 = iter_idx * kNumSendUnrolls * 32; // i is dynamic...
                            COMBINE_SEND_TMA(tma_store_1d, tma_buffers[stage_idx], cpy_dst_int4_ptr + w_offset_int4, get_num_tma_bytes(w_offset_int4));
                            //tma_store_1d(tma_buffers[stage_idx], cpy_dst_int4_ptr + w_offset_int4, get_num_tma_bytes(w_offset_int4));
                        }
                    }
                    __syncwarp();
                }

                // Store metadata (min/max values) for LogFMT
                if constexpr (kUseLogFMT) {
                    num_send_bytes = tma_offset_bytes;
                    if (elect_one_sync(lane_id))
                        tma_store_1d(meta_buffers, cpy_dst_int4_ptr, kNumMetaBytes);
                }

                if constexpr (kEager_combine != EAGER_OFF) {
                    // inter_node: write tag save value
                    
                    if (!intra_node) {
                        // reduce or because tags may be stored in different lanes, maybe there is a better way, why just use lane 0 to do tma? If so, reduce can be replaced by shfl_sync
                        __syncwarp();
                        #pragma unroll
                        for (int __pn = 0; __pn < MAX_PAGES; ++__pn) {
                            //__tail_tags[__pn] = warp_reduce_or(__tail_tags[__pn]);
                            __tail_tags[__pn] = __shfl_sync(0xffffffff, __tail_tags[__pn], 0);
                        }

                        //EP_DEVICE_ASSERT(lane_id == get_lane_id());
                        EP_STATIC_ASSERT((kHidden * sizeof(hip_bfloat16)) % sizeof(int4) == 0, "combine message len (no logfmt) shall be a multiple of int4");
                        if (lane_id < MAX_PAGES_DIV4 + 1) {
                            auto target_ptr = cpy_dst_int4_ptr + hidden_bf16_int4 + lane_id;
                            //EP_DEVICE_ASSERT((target_ptr + 1 - cpy_dst_int4_ptr) * sizeof(int4) <= msg_distance);
                            //printf("[rank %d]: exp %d send back rank %d token %d, put stored value and last tag, offset %lu\n", rank, global_expert_idx, dst_rank, src_idx, (target_ptr - cpy_dst_int4_ptr) * sizeof(int4));
                            // if (lane_id == MAX_PAGES_DIV4) EP_DEVICE_ASSERT(__tail_tags_int4[lane_id].x == ZTAG(combine_round_n));
                            // if (lane_id < MAX_PAGES_DIV4) {
                            //     INT_VALUE_NO_NAN(__tail_tags_int4[lane_id].x);
                            //     INT_VALUE_NO_NAN(__tail_tags_int4[lane_id].y);
                            //     INT_VALUE_NO_NAN(__tail_tags_int4[lane_id].z);
                            //     INT_VALUE_NO_NAN(__tail_tags_int4[lane_id].w);
                            // }
                            st_na_release(target_ptr, __tail_tags_int4[lane_id]);
                        }
                        __syncwarp();
                        num_send_bytes = CEIL_ALIGN(num_send_bytes, sizeof(int4)) + (MAX_PAGES_DIV4 + 1) * sizeof(int4); // extra tag store space
                    }
                }

                // Flush all stores
                tma_store_wait();
                __syncwarp();
                if constexpr (kEager_combine != EAGER_OFF) {
                    if (intra_node && lane_id == 0) {
                        // intranode: only write last tag to signal token ready!
                        st_release_sys_global(reinterpret_cast<int*>(cpy_dst_int4_ptr + hidden_bf16_int4 + MAX_PAGES_DIV4), ZTAG(combine_round_n));
                    }
                }
            }

            // Issue RDMA
            // NOTES: for zero-copy mode, we assume the data is already in the send buffer
            if (dst_p2p_ptr == 0)
                nvshmemi_ibgda_put_nbi_warp(dst_ptr, buf_ptr, num_send_bytes, dst_rank, local_expert_idx, lane_id, message_idx);
        }

        // Put the finishing flag
        EP_DEVICE_ASSERT(num_warps_per_group > 1 and num_warp_groups < 16);
        asm volatile("bar.sync %0, %1;" :: "r"(warp_group_id + 1), "r"(num_warps_per_group * 32));
        if (sub_warp_id == 1 and lane_id == 0) {
            if (kEager_combine != EAGER_FULL) {
                while (ld_acquire_global(atomic_clean_flag) == 0);
            }
            auto dst_ptr = reinterpret_cast<uint64_t>(rdma_recv_flag + global_expert_idx);
            auto dst_p2p_ptr = nvshmemi_get_p2p_ptr(dst_ptr, rank, dst_rank);
            if (dst_p2p_ptr == 0) {
                if (kEager <= EAGER_OFF) {
                    nvshmemi_ibgda_amo_nonfetch_add(reinterpret_cast<int*>(dst_ptr), 1, dst_rank, local_expert_idx);
                } else {
                    nvshmemi_ibgda_rma_p(reinterpret_cast<int*>(dst_ptr), 1, dst_rank, local_expert_idx);
                }
            } else {
                st_release_sys_global(reinterpret_cast<int*>(dst_p2p_ptr), 1);
            }
            if (kEager_combine != EAGER_FULL) {
                atomic_add_release_global(atomic_clean_flag, -1);
            }
        }
        __syncwarp();

        // Destroy m-barriers
        if (lane_id < kNumStages) {
            mbarrier_inval(full_barriers[lane_id]);
            fence_view_async_shared();
            fence_barrier_init();
        }
        __syncwarp();
    }

    // Receiving phase
    LOW_LATENCY_COMBINE_RECV:
    if ((phases & LOW_LATENCY_RECV_PHASE) == 0)
        return;
    
    if constexpr (kEager_combine != EAGER_FULL) {
        // Wait all ranks to arrive
        if (responsible_expert_idx < num_experts) {
            EP_DEVICE_ASSERT(num_warps_per_group > 1);
            if (sub_warp_id == 0 and lane_id == 0) {
                auto start_time = clock64();
                while (ld_acquire_sys_global(rdma_recv_flag + responsible_expert_idx) == 0);
                auto wait_recv_cost = clock64() - start_time;
                if (combine_wait_recv_cost_stats != nullptr) {
                    const auto& src_rank = responsible_expert_idx / num_local_experts;
                    atomicAdd(reinterpret_cast<unsigned long long*>(combine_wait_recv_cost_stats + src_rank), wait_recv_cost);
                }
            }
        }
        cg::this_grid().sync();
    } else {
        __syncthreads(); // two types of recv warp must be synchronized
    }
    

    // Reassign warp groups
    constexpr int kMaxNumGroups = 2;
    const int num_decode_warps = hidden_bf16_int4_pad / (kNumRecvUnrolls * 32);
    const int num_groups = min(kMaxNumGroups, (num_threads / 32) / (num_decode_warps + 1));
    const int decode_warp_idx = __shfl_sync(0xffffffff, warp_id % (num_decode_warps + 1), 0);
    const int group_idx = __shfl_sync(0xffffffff, warp_id / (num_decode_warps + 1), 0);
    EP_STATIC_ASSERT(kHidden % (32 * kNumElemsPerInt4) == 0, "Invalid vectorization");
    EP_DEVICE_ASSERT(num_topk <= 32);
    EP_DEVICE_ASSERT(num_groups > 0);

    if (group_idx < num_groups) {
        constexpr int kNumStages = 3;
        constexpr int kNumTMABufferBytes = 16 * 2 + kHidden * 2;
        constexpr int kNumBF16PerWarpBytes = 32 * kNumRecvUnrolls * kNumElemsPerInt4 * 2;
        constexpr int kNumLogFMTPerWarpBytes = kNumBF16PerWarpBytes / 16 * 10;
        constexpr int kNumDivisionBytes = kNumDivisions * sizeof(uint32_t);
        constexpr int kNumBytesPerGroup = kNumStages * kNumTMABufferBytes + kHidden * 2 + kNumStages * kNumDivisionBytes * 3;

        __shared__ int4 ld_tags_int4[kMaxNumGroups][kNumStages][MAX_PAGES_DIV4];

        // Reallocate shared memory
        const auto smem_group_buffer = smem_buffer + kNumBytesPerGroup * group_idx;
        auto full_barriers  = PatternVisitor([=](const int& i) { return reinterpret_cast<uint64_t*>(smem_group_buffer + i * kNumTMABufferBytes); });
        auto empty_barriers = PatternVisitor([=](const int& i) { return reinterpret_cast<uint64_t*>(smem_group_buffer + i * kNumTMABufferBytes + 8); });
        auto tma_ld_buffers = PatternVisitor([=](const int& i) { return reinterpret_cast<uint8_t* >(smem_group_buffer + i * kNumTMABufferBytes + 16); });
        auto tma_st_buffers = PatternVisitor([=](const int& i) { return reinterpret_cast<uint32_t*>(smem_group_buffer + kNumStages * kNumTMABufferBytes + i * kNumBF16PerWarpBytes); });

        // Redundant when logfmt is disabled
        const auto smem_group_ptr = smem_group_buffer + kNumStages * kNumTMABufferBytes + kHidden * 2;
        auto log_amax_buffers  = PatternVisitor([=](const int& i) { return reinterpret_cast<float*>(smem_group_ptr + i * kNumDivisionBytes); });
        auto log_amin_buffers  = PatternVisitor([=](const int& i) { return reinterpret_cast<float*>(smem_group_ptr + kNumStages * kNumDivisionBytes + i * kNumDivisionBytes); });
        auto cast_info_buffers = PatternVisitor([=](const int& i) { return reinterpret_cast<int*>  (smem_group_ptr + kNumStages * kNumDivisionBytes * 2 + i * kNumDivisionBytes); });

        uint32_t tma_phase = 0;
        EP_STATIC_ASSERT(kNumStages < 32, "Too many stages");
        if (decode_warp_idx == num_decode_warps)
            tma_phase = (1 << kNumStages) - 1;

        // Initialize m-barriers
        if (decode_warp_idx == num_decode_warps and lane_id < kNumStages) {
            mbarrier_init(full_barriers[lane_id], 1);
            mbarrier_init(empty_barriers[lane_id], num_decode_warps);
        }
        asm volatile("bar.sync %0, %1;" :: "r"(group_idx + 1), "r"((num_decode_warps + 1) * 32));

        int stage_idx = 0, topk_idx_by_lane = 0;
        EP_STATIC_ASSERT(kNumMaxTopk <= 32, "Invalid number of topks");
        if (decode_warp_idx == num_decode_warps) {
            // TMA load warp
            //if (sm_id == 0 && rank == 0 && group_idx == 0 && lane_id == 0) printf("tma load warp working\n");
            for (int token_idx = sm_id + num_sms * group_idx; token_idx < num_combined_tokens; token_idx += num_sms * num_groups) {
                if (lane_id < num_topk)
                    topk_idx_by_lane = static_cast<int>(__ldg(topk_idx + token_idx * num_topk + lane_id));
                for (int i = 0; i < num_topk; ++ i) {
                    int topk_idx_reg = __shfl_sync(0xffffffff, topk_idx_by_lane, i);
                    if (topk_idx_reg < 0)
                        continue;
                    const int src_rank = topk_idx_reg / num_local_experts;
                    const auto intra_node = (src_rank >> 3) == (rank >> 3);
                    //if (!intra_node) printf("[rank %d]: an inter-node combine recv\n", rank);
                    //if (sm_id == 0 && rank == 0 && group_idx == 0 && lane_id == 0) printf("tma load warp waiting on mbarrier\n");
                    mbarrier_wait<true>(empty_barriers[stage_idx], tma_phase, stage_idx);
                    auto buffer = static_cast<uint8_t*>(rdma_recv_x) + (topk_idx_reg * num_max_dispatch_tokens_per_rank + token_idx) * msg_distance;
                    //if (sm_id == 0 && rank == 0 && group_idx == 0 && lane_id == 0) printf("tma load warp pass mbarrier\n");
                    if constexpr (kEager_combine != EAGER_OFF) {
                        constexpr int pages = CEIL_DIV(kHidden * sizeof(hip_bfloat16), PCIE_SEG_LEN);
                        constexpr int last_page_part = (kHidden * sizeof(hip_bfloat16)) & PCIE_SEG_LEN_MASK;
                        constexpr int full_last_page = last_page_part == 0 ? 1 : 0;
                        if (intra_node) {
                            // intra node: just check tail tag
                            if (lane_id == 0) {
                                int *__check_ptr = reinterpret_cast<int*>(buffer + kHidden * sizeof(hip_bfloat16) + MAX_PAGES_DIV4 * sizeof(int4));
                                while (ld_acquire_sys_global(__check_ptr) != ZTAG(combine_round_n));
                            }
                        } else {
                            // inter node: check tags of all pages
                            #pragma unroll
                            for (int __pn = lane_id; __pn < pages + full_last_page; __pn += 32) {
                                int *__check_ptr = reinterpret_cast<int*>(buffer + (__pn << PCIE_SEG_LEN_LOG) + ((__pn == pages + full_last_page - 1) ? (last_page_part + MAX_PAGES_DIV4 * sizeof(int4)) : (PCIE_SEG_LEN - PCIE_TAIL_SZ)));
                                //EP_DEVICE_ASSERT(reinterpret_cast<uint8_t*>(__check_ptr) - buffer + sizeof(int) <= msg_distance);
                                //printf("[rank %d]: token %d topk %d from exp %d at rank %d, check offset %lu\n", rank, token_idx, i, topk_idx_reg, src_rank, reinterpret_cast<uint8_t*>(__check_ptr) - buffer);
                                while (ld_acquire_sys_global(__check_ptr) != ZTAG(combine_round_n));
                                //printf("[rank %d]: token %d topk %d from exp %d at rank %d, check offset %lu, OK\n", rank, token_idx, i, topk_idx_reg, src_rank, reinterpret_cast<uint8_t*>(__check_ptr) - buffer);
                            }
                        }
                        __syncwarp();
                        if (!intra_node) {
                            if (lane_id < MAX_PAGES_DIV4) {
                                auto tag_pos = buffer + sizeof(hip_bfloat16) * kHidden + lane_id * sizeof(int4);
                                //EP_DEVICE_ASSERT(tag_pos - buffer + sizeof(int4) <= msg_distance);
                                st_release_cta(&ld_tags_int4[group_idx][stage_idx][lane_id], ld_nc_global(reinterpret_cast<int4*>(tag_pos)));
                            }
                        }
                        __syncwarp();
                    }
                    if constexpr (kUseLogFMT) {
                        logfmt_check_amaxmin<kNumDivisions / 2, kNumSendUnrolls, kNumRecvUnrolls>(
                            buffer, reinterpret_cast<float2*>(log_amax_buffers[stage_idx]),
                            reinterpret_cast<float2*>(log_amin_buffers[stage_idx]), cast_info_buffers[stage_idx], lane_id);
                    }
                    //if (sm_id == 0 && rank == 0 && group_idx == 0 && lane_id == 0) printf("tma load warp launching tma load\n");
                    if (elect_one_sync(lane_id)) {
                        int num_casted = 0;
                        if constexpr (kUseLogFMT) {
                            const auto& info = cast_info_buffers[stage_idx][num_decode_warps - 1];
                            num_casted = (info >> 1) + (info & 1);
                        }
                        int num_tma_bytes = num_casted * kNumLogFMTPerWarpBytes + (num_decode_warps - num_casted) * kNumBF16PerWarpBytes;
                        tma_load_1d(tma_ld_buffers[stage_idx], buffer + (kUseLogFMT ? kNumMetaBytes : 0), full_barriers[stage_idx], num_tma_bytes);
                        mbarrier_arrive_and_expect_tx(full_barriers[stage_idx], num_tma_bytes);
                    }
                    //if (sm_id == 0 && rank == 0 && group_idx == 0 && lane_id == 0) printf("tma load warp started tma load\n");
                    // if constexpr (kEager != EAGER_OFF) {
                    //     // inter node: tag postion shall be restored, load them first
                    //     if (!intra_node && lane_id < MAX_PAGES_DIV4) {
                    //         auto tag_pos = buffer + sizeof(hip_bfloat16) * kHidden + lane_id * sizeof(int4);
                    //         EP_DEVICE_ASSERT(tag_pos - buffer + sizeof(int4) <= msg_distance);
                    //         ld_tags_int4[group_idx][stage_idx][lane_id] = ld_nc_global(reinterpret_cast<int4*>(tag_pos));
                    //     }
                    // }
                    __syncwarp();
                    stage_idx = (stage_idx + 1) % kNumStages;
                    //if (sm_id == 0 && rank == 0 && group_idx == 0 && lane_id == 0) printf("tma load warp loop end\n");
                }
            }
        } else {
            // Reduction warps
            //if (sm_id == 0 && rank == 0 && group_idx == 0 && lane_id == 0 && decode_warp_idx == 0) printf("reduce warp working\n");
            float topk_weights_by_lane;
            for (int token_idx = sm_id + num_sms * group_idx; token_idx < num_combined_tokens; token_idx += num_sms * num_groups) {
                if (lane_id < num_topk) {
                    topk_idx_by_lane = static_cast<int>(__ldg(topk_idx + token_idx * num_topk + lane_id));
                    topk_weights_by_lane = __ldg(topk_weights + token_idx * num_topk + lane_id);
                }
                __syncwarp();
                //if (sm_id == 0 && rank == 0 && group_idx == 0 && lane_id == 0 && decode_warp_idx == 0) printf("reduce warp got topk info\n");
                float combined_values[kNumElemsPerInt4 * kNumRecvUnrolls] = {0.0f};
                for (int i = 0; i < num_topk; ++ i) {
                    const auto topk_idx = __shfl_sync(0xffffffff, topk_idx_by_lane, i);
                    if (topk_idx < 0)
                        continue;
                    const auto& topk_weight = __shfl_sync(0xffffffff, topk_weights_by_lane, i);
                    const auto intra_node = ((topk_idx / num_local_experts) >> 3) == (rank >> 3);
                    //if (sm_id == 0 && rank == 0 && group_idx == 0 && lane_id == 0 && decode_warp_idx == 0) printf("reduce warp wait on barrier\n");
                    mbarrier_wait<true>(full_barriers[stage_idx], tma_phase, stage_idx);
                    //if (sm_id == 0 && rank == 0 && group_idx == 0 && lane_id == 0 && decode_warp_idx == 0) printf("reduce warp pass barrier\n");
                    if constexpr (kEager_combine != EAGER_OFF) {
                        auto buffer = static_cast<uint8_t*>(rdma_recv_x) + (topk_idx * num_max_dispatch_tokens_per_rank + token_idx) * msg_distance;
                        constexpr int pages = (kHidden * sizeof(hip_bfloat16)) >> PCIE_SEG_LEN_LOG;
                        if (!intra_node) {
                            if (decode_warp_idx == 0 && lane_id < pages) {
                                //EP_DEVICE_ASSERT((lane_id << PCIE_SEG_LEN_LOG) + (PCIE_SEG_LEN - PCIE_TAIL_SZ) < kHidden * sizeof(hip_bfloat16) - sizeof(int));
                                // int space;
                                // hip_bfloat16* space_ptr = reinterpret_cast<hip_bfloat16*>(&space);
                                // space_ptr[0] = static_cast<hip_bfloat16>(rank - 128);
                                // space_ptr[1] = space_ptr[0];
                                //EP_DEVICE_ASSERT(reinterpret_cast<int*>(ld_tags_int4[group_idx][stage_idx])[lane_id] == space);
                                //printf("[rank %d]: debug: wriring shmem\n", rank);
                                int save_value = ld_nc_global(reinterpret_cast<int*>(buffer + kHidden * sizeof(hip_bfloat16) + lane_id * sizeof(int))); // reinterpret_cast<int*>(ld_tags_int4[group_idx][stage_idx])[lane_id]
                                //INT_VALUE_NO_NAN(save_value);
                                st_release_cta(reinterpret_cast<int*>(tma_ld_buffers[stage_idx] + (lane_id << PCIE_SEG_LEN_LOG) + (PCIE_SEG_LEN - PCIE_TAIL_SZ)), save_value);
                            }
                            asm volatile("bar.sync %0, %1;" :: "r"(group_idx + 2), "r"(num_decode_warps * 32));
                        }
                    }
                    //if (sm_id == 0 && rank == 0 && group_idx == 0 && lane_id == 0 && decode_warp_idx == 0) printf("reduce warp begin reduce\n");
                    if constexpr (kUseLogFMT) {
                        const auto& info = cast_info_buffers[stage_idx][decode_warp_idx];
                        bool enable_cast = info & 1;
                        int num_casted_prefix = info >> 1;
                        int tma_offset = kNumLogFMTPerWarpBytes * num_casted_prefix + kNumBF16PerWarpBytes * (decode_warp_idx - num_casted_prefix);
                        int division_idx = decode_warp_idx * (kNumRecvUnrolls * 2) + lane_id * kNumRecvUnrolls / 16;
                        decode_and_accumulate<kNumRecvUnrolls>(
                            reinterpret_cast<uint32_t*>(tma_ld_buffers[stage_idx] + tma_offset + (enable_cast ? kNumLogFMTPerWarpBytes : kNumBF16PerWarpBytes) / 32 * lane_id),
                            combined_values, log_amax_buffers[stage_idx][division_idx], log_amin_buffers[stage_idx][division_idx], enable_cast, topk_weight
                        );
                    } else {
                        int tma_offset = kNumBF16PerWarpBytes * decode_warp_idx;
                        decode_and_accumulate<kNumRecvUnrolls>(
                            reinterpret_cast<uint32_t*>(tma_ld_buffers[stage_idx] + tma_offset + kNumBF16PerWarpBytes / 32 * lane_id),
                            combined_values, 0, 0, false, topk_weight
                        );
                    }
                    //if (sm_id == 0 && rank == 0 && group_idx == 0 && lane_id == 0 && decode_warp_idx == 0) printf("reduce warp finish reduce\n");
                    if (elect_one_sync(lane_id))
                        mbarrier_arrive(empty_barriers[stage_idx]);
                    stage_idx = (stage_idx + 1) % kNumStages;
                }
                //asm volatile("bar.sync %0, %1;" :: "r"(group_idx + 2), "r"(num_decode_warps * 32));
                //if (sm_id == 0 && rank == 0 && group_idx == 0 && lane_id == 0 && decode_warp_idx == 0) printf("reduce warp finish all reduce, waiting for tma store done\n");

                tma_store_wait<0>();

                //if (sm_id == 0 && rank == 0 && group_idx == 0 && lane_id == 0 && decode_warp_idx == 0) printf("reduce warp finish all reduce, load reduce result to share mem\n");
                #pragma unroll
                for (int k = 0; k < kNumRecvUnrolls * 4; ++ k) {
                    auto combined_pack = __hip_bfloat162(combined_values[k * 2], combined_values[k * 2 + 1]);
                    tma_st_buffers[decode_warp_idx][kNumRecvUnrolls * 4 * lane_id + k] = *reinterpret_cast<uint32_t*>(&combined_pack);
                }
                tma_store_fence();
                //if (sm_id == 0 && rank == 0 && group_idx == 0 && lane_id == 0 && decode_warp_idx == 0) printf("reduce warp finish all reduce, start tma store\n");
                if (elect_one_sync(lane_id)) {
                    tma_store_1d(tma_st_buffers[decode_warp_idx],
                                 static_cast<int4*>(combined_x) + token_idx * hidden_bf16_int4 + decode_warp_idx * kNumRecvUnrolls * 32,
                                 kNumBF16PerWarpBytes);
                }
                __syncwarp();
                //if (sm_id == 0 && rank == 0 && group_idx == 0 && lane_id == 0 && decode_warp_idx == 0) printf("reduce warp finish a token\n");
            }
        }

        // Flush all stores
        tma_store_wait<0>();
    }
}

void combine(void* combined_x,
             void* rdma_recv_x, int* rdma_recv_flag, void* rdma_send_x,
             const void* x, const int64_t* topk_idx, const float* topk_weights,
             const int* per_rank_src_count,
             const int* src_info, const int64_t* layout_range,
             int64_t* combine_wait_recv_cost_stats,
             int* next_clean, int num_next_clean_int,
             int num_combined_tokens, int hidden, int num_max_dispatch_tokens_per_rank,
             int num_topk, int num_experts, int rank, int num_ranks,
             bool use_logfmt,
             void* workspace, int num_device_sms,
             hipStream_t stream, int phases, bool zero_copy,
             int combine_round_n, const int eager_opt) {
    constexpr int kNumMaxTopk = 9;
    const int num_warp_groups = ceil_div(num_experts, num_device_sms);
    const int num_warps_per_group = 32 / num_warp_groups;
    const int num_recv_per_sm = ceil_div(num_combined_tokens, num_device_sms);
    EP_HOST_ASSERT(num_warp_groups > 0 and num_warps_per_group > 0 and num_recv_per_sm > 0);

    const auto num_warps = num_warp_groups * num_warps_per_group;
    const auto num_sms = max(ceil_div(num_experts, num_warp_groups), ceil_div(num_combined_tokens, num_recv_per_sm));

    // Check workspace
    auto atomic_clean_flag = static_cast<int*>(workspace);
    EP_HOST_ASSERT(sizeof(int) <= NUM_WORKSPACE_BYTES);
    EP_HOST_ASSERT(num_topk <= kNumMaxTopk);

    // Online cast cannot use zero-copy
    EP_HOST_ASSERT(not (zero_copy and use_logfmt));

    constexpr int kNumStages = 3;
    constexpr int kNumMaxUnrolls = 4;
    constexpr int kMaxNumGroups = 2;

    // Send buffer size
    const int num_meta_bytes = hidden / 128 * 4;
    const int num_send_tma_bytes = 32 * sizeof(int4) * kNumMaxUnrolls + 16;
    const int smem_send_size = num_warps * (kNumStages * num_send_tma_bytes + num_meta_bytes);

    // Receive buffer size
    const int num_recv_tma_bytes = 16 + hidden * 2;
    const int smem_recv_size = kMaxNumGroups * (kNumStages * num_recv_tma_bytes + hidden * 2 + kNumStages * num_meta_bytes * 3);

    // Total requirement
    const int smem_size = max(smem_send_size, smem_recv_size);

#define COMBINE_LAUNCH_CASE(eager_opt, hidden) { \
auto combine_func = use_logfmt ? \
    combine<true, hidden, kNumMaxTopk, kNumMaxUnrolls, EAGER_OFF> : \
    combine<false, hidden, kNumMaxTopk, kNumMaxUnrolls, eager_opt>; \
SET_SHARED_MEMORY_FOR_TMA(combine_func); \
LAUNCH_KERNEL(&cfg, combine_func, \
              combined_x, \
              rdma_recv_x, rdma_recv_flag, rdma_send_x, \
              x, topk_idx, topk_weights, per_rank_src_count, src_info, layout_range, \
              combine_wait_recv_cost_stats, \
              next_clean, num_next_clean_int, \
              atomic_clean_flag, \
              num_combined_tokens, hidden, num_topk, \
              num_max_dispatch_tokens_per_rank, \
              num_experts, rank, num_ranks, \
              num_warp_groups, num_warps_per_group, \
              phases, zero_copy, combine_round_n); } break

    SETUP_LAUNCH_CONFIG(num_sms, num_warps * 32, stream);

    #define EAGER_COMBINE_LAUNCH_CASE(hidden) SWITCH_EAGER(COMBINE_LAUNCH_CASE, hidden)

    SWITCH_HIDDEN(EAGER_COMBINE_LAUNCH_CASE);
#undef COMBINE_LAUNCH_CASE
}

} // namespace internode_ll

} // namespace deep_ep
